#include "hip/hip_runtime.h"
/*
 * X17 algorithm built on cbuchner1's original X11
 *
 */

extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"

#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"

#include "sph/sph_sha2.h"
#include "sph/sph_haval.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];
static uint32_t endiandata[MAX_GPUS][20];


extern void quark_blake512_cpu_init(int thr_id);
extern void quark_blake512_cpu_setBlock_80(uint64_t *pdata);
extern void quark_blake512_cpu_setBlock_80_multi(uint32_t thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_bmw512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_groestl512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void cuda_jh512Keccak512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_luffaCubehash512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t luffacubethreads);

extern void x11_shavite512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t shavitethreads);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const uint32_t simdthreads);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x13_hamsi512_cpu_init(int thr_id, uint32_t threads);
extern void x13_hamsi512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int flag);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x17_sha512_cpu_init(int thr_id, uint32_t threads);
extern void x17_sha512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce,uint32_t *d_hash);

extern void x17_haval256_cpu_init(int thr_id, uint32_t threads);
extern void x17_haval256_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes,
											uint32_t *d_noncesTrue, uint32_t *nrmTrue, uint32_t *d_noncesFalse, uint32_t *nrmFalse);

// Xevan
extern "C" void xevan_hash(void *output, const void *input)
{
	// blake1-bmw2-grs3-skein4-jh5-keccak6-luffa7-cubehash8-shavite9-simd10-echo11-hamsi12-fugue13-shabal14-whirlpool15-sha512-haval17

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_fugue512_context ctx_fugue;
	sph_shabal512_context ctx_shabal;
	sph_whirlpool_context ctx_whirlpool;
	sph_sha512_context ctx_sha512;
	sph_haval256_5_context ctx_haval;

	uint32_t hashA[32], hashB[32];
	memset(hashA , 0, 128);
	memset(hashB , 0, 128);

    sph_blake512_init(&ctx_blake);
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close (&ctx_blake, hashA);

    sph_bmw512_init(&ctx_bmw);
    sph_bmw512 (&ctx_bmw, hashA, 128);
    sph_bmw512_close(&ctx_bmw, hashB);

    sph_groestl512_init(&ctx_groestl);
    sph_groestl512 (&ctx_groestl, hashB, 128);
    sph_groestl512_close(&ctx_groestl, hashA);

    sph_skein512_init(&ctx_skein);
    sph_skein512 (&ctx_skein, hashA, 128);
    sph_skein512_close (&ctx_skein, hashB);

    sph_jh512_init(&ctx_jh);
    sph_jh512 (&ctx_jh, hashB, 128);
    sph_jh512_close(&ctx_jh, hashA);

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, hashA, 128);
    sph_keccak512_close(&ctx_keccak, hashB);

    sph_luffa512_init (&ctx_luffa1);
    sph_luffa512 (&ctx_luffa1, hashB, 128);
    sph_luffa512_close (&ctx_luffa1, hashA);

    sph_cubehash512_init (&ctx_cubehash1);
    sph_cubehash512 (&ctx_cubehash1, hashA, 128);
    sph_cubehash512_close(&ctx_cubehash1, hashB);

    sph_shavite512_init (&ctx_shavite1);
    sph_shavite512 (&ctx_shavite1, hashB, 128);
    sph_shavite512_close(&ctx_shavite1, hashA);

    sph_simd512_init (&ctx_simd1);
    sph_simd512 (&ctx_simd1, hashA, 128);
    sph_simd512_close(&ctx_simd1, hashB);

    sph_echo512_init (&ctx_echo1);
    sph_echo512 (&ctx_echo1, hashB, 128);
    sph_echo512_close(&ctx_echo1, hashA);
	
	
	sph_hamsi512_init(&ctx_hamsi);
    sph_hamsi512 (&ctx_hamsi, hashA, 128);
    sph_hamsi512_close(&ctx_hamsi, hashB);

    sph_fugue512_init(&ctx_fugue);
    sph_fugue512 (&ctx_fugue, hashB, 128);
    sph_fugue512_close(&ctx_fugue, hashA);

    sph_shabal512_init(&ctx_shabal);
    sph_shabal512 (&ctx_shabal, hashA, 128);
    sph_shabal512_close(&ctx_shabal, hashB);

    sph_whirlpool_init(&ctx_whirlpool);
    sph_whirlpool (&ctx_whirlpool, hashB, 128);
    sph_whirlpool_close(&ctx_whirlpool, hashA);

    sph_sha512_init(&ctx_sha2);
    sph_sha512 (&ctx_sha2, hashA, 128);
    sph_sha512_close(&ctx_sha2, hashB);

    sph_haval256_5_init(&ctx_haval);
    sph_haval256_5 (&ctx_haval, hashB, 128);
    sph_haval256_5_close(&ctx_haval, hashA);
	memset(&hashA[8], 0, 128 - 32);
	
	
    ///  Part2
    sph_blake512_init(&ctx_blake);
    sph_blake512 (&ctx_blake, hashA, 128);
    sph_blake512_close(&ctx_blake, hashB);
    
    sph_bmw512_init(&ctx_bmw);
    sph_bmw512 (&ctx_bmw, hashB, 128);
    sph_bmw512_close(&ctx_bmw, hashA);

    sph_groestl512_init(&ctx_groestl);
    sph_groestl512 (&ctx_groestl, hashA, 128);
    sph_groestl512_close(&ctx_groestl, hashB);

    sph_skein512_init(&ctx_skein);
    sph_skein512 (&ctx_skein, hashB, 128);
    sph_skein512_close(&ctx_skein, hashA);
    
    sph_jh512_init(&ctx_jh);
    sph_jh512 (&ctx_jh, hashA, 128);
    sph_jh512_close(&ctx_jh, hashB);
    
    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, hashB, 128);
    sph_keccak512_close(&ctx_keccak, hashA);

    sph_luffa512_init(&ctx_luffa1);
    sph_luffa512 (&ctx_luffa1, hashA, 128);
    sph_luffa512_close(&ctx_luffa1, hashB);
    
    sph_cubehash512_init(&ctx_cubehash1);
    sph_cubehash512 (&ctx_cubehash1, hashB, 128);
    sph_cubehash512_close(&ctx_cubehash1, hashA);
    
    sph_shavite512_init(&ctx_shavite1);
    sph_shavite512(&ctx_shavite1, hashA, 128);
    sph_shavite512_close(&ctx_shavite1, hashB);
        
    sph_simd512_init(&ctx_simd1);
    sph_simd512 (&ctx_simd1, hashB, 128);
    sph_simd512_close(&ctx_simd1, hashA);

    sph_echo512_init(&ctx_echo1);
    sph_echo512 (&ctx_echo1, hashA, 128);
    sph_echo512_close(&ctx_echo1, hashB);

    sph_hamsi512_init(&ctx_hamsi);
    sph_hamsi512 (&ctx_hamsi, hashB, 128);
    sph_hamsi512_close(&ctx_hamsi, hashA);

    sph_fugue512_init(&ctx_fugue);
    sph_fugue512 (&ctx_fugue, hashA, 128);
    sph_fugue512_close(&ctx_fugue, hashB);

    sph_shabal512_init(&ctx_shabal);
    sph_shabal512 (&ctx_shabal, hashB, 128);
    sph_shabal512_close(&ctx_shabal, hashA);

    sph_whirlpool_init(&ctx_whirlpool);
    sph_whirlpool (&ctx_whirlpool, hashA, 128);
    sph_whirlpool_close(&ctx_whirlpool, hashB);

    sph_sha512_init(&ctx_sha2);
    sph_sha512 (&ctx_sha2, hashB, 128);
    sph_sha512_close(&ctx_sha2, hashA);

    sph_haval256_5_init(&ctx_haval);
    sph_haval256_5 (&ctx_haval, hashA, 128);
    sph_haval256_5_close(&ctx_haval, hashB);


    memcpy(output, hashB, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_x17(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	int intensity = 256 * 256 * 13;
	if (device_sm[device_map[thr_id]] == 520)  intensity = 256 * 256 * 22;
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 19=256*256*8;
	throughput = min(throughput, (max_nonce - first_nonce));
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 64 : 32;
	uint32_t shavitethreads = (device_sm[device_map[thr_id]] == 500) ? 256 : 320;
	uint32_t luffacubehashthreads = (device_sm[device_map[thr_id]] == 500) ? 512 : 256;

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0xff;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);

		quark_skein512_cpu_init(thr_id);
		x11_simd512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);
		x13_hamsi512_cpu_init(thr_id, throughput);
		x13_fugue512_cpu_init(thr_id, throughput);
		x15_whirlpool_cpu_init(thr_id, throughput, 0);
		x17_sha512_cpu_init(thr_id, throughput);
		x17_haval256_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput), 0);
		quark_blake512_cpu_init(thr_id);

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[thr_id][k], ((uint32_t*)pdata)[k]);

	if (opt_n_gputhreads > 1)
	{
		quark_blake512_cpu_setBlock_80_multi(thr_id, (uint64_t *)endiandata[thr_id]);
	}
	else
	{
		quark_blake512_cpu_setBlock_80((uint64_t *)endiandata[thr_id]);
	}
	cuda_check_cpu_setTarget(ptarget);

	do {

		quark_blake512_cpu_hash_80(throughput, pdata[19], d_hash[thr_id]);
		quark_bmw512_cpu_hash_64(throughput, pdata[19], NULL, d_hash[thr_id]);
		quark_groestl512_cpu_hash_64(throughput, pdata[19], NULL, d_hash[thr_id]);
		quark_skein512_cpu_hash_64(throughput, pdata[19], NULL, d_hash[thr_id]);
		cuda_jh512Keccak512_cpu_hash_64(throughput, pdata[19], d_hash[thr_id]);

		x11_luffaCubehash512_cpu_hash_64(throughput, pdata[19], d_hash[thr_id], luffacubehashthreads);
		x11_shavite512_cpu_hash_64(throughput, pdata[19], d_hash[thr_id], shavitethreads);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], simdthreads);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x13_hamsi512_cpu_hash_64(throughput, pdata[19], d_hash[thr_id]);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x17_sha512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x17_haval256_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);

		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			/* check now with the CPU to confirm */
			be32enc(&endiandata[thr_id][19], foundNonce);
			x17hash(vhash64, endiandata[thr_id]);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], foundNonce);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (secNonce != 0) {
					pdata[21] = secNonce;
					res++;
				}
				if (opt_benchmark) applog(LOG_INFO, "found nounce", thr_id, foundNonce, vhash64[7], Htarg);
				pdata[19] = foundNonce;
				return res;
			}
			else
			{
				applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}

		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
