#include "hip/hip_runtime.h"
/**
 * This code compares final hash against target
 */
#include <stdio.h>
#include <memory.h>

#include "miner.h"

#include "cuda_helper.h"

__constant__ uint32_t pTarget[8]; // 32 bytes

// store MAX_GPUS device arrays of 8 nonces
<<<<<<< HEAD
static uint32_t* h_resNonces[MAX_GPUS] = { NULL };
static uint32_t* d_resNonces[MAX_GPUS] = { NULL };
static __thread bool init_done = false;
=======
static uint32_t* h_resNonces[MAX_GPUS];
static uint32_t* d_resNonces[MAX_GPUS];
>>>>>>> 8c320ca... added xevan

__host__
void cuda_check_cpu_init(int thr_id, uint32_t threads)
{
<<<<<<< HEAD
    CUDA_CALL_OR_RET(hipMalloc(&d_resNonces[thr_id], 32));
    CUDA_SAFE_CALL(hipHostMalloc(&h_resNonces[thr_id], 32));
    init_done = true;
}

__host__
void cuda_check_cpu_free(int thr_id)
{
	if (!init_done) return;
	hipFree(d_resNonces[thr_id]);
	hipHostFree(h_resNonces[thr_id]);
	d_resNonces[thr_id] = NULL;
	h_resNonces[thr_id] = NULL;
	init_done = false;
=======
    CUDA_CALL_OR_RET(hipHostMalloc(&h_resNonces[thr_id], 8*sizeof(uint32_t)));
	CUDA_CALL_OR_RET(hipMalloc(&d_resNonces[thr_id], 8 * sizeof(uint32_t)));
>>>>>>> 8c320ca... added xevan
}

// Target Difficulty
__host__
void cuda_check_cpu_setTarget(const void *ptarget)
{
<<<<<<< HEAD
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 32, 0, hipMemcpyHostToDevice));
=======
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
>>>>>>> 8c320ca... added xevan
}

/* --------------------------------------------------------------------------------------------- */

__device__ __forceinline__
static bool hashbelowtarget(const uint32_t *const __restrict__ hash, const uint32_t *const __restrict__ target)
{
	if (hash[7] > target[7])
		return false;
	if (hash[7] < target[7])
		return true;
	if (hash[6] > target[6])
		return false;
	if (hash[6] < target[6])
		return true;

	if (hash[5] > target[5])
		return false;
	if (hash[5] < target[5])
		return true;
	if (hash[4] > target[4])
		return false;
	if (hash[4] < target[4])
		return true;

	if (hash[3] > target[3])
		return false;
	if (hash[3] < target[3])
		return true;
	if (hash[2] > target[2])
		return false;
	if (hash[2] < target[2])
		return true;

	if (hash[1] > target[1])
		return false;
	if (hash[1] < target[1])
		return true;
	if (hash[0] > target[0])
		return false;

	return true;
}

__global__ __launch_bounds__(512, 4)
void cuda_checkhash_64(uint32_t threads, uint32_t startNounce, uint32_t *hash, uint32_t *resNonces)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// shl 4 = *16 x 4 (uint32) = 64 bytes
		// todo: use only 32 bytes * threads if possible
		uint32_t *inpHash = &hash[thread << 4];

		if (resNonces[0] == UINT32_MAX) {
			if (hashbelowtarget(inpHash, pTarget))
				resNonces[0] = (startNounce + thread);
		}
	}
}

<<<<<<< HEAD
__global__ __launch_bounds__(512, 4)
void cuda_checkhash_32(uint32_t threads, uint32_t startNounce, uint32_t *hash, uint32_t *resNonces)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *inpHash = &hash[thread << 3];

		if (resNonces[0] == UINT32_MAX) {
			if (hashbelowtarget(inpHash, pTarget))
				resNonces[0] = (startNounce + thread);
		}
	}
}

=======
>>>>>>> 8c320ca... added xevan
__host__
uint32_t cuda_check_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash)
{
	hipMemset(d_resNonces[thr_id], 0xff, sizeof(uint32_t));

	const uint32_t threadsperblock = 512;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

<<<<<<< HEAD
	if (bench_algo >= 0) // dont interrupt the global benchmark
		return UINT32_MAX;

	if (!init_done) {
		applog(LOG_ERR, "missing call to cuda_check_cpu_init");
		return UINT32_MAX;
	}

	cuda_checkhash_64 <<<grid, block>>> (threads, startNounce, d_inputHash, d_resNonces[thr_id]);
	hipDeviceSynchronize();

	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
	return h_resNonces[thr_id][0];
}

__host__
uint32_t cuda_check_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash)
{
	hipMemset(d_resNonces[thr_id], 0xff, sizeof(uint32_t));

	const uint32_t threadsperblock = 512;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	if (bench_algo >= 0) // dont interrupt the global benchmark
		return UINT32_MAX;

	if (!init_done) {
		applog(LOG_ERR, "missing call to cuda_check_cpu_init");
		return UINT32_MAX;
	}

	cuda_checkhash_32 <<<grid, block>>> (threads, startNounce, d_inputHash, d_resNonces[thr_id]);
	hipDeviceSynchronize();

	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
=======
	cuda_checkhash_64 <<<grid, block>>> (threads, startNounce, d_inputHash, d_resNonces[thr_id]);

	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

>>>>>>> 8c320ca... added xevan
	return h_resNonces[thr_id][0];
}

/* --------------------------------------------------------------------------------------------- */

__global__ __launch_bounds__(512, 4)
void cuda_checkhash_64_suppl(uint32_t startNounce, uint32_t *hash, uint32_t *resNonces)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	uint32_t *inpHash = &hash[thread << 4];

	if (hashbelowtarget(inpHash, pTarget)) {
<<<<<<< HEAD
		int resNum = ++resNonces[0];
		__threadfence();
=======
		int resNum = atomicAdd(resNonces,1)+1;
>>>>>>> 8c320ca... added xevan
		if (resNum < 8)
			resNonces[resNum] = (startNounce + thread);
	}
}

__host__
<<<<<<< HEAD
uint32_t cuda_check_hash_suppl(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash, uint8_t numNonce)
=======
uint32_t cuda_check_hash_suppl(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_inputHash, uint32_t foundnonce)
>>>>>>> 8c320ca... added xevan
{
	uint32_t rescnt, result = 0;

	const uint32_t threadsperblock = 512;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

<<<<<<< HEAD
	if (!init_done) {
		applog(LOG_ERR, "missing call to cuda_check_cpu_init");
		return 0;
	}

=======
>>>>>>> 8c320ca... added xevan
	// first element stores the count of found nonces
	hipMemset(d_resNonces[thr_id], 0, sizeof(uint32_t));

	cuda_checkhash_64_suppl <<<grid, block>>> (startNounce, d_inputHash, d_resNonces[thr_id]);
<<<<<<< HEAD
	hipDeviceSynchronize();

	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], 32, hipMemcpyDeviceToHost);
	rescnt = h_resNonces[thr_id][0];
	if (rescnt > numNonce) {
		if (numNonce <= rescnt) {
			result = h_resNonces[thr_id][numNonce+1];
		}
		if (opt_debug)
			applog(LOG_WARNING, "Found %d nonces: %x + %x", rescnt, h_resNonces[thr_id][1], result);
	}

=======
	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], 8*sizeof(uint32_t), hipMemcpyDeviceToHost);

	rescnt = h_resNonces[thr_id][0];
	if (rescnt > 1)
	{
		do
		{
			if (h_resNonces[thr_id][rescnt] != foundnonce)
			{
				result = h_resNonces[thr_id][rescnt];
				break;
			}
			rescnt--;
		} while (rescnt > 0);
	}
>>>>>>> 8c320ca... added xevan
	return result;
}

/* --------------------------------------------------------------------------------------------- */

__global__
void cuda_check_hash_branch_64(uint32_t threads, uint32_t startNounce, uint32_t *g_nonceVector, uint32_t *g_hash, uint32_t *resNounce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = g_nonceVector[thread];
		uint32_t hashPosition = (nounce - startNounce) << 4;
		uint32_t *inpHash = &g_hash[hashPosition];

<<<<<<< HEAD
		for (int i = 7; i >= 0; i--) {
			if (inpHash[i] > pTarget[i]) {
				return;
			}
			if (inpHash[i] < pTarget[i]) {
				break;
			}
		}
		if (resNounce[0] > nounce)
			resNounce[0] = nounce;
	}
}

__host__
uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order)
{
	const uint32_t threadsperblock = 256;

	uint32_t result = UINT32_MAX;

	if (bench_algo >= 0) // dont interrupt the global benchmark
		return result;

	if (!init_done) {
		applog(LOG_ERR, "missing call to cuda_check_cpu_init");
		return result;
	}

	hipMemset(d_resNonces[thr_id], 0xff, sizeof(uint32_t));

=======
		if (hashbelowtarget(inpHash, pTarget))
		{
			if (resNounce[0] > nounce)
				resNounce[0] = nounce;
		}
	}
}

__global__
void cuda_check_quarkcoin_64(uint32_t threads, uint32_t startNounce, uint32_t *g_nonceVector, uint32_t *g_hash, uint32_t *resNounce)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = g_nonceVector[thread];
		uint32_t hashPosition = (nounce - startNounce) << 4;
		uint32_t *inpHash = &g_hash[hashPosition];

		if (inpHash[7] <= pTarget[7])
		{
			uint32_t tmp = atomicCAS(resNounce, 0xffffffff, nounce);
			if (tmp != 0xffffffff)
				resNounce[1] = nounce;
		}
	}
}

__host__
uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash)
{
	uint32_t result = 0xffffffff;
	hipMemset(d_resNonces[thr_id], 0xff, sizeof(uint32_t));

	const uint32_t threadsperblock = 256;

>>>>>>> 8c320ca... added xevan
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cuda_check_hash_branch_64 <<<grid, block>>> (threads, startNounce, d_nonceVector, d_inputHash, d_resNonces[thr_id]);

<<<<<<< HEAD
	MyStreamSynchronize(NULL, order, thr_id);

	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
=======
	hipMemcpy(h_resNonces[thr_id], d_resNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

>>>>>>> 8c320ca... added xevan
	result = *h_resNonces[thr_id];

	return result;
}
<<<<<<< HEAD

/* Function to get the compiled Shader Model version */
int cuda_arch[MAX_GPUS] = { 0 };
__global__ void nvcc_get_arch(int *d_version)
{
	*d_version = 0;
#ifdef __CUDA_ARCH__
	*d_version = __CUDA_ARCH__;
#endif
}

__host__
int cuda_get_arch(int thr_id)
{
	int *d_version;
	int dev_id = device_map[thr_id];
	if (cuda_arch[dev_id] == 0) {
		// only do it once...
		hipMalloc(&d_version, sizeof(int));
		nvcc_get_arch <<< 1, 1 >>> (d_version);
		hipMemcpy(&cuda_arch[dev_id], d_version, sizeof(int), hipMemcpyDeviceToHost);
		hipFree(d_version);
	}
	return cuda_arch[dev_id];
}
=======
__host__
void cuda_check_quarkcoin(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, uint32_t *resNonces)
{
	hipMemset(d_resNonces[thr_id], 0xff, 2*sizeof(uint32_t));

	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	cuda_check_quarkcoin_64 << <grid, block >> > (threads, startNounce, d_nonceVector, d_inputHash, d_resNonces[thr_id]);

	hipMemcpy(resNonces, d_resNonces[thr_id], 2*sizeof(uint32_t), hipMemcpyDeviceToHost);
}
>>>>>>> 8c320ca... added xevan
