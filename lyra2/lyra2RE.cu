#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "lyra2/Lyra2.h"
}

<<<<<<< HEAD
#include <miner.h>
#include <cuda_helper.h>

static uint64_t* d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void keccak256_cpu_free(int thr_id);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix);
extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, bool gtx750ti);

extern void groestl256_cpu_init(int thr_id, uint32_t threads);
extern void groestl256_cpu_free(int thr_id);
extern void groestl256_setTarget(const void *ptarget);
extern uint32_t groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, int order);
extern uint32_t groestl256_getSecNonce(int thr_id, int num);


extern "C" void lyra2re_hash(void *state, const void *input)
{
	uint32_t hashA[8], hashB[8];

=======
#include "miner.h"
#include "cuda_helper.h"

static _ALIGN(64) uint64_t *d_hash[MAX_GPUS];
static  uint64_t *d_hash2[MAX_GPUS];


extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);
extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void lyra2_cpu_hash_32_multi(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);

extern void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t* matrix);
extern void lyra2_cpu_init_multi(int thr_id, uint32_t threads, uint64_t *hash, uint64_t* hash2);
extern void groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, uint32_t *resultnonces,uint32_t target);
extern void groestl256_cpu_init(int thr_id, uint32_t threads);

extern "C" void lyra2_hash(void *state, const void *input)
{
>>>>>>> 8c320ca... added xevan
	sph_blake256_context     ctx_blake;
	sph_keccak256_context    ctx_keccak;
	sph_skein256_context     ctx_skein;
	sph_groestl256_context   ctx_groestl;

<<<<<<< HEAD
	sph_blake256_set_rounds(14);
=======
	uint32_t hashA[8], hashB[8];
>>>>>>> 8c320ca... added xevan

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

<<<<<<< HEAD
	LYRA2(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8);

=======
	LYRA2_old(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8);
>>>>>>> 8c320ca... added xevan
	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashA, 32);
	sph_skein256_close(&ctx_skein, hashB);

	sph_groestl256_init(&ctx_groestl);
	sph_groestl256(&ctx_groestl, hashB, 32);
	sph_groestl256_close(&ctx_groestl, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };
<<<<<<< HEAD
static __thread uint32_t throughput = 0;

extern "C" int scanhash_lyra2(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

	static __thread bool gtx750ti;
	if (!init[thr_id])
	{
		int dev_id = device_map[thr_id];
		hipSetDevice(dev_id);
		CUDA_LOG_ERROR();

		int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 17 : 16;
		if (device_sm[device_map[thr_id]] == 500) intensity = 15;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);

		if (strstr(props.name, "750 Ti")) gtx750ti = true;
		else gtx750ti = false;

		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		blake256_cpu_init(thr_id, throughput);
		keccak256_cpu_init(thr_id, throughput);
		skein256_cpu_init(thr_id, throughput);
		groestl256_cpu_init(thr_id, throughput);

		if (device_sm[dev_id] >= 500)
		{
			size_t matrix_sz = device_sm[dev_id] > 500 ? sizeof(uint64_t) * 4 * 4 : sizeof(uint64_t) * 8 * 8 * 3 * 4;
			CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
			lyra2_cpu_init(thr_id, throughput, d_matrix[thr_id]);
		}

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		init[thr_id] = true;
	}

	uint32_t _ALIGN(128) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	groestl256_setTarget(ptarget);

	do {
		int order = 0;

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], gtx750ti);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = groestl256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			lyra2re_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = groestl256_getSecNonce(thr_id, 1);
				if (work->nonces[1] != UINT32_MAX) {
					be32enc(&endiandata[19], work->nonces[1]);
					lyra2re_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);
=======

extern "C" int scanhash_lyra2(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
//	uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 256 * 256 * 4 : 256 * 256 * 4 ;
 //   intensity = (device_sm[device_map[thr_id]] == 500) ? 256 * 256 * 2 : intensity;
//	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 18=256*256*4;

	uint32_t intensity = 256 * 256 * 2;

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);
	if (strstr(props.name, "970"))
	{
		intensity = 256 * 256 * 4;
	}
	else if (strstr(props.name, "980 Ti"))
	{
		intensity = 256 * 256 * 4;
	}
	else if (strstr(props.name, "980"))
	{
		intensity = 256 * 256 * 4;
	}
	else if (strstr(props.name, "750 Ti"))
	{
		intensity = 256 * 256 * 2;
	}
	else if (strstr(props.name, "750"))
	{
		intensity = 256 * 256 * 2;
	}
	else if (strstr(props.name, "960"))
	{
		intensity = 256 * 256 * 3;
	}
	else if (strstr(props.name, "950"))
	{
		intensity = 256 * 256 * 2;
	}

	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;

	
	if (!init[thr_id])
	{ 
		hipSetDevice(device_map[thr_id]);
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		groestl256_cpu_init(thr_id, throughput);
		
		CUDA_SAFE_CALL(hipMalloc(&d_hash2[thr_id], 16  * 8 * 8 * sizeof(uint64_t) * throughput));

        lyra2_cpu_init(thr_id, throughput,d_hash2[thr_id]);


		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint32_t) * throughput));

		init[thr_id] = true; 
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	blake256_cpu_setBlock_80(pdata);
	do {
		uint32_t foundNonce[2] = { 0, 0 };

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		lyra2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		groestl256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], foundNonce,ptarget[7]);
		if (foundNonce[0] != 0xffffffff)
		{
			CUDA_SAFE_CALL(hipGetLastError());
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce[0]);
			lyra2_hash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != 0xffffffff)
				{
					pdata[21] = foundNonce[1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, foundNonce[1], vhash64[7], Htarg);
				}
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d Found nounce % 08x", thr_id, foundNonce[0], vhash64[7], Htarg);
//				MyStreamSynchronize(NULL, NULL, device_map[thr_id]);
				return res;
			}
			else
			{
				if (vhash64[7] > Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_WARNING, "GPU #%d: result does not validate on CPU!", thr_id);
			}
		}

		pdata[19] += throughput;

	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
>>>>>>> 8c320ca... added xevan

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
<<<<<<< HEAD

// cleanup
extern "C" void free_lyra2(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_matrix[thr_id]);

	keccak256_cpu_free(thr_id);
	groestl256_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
=======
>>>>>>> 8c320ca... added xevan
