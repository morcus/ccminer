#include "hip/hip_runtime.h"
<<<<<<< HEAD
/**
 * Lyra2 (v2) CUDA Implementation
 *
 * Based on djm34/VTC sources and incredible 2x boost by Nanashi Meiyo-Meijin (May 2016)
 */
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cuda_lyra2v2_sm3.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 500
#endif

#define TPB 32

#if __CUDA_ARCH__ >= 500

#include "cuda_lyra2_vectors.h"

#define Nrow 4
#define Ncol 4
#define memshift 3

__device__ uint2x4 *DMatrix;

__device__ __forceinline__ uint2 LD4S(const int index)
{
	extern __shared__ uint2 shared_mem[];
	return shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
}

__device__ __forceinline__ void ST4S(const int index, const uint2 data)
{
	extern __shared__ uint2 shared_mem[];
	shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data;
}

__device__ __forceinline__ uint2 shuffle2(uint2 a, uint32_t b, uint32_t c)
{
	return make_uint2(__shfl(a.x, b, c), __shfl(a.y, b, c));
}

__device__ __forceinline__
void Gfunc_v5(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; d ^= a; d = SWAPUINT2(d);
	c += d; b ^= c; b = ROR2(b, 24);
	a += b; d ^= a; d = ROR2(d, 16);
	c += d; b ^= c; b = ROR2(b, 63);
}

__device__ __forceinline__
void round_lyra_v5(uint2x4 s[4])
{
	Gfunc_v5(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v5(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v5(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v5(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v5(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v5(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v5(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v5(s[0].w, s[1].x, s[2].y, s[3].z);
}

__device__ __forceinline__
void round_lyra_v5(uint2 s[4])
{
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	s[1] = shuffle2(s[1], threadIdx.x + 1, 4);
	s[2] = shuffle2(s[2], threadIdx.x + 2, 4);
	s[3] = shuffle2(s[3], threadIdx.x + 3, 4);
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	s[1] = shuffle2(s[1], threadIdx.x + 3, 4);
	s[2] = shuffle2(s[2], threadIdx.x + 2, 4);
	s[3] = shuffle2(s[3], threadIdx.x + 1, 4);
}

__device__ __forceinline__
void reduceDuplexRowSetup2(uint2 state[4])
{
	uint2 state1[Ncol][3], state0[Ncol][3], state2[3];
	int i, j;

	#pragma unroll
	for (int i = 0; i < Ncol; i++)
	{
		#pragma unroll
		for (j = 0; j < 3; j++)
			state0[Ncol - i - 1][j] = state[j];
		round_lyra_v5(state);
	}

	//#pragma unroll 4
	for (i = 0; i < Ncol; i++)
	{
		#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= state0[i][j];

		round_lyra_v5(state);

		#pragma unroll
		for (j = 0; j < 3; j++)
			state1[Ncol - i - 1][j] = state0[i][j];

		#pragma unroll
		for (j = 0; j < 3; j++)
			state1[Ncol - i - 1][j] ^= state[j];
	}

	for (i = 0; i < Ncol; i++)
	{
		const uint32_t s0 = memshift * Ncol * 0 + i * memshift;
		const uint32_t s2 = memshift * Ncol * 2 + memshift * (Ncol - 1) - i*memshift;

		#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= state1[i][j] + state0[i][j];

		round_lyra_v5(state);

		#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] = state1[i][j];

		#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] ^= state[j];

		#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s2 + j, state2[j]);

		uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
		uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
		uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

		if (threadIdx.x == 0) {
			state0[i][0] ^= Data2;
			state0[i][1] ^= Data0;
			state0[i][2] ^= Data1;
		} else {
			state0[i][0] ^= Data0;
			state0[i][1] ^= Data1;
			state0[i][2] ^= Data2;
		}

		#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s0 + j, state0[i][j]);

		#pragma unroll
		for (j = 0; j < 3; j++)
			state0[i][j] = state2[j];

	}

	for (i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = memshift * Ncol * 1 + i*memshift;
		const uint32_t s3 = memshift * Ncol * 3 + memshift * (Ncol - 1) - i*memshift;

		#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= state1[i][j] + state0[Ncol - i - 1][j];

		round_lyra_v5(state);

		#pragma unroll
		for (j = 0; j < 3; j++)
			state0[Ncol - i - 1][j] ^= state[j];

		#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s3 + j, state0[Ncol - i - 1][j]);

		uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
		uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
		uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

		if (threadIdx.x == 0) {
			state1[i][0] ^= Data2;
			state1[i][1] ^= Data0;
			state1[i][2] ^= Data1;
		} else  {
			state1[i][0] ^= Data0;
			state1[i][1] ^= Data1;
			state1[i][2] ^= Data2;
		}

		#pragma unroll
		for (j = 0; j < 3; j++)
			ST4S(s1 + j, state1[i][j]);
	}
}

__device__
void reduceDuplexRowt2(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4])
{
	uint2 state1[3], state2[3];
	const uint32_t ps1 = memshift * Ncol * rowIn;
	const uint32_t ps2 = memshift * Ncol * rowInOut;
	const uint32_t ps3 = memshift * Ncol * rowOut;

	for (int i = 0; i < Ncol; i++)
=======


#include <stdio.h>
#include <memory.h>
#include "cuda_vector.h"
#define TPB52 256
#define TPB50 64

 
#define Nrow 4
#define Ncol 4
#define u64type uint2
#define vectype uint28
#define memshift 3
__device__ vectype  *DMatrix;

 
__device__ __forceinline__ void Gfunc_v35(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
{

	a += b; d = eorswap32 (a, d);
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);

}

__device__ __forceinline__ void round_lyra_v35(vectype* s)
{

	Gfunc_v35(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v35(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v35(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v35(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v35(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v35(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v35(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v35(s[0].w, s[1].x, s[2].y, s[3].z);

}


 

__device__ __forceinline__ void reduceDuplex50(vectype state[4], uint32_t thread)
{
	const uint32_t ps1 = (Nrow * Ncol * memshift * thread);
	const uint32_t ps2 = (memshift * (Ncol - 1) + memshift * Ncol + Nrow * Ncol * memshift * thread);
	uint28 tmp[3];

//#pragma unroll 4
	for (int i = 0; i < Ncol; i++)
	{
#if __CUDA_ARCH__ == 500

		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 - i*memshift;

#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + s1)[j]);

		round_lyra_v35(state);

#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = __ldg4(&(DMatrix + s1)[j]) ^ state[j];
#else
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 - i*memshift;
		tmp[0] = __ldg4(&(DMatrix + s1)[0]);
		tmp[1] = __ldg4(&(DMatrix + s1)[1]);
		tmp[2] = __ldg4(&(DMatrix + s1)[2]);
		state[0] ^= tmp[0];
		state[1] ^= tmp[1];
		state[2] ^= tmp[2];

		round_lyra_v35(state);

#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = tmp[j] ^ state[j];
#endif

	}
}
__device__  void reduceDuplexRowSetupV2(const int rowIn, const int rowInOut, const int rowOut, vectype state[4], uint32_t thread)
{

	int i, j;
		vectype state2[3],state1[3];

		const uint32_t ps1 = (memshift * Ncol * rowIn + Nrow * Ncol * memshift * thread);
		const uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
		const uint32_t ps3 = (memshift * (Ncol-1) + memshift * Ncol * rowOut + Nrow * Ncol * memshift * thread);
	for (i = 0; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;
		const uint32_t s3 = ps3 - i*memshift;

		#if __CUDA_ARCH__ == 500
		#pragma unroll
		for (j = 0; j < 3; j++)
		{
			state[j] = state[j] ^ (__ldg4(&(DMatrix + s1)[j]) + __ldg4(&(DMatrix + s2)[j]));
		}
		
		round_lyra_v35(state);
#pragma unroll
		for (j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);

#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);

#pragma unroll
		for (j = 0; j < 3; j++) 
		{
			(DMatrix + s3)[j] =state[j]^state1[j];
		}
		#else

#pragma unroll
		for (j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);
#pragma unroll
		for (j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);
#pragma unroll
		for (j = 0; j < 3; j++)
		{
			state[j] ^= state1[j] + state2[j];
		}

		round_lyra_v35(state);

#pragma unroll
		for (j = 0; j < 3; j++)
		{			
			(DMatrix + s3)[j] = state1[j]^ state[j];;
		}

		#endif

		   ((uint2*)state2)[0] ^= ((uint2*)state)[11];
		   #pragma unroll
		   for (j = 0; j < 11; j++)
			((uint2*)state2)[j+1] ^= ((uint2*)state)[j];


		#pragma unroll
		for (j = 0; j < 3; j++)
		    (DMatrix + s2)[j] = state2[j];
	}
}



__device__ void reduceDuplexRowtV2(const int rowIn, const int rowInOut, const int rowOut, vectype* state, uint32_t thread)
{
	int i,j;
		vectype state2[3];
		const uint32_t ps1 = (memshift * Ncol * rowIn + Nrow * Ncol * memshift * thread);
		const uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
		const uint32_t ps3 = (memshift * Ncol * rowOut + Nrow * Ncol * memshift * thread);
	
	for (i = 0; i < Ncol; i++)
>>>>>>> 8c320ca... added xevan
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;
		const uint32_t s3 = ps3 + i*memshift;

<<<<<<< HEAD
		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] = LD4S(s1 + j);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = LD4S(s2 + j);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra_v5(state);

		uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
		uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
		uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

		if (threadIdx.x == 0) {
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		} else {
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		#pragma unroll
		for (int j = 0; j < 3; j++)
			ST4S(s2 + j, state2[j]);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			ST4S(s3 + j, LD4S(s3 + j) ^ state[j]);
	}
}

__device__
void reduceDuplexRowt2x4(const int rowInOut, uint2 state[4])
{
	const int rowIn = 2;
	const int rowOut = 3;

	int i, j;
	uint2 last[3];
	const uint32_t ps1 = memshift * Ncol * rowIn;
	const uint32_t ps2 = memshift * Ncol * rowInOut;

	#pragma unroll
	for (int j = 0; j < 3; j++)
		last[j] = LD4S(ps2 + j);

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= LD4S(ps1 + j) + last[j];

	round_lyra_v5(state);

	uint2 Data0 = shuffle2(state[0], threadIdx.x - 1, 4);
	uint2 Data1 = shuffle2(state[1], threadIdx.x - 1, 4);
	uint2 Data2 = shuffle2(state[2], threadIdx.x - 1, 4);

	if (threadIdx.x == 0) {
		last[0] ^= Data2;
		last[1] ^= Data0;
		last[2] ^= Data1;
	} else {
		last[0] ^= Data0;
		last[1] ^= Data1;
		last[2] ^= Data2;
	}

	if (rowInOut == rowOut)
	{
		#pragma unroll
		for (j = 0; j < 3; j++)
			last[j] ^= state[j];
	}

	for (i = 1; i < Ncol; i++)
	{
		const uint32_t s1 = ps1 + i*memshift;
		const uint32_t s2 = ps2 + i*memshift;

		#pragma unroll
		for (j = 0; j < 3; j++)
			state[j] ^= LD4S(s1 + j) + LD4S(s2 + j);

		round_lyra_v5(state);
	}

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= last[j];
}

__global__
__launch_bounds__(TPB, 1)
void lyra2v2_gpu_hash_32_1(uint32_t threads, uint2 *inputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	const uint2x4 blake2b_IV[2] = {
		0xf3bcc908UL, 0x6a09e667UL, 0x84caa73bUL, 0xbb67ae85UL,
		0xfe94f82bUL, 0x3c6ef372UL, 0x5f1d36f1UL, 0xa54ff53aUL,
		0xade682d1UL, 0x510e527fUL, 0x2b3e6c1fUL, 0x9b05688cUL,
		0xfb41bd6bUL, 0x1f83d9abUL, 0x137e2179UL, 0x5be0cd19UL
	};

	const uint2x4 Mask[2] = {
		0x00000020UL, 0x00000000UL, 0x00000020UL, 0x00000000UL,
		0x00000020UL, 0x00000000UL, 0x00000001UL, 0x00000000UL,
		0x00000004UL, 0x00000000UL, 0x00000004UL, 0x00000000UL,
		0x00000080UL, 0x00000000UL, 0x00000000UL, 0x01000000UL
	};

	uint2x4 state[4];

	if (thread < threads)
	{
		state[0].x = state[1].x = __ldg(&inputHash[thread + threads * 0]);
		state[0].y = state[1].y = __ldg(&inputHash[thread + threads * 1]);
		state[0].z = state[1].z = __ldg(&inputHash[thread + threads * 2]);
		state[0].w = state[1].w = __ldg(&inputHash[thread + threads * 3]);
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

		for (int i = 0; i<12; i++)
			round_lyra_v5(state);

		state[0] ^= Mask[0];
		state[1] ^= Mask[1];

		for (int i = 0; i<12; i++)
			round_lyra_v5(state);

		DMatrix[blockDim.x * gridDim.x * 0 + thread] = state[0];
		DMatrix[blockDim.x * gridDim.x * 1 + thread] = state[1];
		DMatrix[blockDim.x * gridDim.x * 2 + thread] = state[2];
		DMatrix[blockDim.x * gridDim.x * 3 + thread] = state[3];
	}
}

__global__
__launch_bounds__(TPB, 1)
void lyra2v2_gpu_hash_32_2(uint32_t threads)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;

	if (thread < threads)
	{
		uint2 state[4];
		state[0] = ((uint2*)DMatrix)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[1] = ((uint2*)DMatrix)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[2] = ((uint2*)DMatrix)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[3] = ((uint2*)DMatrix)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];

		reduceDuplexRowSetup2(state);

		uint32_t rowa;
		int prev = 3;

		for (int i = 0; i < 3; i++)
		{
			rowa = __shfl(state[0].x, 0, 4) & 3;
			reduceDuplexRowt2(prev, rowa, i, state);
			prev = i;
		}

		rowa = __shfl(state[0].x, 0, 4) & 3;
		reduceDuplexRowt2x4(rowa, state);

		((uint2*)DMatrix)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[0];
		((uint2*)DMatrix)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[1];
		((uint2*)DMatrix)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[2];
		((uint2*)DMatrix)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[3];
	}
}

__global__
__launch_bounds__(TPB, 1)
void lyra2v2_gpu_hash_32_3(uint32_t threads, uint2 *outputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint2x4 state[4];

	if (thread < threads)
	{
		state[0] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 0 + thread]);
		state[1] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 1 + thread]);
		state[2] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 2 + thread]);
		state[3] = __ldg4(&DMatrix[blockDim.x * gridDim.x * 3 + thread]);

		for (int i = 0; i < 12; i++)
			round_lyra_v5(state);

		outputHash[thread + threads * 0] = state[0].x;
		outputHash[thread + threads * 1] = state[0].y;
		outputHash[thread + threads * 2] = state[0].z;
		outputHash[thread + threads * 3] = state[0].w;
	}
}

#else
#include "cuda_helper.h"
#if __CUDA_ARCH__ < 200
__device__ void* DMatrix;
#endif
__global__ void lyra2v2_gpu_hash_32_1(uint32_t threads, uint2 *inputHash) {}
__global__ void lyra2v2_gpu_hash_32_2(uint32_t threads) {}
__global__ void lyra2v2_gpu_hash_32_3(uint32_t threads, uint2 *outputHash) {}
#endif


__host__
void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix)
{
	cuda_get_arch(thr_id);
	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
}

__host__
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, int order)
{
	int dev_id = device_map[thr_id % MAX_GPUS];

	if (device_sm[dev_id] >= 500) {

		const uint32_t tpb = TPB;

		dim3 grid2((threads + tpb - 1) / tpb);
		dim3 block2(tpb);
		dim3 grid4((threads * 4 + tpb - 1) / tpb);
		dim3 block4(4, tpb / 4);

		lyra2v2_gpu_hash_32_1 <<< grid2, block2 >>> (threads, (uint2*)g_hash);
		lyra2v2_gpu_hash_32_2 <<< grid4, block4, 48 * sizeof(uint2) * tpb >>> (threads);
		lyra2v2_gpu_hash_32_3 <<< grid2, block2 >>> (threads, (uint2*)g_hash);

	} else {

		uint32_t tpb = 16;
		if (cuda_arch[dev_id] >= 350) tpb = TPB35;
		else if (cuda_arch[dev_id] >= 300) tpb = TPB30;
		else if (cuda_arch[dev_id] >= 200) tpb = TPB20;

		dim3 grid((threads + tpb - 1) / tpb);
		dim3 block(tpb);
		lyra2v2_gpu_hash_32_v3 <<< grid, block >>> (threads, startNounce, (uint2*)g_hash);

	}
}
=======
		#pragma unroll 
		for (j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);

		#pragma unroll 
		for (j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + s1)[j]) + state2[j];

		round_lyra_v35(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		#pragma unroll 
		for (j = 0; j < 11; j++)
		((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];
#if __CUDA_ARCH__ == 500
		if (rowInOut != rowOut) 
		{
			#pragma unroll 
			for ( j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		} 
		if (rowInOut == rowOut)
		{
			#pragma unroll 
			for (j = 0; j < 3; j++)
			state2[j] ^= state[j];
		}
#else
		if (rowInOut != rowOut)
		{
			#pragma unroll 
			for (j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		} else
		{
			#pragma unroll 
			for (j = 0; j < 3; j++)
				state2[j] ^= state[j];
		}
#endif

		#pragma unroll 
		for (j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state2[j];
	}
}

__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(128, 1)
#endif
void lyra2v2_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);



	vectype state[4];

	if (thread < threads)
	{
		const uint28 blake2b_IV[2] =
		{
			0xf3bcc908, 0x6a09e667,
			0x84caa73b, 0xbb67ae85,
			0xfe94f82b, 0x3c6ef372,
			0x5f1d36f1, 0xa54ff53a,
			0xade682d1, 0x510e527f,
			0x2b3e6c1f, 0x9b05688c,
			0xfb41bd6b, 0x1f83d9ab,
			0x137e2179, 0x5be0cd19
		};

		state[2] = ((blake2b_IV)[0]);
		state[3] = ((blake2b_IV)[1]);

		((uint2*)state)[0] = __ldg(&outputHash[thread]);
		((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		((uint2*)state)[2] = __ldg(&outputHash[thread + 2 * threads]);
		((uint2*)state)[3] = __ldg(&outputHash[thread + 3 * threads]);

		 state[1] = state[0];

		 for (int i = 0; i<12; i++)
			 round_lyra_v35(state);
		 ((uint2*)state)[0].x ^= 0x20;
		 ((uint2*)state)[1].x ^= 0x20;
		 ((uint2*)state)[2].x ^= 0x20;
		 ((uint2*)state)[3].x ^= 0x01;
		 ((uint2*)state)[4].x ^= 0x04;
		 ((uint2*)state)[5].x ^= 0x04;
		 ((uint2*)state)[6].x ^= 0x80;
		 ((uint2*)state)[7].y ^= 0x01000000;

		 for (int i = 0; i<12; i++)
			 round_lyra_v35(state);

		const uint32_t ps1 = (memshift * (Ncol - 1) + Nrow * Ncol * memshift * thread);

#if __CUDA_ARCH__ > 500
#pragma unroll
#endif
		for (int i = 0; i < Ncol; i++)
		{
			const uint32_t s1 = ps1 - memshift * i;
			DMatrix[s1] = state[0];
			DMatrix[s1+1] = state[1];
			DMatrix[s1+2] = state[2];
			round_lyra_v35(state);
		}

		reduceDuplex50(state, thread);

		reduceDuplexRowSetupV2(1, 0, 2, state,  thread);
		reduceDuplexRowSetupV2(2, 1, 3, state,  thread);

		uint32_t rowa;
		int prev=3;

        for (int i = 0; i < 4; i++)
        {
	     rowa = ((uint2*)state)[0].x & 3;  
		 reduceDuplexRowtV2(prev, rowa, i, state, thread);
         prev=i;
        }


		const uint32_t shift = (memshift * Ncol * rowa + Nrow * Ncol * memshift * thread);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
        	round_lyra_v35(state);
		
		outputHash[thread] = ((uint2*)state)[0];
		outputHash[thread + threads] = ((uint2*)state)[1];
		outputHash[thread + 2 * threads] = ((uint2*)state)[2];
		outputHash[thread + 3 * threads] = ((uint2*)state)[3];
//		((vectype*)outputHash)[thread] = state[0];

	} //thread
}


__host__
void lyra2v2_cpu_init(int thr_id, uint32_t threads,uint64_t *hash)
{
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
}



__host__ 
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, uint32_t tpb)
{
	dim3 grid((threads + tpb - 1) / tpb);
	dim3 block(tpb);

	lyra2v2_gpu_hash_32 << <grid, block >> > (threads, startNounce, (uint2*)d_outputHash);
}

  
>>>>>>> 8c320ca... added xevan
