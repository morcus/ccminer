#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
#include "lyra2/Lyra2.h"
}

<<<<<<< HEAD
#include <miner.h>
#include <cuda_helper.h>

static uint64_t *d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);
extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void keccak256_cpu_free(int thr_id);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);
extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, int order);

extern void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t* d_matrix);

extern void bmw256_setTarget(const void *ptarget);
extern void bmw256_cpu_init(int thr_id, uint32_t threads);
extern void bmw256_cpu_free(int thr_id);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces);

void lyra2v2_hash(void *state, const void *input)
{
	uint32_t hashA[8], hashB[8];

=======
#include "miner.h"
#include "cuda_helper.h"


static _ALIGN(64) uint64_t *d_hash[MAX_GPUS];
static  uint64_t *d_hash2[MAX_GPUS];

extern void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_hash_80(int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void Keccak256_cpu_hash_32(int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);

extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void skeinCube256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);


extern void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, uint32_t tpb);
extern void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t* matrix);

extern void bmw256_cpu_init(int thr_id, uint32_t threads);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces, uint32_t target);

extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash);

extern "C" void lyra2v2_hash(void *state, const void *input)
{
>>>>>>> 8c320ca... added xevan
	sph_blake256_context      ctx_blake;
	sph_keccak256_context     ctx_keccak;
	sph_skein256_context      ctx_skein;
	sph_bmw256_context        ctx_bmw;
	sph_cubehash256_context   ctx_cube;

<<<<<<< HEAD
	sph_blake256_set_rounds(14);
=======
	uint32_t hashA[8], hashB[8];
>>>>>>> 8c320ca... added xevan

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashB, 32);
	sph_cubehash256_close(&ctx_cube, hashA);

<<<<<<< HEAD
=======

>>>>>>> 8c320ca... added xevan
	LYRA2(hashB, 32, hashA, 32, hashA, 32, 1, 4, 4);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashB, 32);
	sph_skein256_close(&ctx_skein, hashA);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashA, 32);
	sph_cubehash256_close(&ctx_cube, hashB);

<<<<<<< HEAD
=======

>>>>>>> 8c320ca... added xevan
	sph_bmw256_init(&ctx_bmw);
	sph_bmw256(&ctx_bmw, hashB, 32);
	sph_bmw256_close(&ctx_bmw, hashA);

	memcpy(state, hashA, 32);
}

static bool init[MAX_GPUS] = { 0 };

<<<<<<< HEAD
extern "C" int scanhash_lyra2v2(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] < 500) ? 18 : is_windows() ? 19 : 20;
	if (strstr(device_name[dev_id], "GTX 10")) intensity = 20;
	uint32_t throughput = cuda_default_throughput(dev_id, 1UL << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id])
	{
		size_t matrix_sz = 16 * sizeof(uint64_t) * 4 * 3;
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		blake256_cpu_init(thr_id, throughput);
		keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		bmw256_cpu_init(thr_id, throughput);

		// SM 3 implentation requires a bit more memory
		if (device_sm[dev_id] < 500 || cuda_arch[dev_id] < 500)
			matrix_sz = 16 * sizeof(uint64_t) * 4 * 4;
			
		CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
		lyra2v2_cpu_init(thr_id, throughput, d_matrix[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		api_set_throughput(thr_id, throughput);
=======
extern "C" int scanhash_lyra2v2(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t intensity = 256 * 256 * 8;
	uint32_t tpb = 8;
//	bool mergeblakekeccak = false;
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);
	if (strstr(props.name, "970"))
	{
		tpb = 10;
		intensity = 256 * 256 * 20;
	}
	else if (strstr(props.name, "980 Ti"))
	{
		tpb = 10;
		intensity = 256 * 256 * 18;
	}
	else if (strstr(props.name, "980"))
	{
		tpb = 10;
		intensity = 256 * 256 * 18;
	}
	else if (strstr(props.name, "750 Ti"))
	{
		intensity = 256 * 256 * 10;
		tpb = 16;
//		mergeblakekeccak = true;
	}
	else if (strstr(props.name, "750"))
	{
		intensity = 256 * 256 * 5;
		tpb = 16;
//		mergeblakekeccak = true;
	}
	else if (strstr(props.name, "960"))
	{
		tpb = 9;
		intensity = 256 * 256 * 18;
	}
	else if (strstr(props.name, "950"))
	{
		intensity = 256 * 256 * 18;
		tpb = 13;
	}

	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		//keccak256_cpu_init(thr_id,throughput);
		skein256_cpu_init(thr_id, throughput);
		bmw256_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash2[thr_id], 16 * 4 * 3 * sizeof(uint64_t) * throughput));
		lyra2v2_cpu_init(thr_id, throughput, d_hash2[thr_id]);
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint32_t) * throughput));
>>>>>>> 8c320ca... added xevan
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
<<<<<<< HEAD
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(pdata);
	bmw256_setTarget(ptarget);

	do {
		int order = 0;

		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2v2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		cubehash256_cpu_hash_32(thr_id, throughput,pdata[19], d_hash[thr_id], order++);

		memset(work->nonces, 0, sizeof(work->nonces));
		bmw256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], work->nonces);

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			lyra2v2_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					lyra2v2_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && !abort_flag);
=======
	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	blake256_cpu_setBlock_80(pdata);

	do {
		uint32_t foundNonce[2] = { 0, 0 };

//		if (mergeblakekeccak)
//		{
			blakeKeccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);

/*		}
		else
		{
			blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
			keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		}
*/
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		lyra2v2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], tpb);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		bmw256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], foundNonce, ptarget[7]);
		//		foundNonce[0] = 0xffffffff;
		if (foundNonce[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce[0]);
			lyra2v2_hash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != 0xffffffff)
				{
					pdata[21] = foundNonce[1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, foundNonce[1], vhash64[7], Htarg);
				}
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d Found nounce % 08x", thr_id, foundNonce[0], vhash64[7], Htarg);
				return res;
			}
			else
			{
				if (vhash64[7] > Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_WARNING, "GPU #%d: result does not validate on CPU!", thr_id);
			}
		}

		pdata[19] += throughput;

	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
>>>>>>> 8c320ca... added xevan

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
<<<<<<< HEAD

// cleanup
extern "C" void free_lyra2v2(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_matrix[thr_id]);

	bmw256_cpu_free(thr_id);
	keccak256_cpu_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
=======
>>>>>>> 8c320ca... added xevan
