#include "hip/hip_runtime.h"
<<<<<<< HEAD
/**
 * Lyra2 (v1) cuda implementation based on djm34 work
 * tpruvot@github 2015, Nanashi 08/2016 (from 1.8-r2)
 */

#include <stdio.h>
#include <memory.h>

#define TPB52 32

#include "cuda_lyra2_sm2.cuh"
#include "cuda_lyra2_sm5.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 520
#endif

#if !defined(__CUDA_ARCH__) ||  __CUDA_ARCH__ > 500

#include "cuda_lyra2_vectors.h"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
__device__ uint32_t __shfl(uint32_t a, uint32_t b, uint32_t c);
#endif

#define Nrow 8
#define Ncol 8
#define memshift 3

#define BUF_COUNT 0

__device__ uint2 *DMatrix;

__device__ __forceinline__ void LD4S(uint2 res[3], const int row, const int col, const int thread, const int threads)
{
#if BUF_COUNT != 8
	extern __shared__ uint2 shared_mem[];
	const int s0 = (Ncol * (row - BUF_COUNT) + col) * memshift;
#endif
#if BUF_COUNT != 0
	const int d0 = (memshift *(Ncol * row + col) * threads + thread)*blockDim.x + threadIdx.x;
#endif

#if BUF_COUNT == 8
	#pragma unroll
	for (int j = 0; j < 3; j++)
		res[j] = *(DMatrix + d0 + j * threads * blockDim.x);
#elif BUF_COUNT == 0
	#pragma unroll
	for (int j = 0; j < 3; j++)
		res[j] = shared_mem[((s0 + j) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
#else
	if (row < BUF_COUNT)
	{
		#pragma unroll
		for (int j = 0; j < 3; j++)
			res[j] = *(DMatrix + d0 + j * threads * blockDim.x);
	}
	else
	{
	#pragma unroll
		for (int j = 0; j < 3; j++)
			res[j] = shared_mem[((s0 + j) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
	}
#endif
}

__device__ __forceinline__ void ST4S(const int row, const int col, const uint2 data[3], const int thread, const int threads)
{
#if BUF_COUNT != 8
	extern __shared__ uint2 shared_mem[];
	const int s0 = (Ncol * (row - BUF_COUNT) + col) * memshift;
#endif
#if BUF_COUNT != 0
	const int d0 = (memshift *(Ncol * row + col) * threads + thread)*blockDim.x + threadIdx.x;
#endif

#if BUF_COUNT == 8
	#pragma unroll
	for (int j = 0; j < 3; j++)
		*(DMatrix + d0 + j * threads * blockDim.x) = data[j];

#elif BUF_COUNT == 0
	#pragma unroll
	for (int j = 0; j < 3; j++)
		shared_mem[((s0 + j) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data[j];

#else
	if (row < BUF_COUNT)
	{
	#pragma unroll
		for (int j = 0; j < 3; j++)
			*(DMatrix + d0 + j * threads * blockDim.x) = data[j];
	}
	else
	{
	#pragma unroll
		for (int j = 0; j < 3; j++)
			shared_mem[((s0 + j) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data[j];
	}
#endif
}

#if __CUDA_ARCH__ >= 300
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	return __shfl(a, b, c);
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	return make_uint2(__shfl(a.x, b, c), __shfl(a.y, b, c));
}

__device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	a1 = WarpShuffle(a1, b1, c);
	a2 = WarpShuffle(a2, b2, c);
	a3 = WarpShuffle(a3, b3, c);
}

#else
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;
	uint32_t *_ptr = (uint32_t*)shared_mem;

	__threadfence_block();
	uint32_t buf = _ptr[thread];

	_ptr[thread] = a;
	__threadfence_block();
	uint32_t result = _ptr[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	_ptr[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a;
	__threadfence_block();
	uint2 result = shared_mem[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a1;
	__threadfence_block();
	a1 = shared_mem[(thread&~(c - 1)) + (b1&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a2;
	__threadfence_block();
	a2 = shared_mem[(thread&~(c - 1)) + (b2&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a3;
	__threadfence_block();
	a3 = shared_mem[(thread&~(c - 1)) + (b3&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;
	__threadfence_block();
}

#endif

#if __CUDA_ARCH__ > 500 || !defined(__CUDA_ARCH)
static __device__ __forceinline__
void Gfunc(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; uint2 tmp = d; d.y = a.x ^ tmp.x; d.x = a.y ^ tmp.y;
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);
}
#endif

__device__ __forceinline__ void round_lyra(uint2 s[4])
{
	Gfunc(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3, 4);
	Gfunc(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1, 4);
}

static __device__ __forceinline__
void round_lyra(uint2x4* s)
{
	Gfunc(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc(s[0].w, s[1].w, s[2].w, s[3].w);
	Gfunc(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc(s[0].w, s[1].x, s[2].y, s[3].z);
}

static __device__ __forceinline__
void reduceDuplex(uint2 state[4], uint32_t thread, const uint32_t threads)
{
	uint2 state1[3];

#if __CUDA_ARCH__ > 500
#pragma unroll
#endif
	for (int i = 0; i < Nrow; i++)
	{
		ST4S(0, Ncol - i - 1, state, thread, threads);

		round_lyra(state);
	}

	#pragma unroll 4
	for (int i = 0; i < Nrow; i++)
	{
		LD4S(state1, 0, i, thread, threads);
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];

		round_lyra(state);

		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];
		ST4S(1, Ncol - i - 1, state1, thread, threads);
	}
}

static __device__ __forceinline__
void reduceDuplexRowSetup(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4], uint32_t thread, const uint32_t threads)
{
	uint2 state1[3], state2[3];

	#pragma unroll 1
	for (int i = 0; i < Nrow; i++)
	{
		LD4S(state1, rowIn, i, thread, threads);
		LD4S(state2, rowInOut, i, thread, threads);
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra(state);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];

		ST4S(rowOut, Ncol - i - 1, state1, thread, threads);

		//一個手前のスレッドからデータを貰う(同時に一個先のスレッドにデータを送る)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		} else {
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		ST4S(rowInOut, i, state2, thread, threads);
	}
}

static __device__ __forceinline__
void reduceDuplexRowt(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4], const uint32_t thread, const uint32_t threads)
{
	for (int i = 0; i < Nrow; i++)
	{
		uint2 state1[3], state2[3];

		LD4S(state1, rowIn, i, thread, threads);
		LD4S(state2, rowInOut, i, thread, threads);

#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra(state);

		//一個手前のスレッドからデータを貰う(同時に一個先のスレッドにデータを送る)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		ST4S(rowInOut, i, state2, thread, threads);

		LD4S(state1, rowOut, i, thread, threads);

#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];

		ST4S(rowOut, i, state1, thread, threads);
	}
}

static __device__ __forceinline__
void reduceDuplexRowt_8(const int rowInOut, uint2* state, const uint32_t thread, const uint32_t threads)
{
	uint2 state1[3], state2[3], last[3];

	LD4S(state1, 2, 0, thread, threads);
	LD4S(last, rowInOut, 0, thread, threads);

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= state1[j] + last[j];

	round_lyra(state);

	//一個手前のスレッドからデータを貰う(同時に一個先のスレッドにデータを送る)
	uint2 Data0 = state[0];
	uint2 Data1 = state[1];
	uint2 Data2 = state[2];
	WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

	if (threadIdx.x == 0)
	{
		last[0] ^= Data2;
		last[1] ^= Data0;
		last[2] ^= Data1;
	} else {
		last[0] ^= Data0;
		last[1] ^= Data1;
		last[2] ^= Data2;
	}

	if (rowInOut == 5)
	{
		#pragma unroll
		for (int j = 0; j < 3; j++)
			last[j] ^= state[j];
	}

	for (int i = 1; i < Nrow; i++)
	{
		LD4S(state1, 2, i, thread, threads);
		LD4S(state2, rowInOut, i, thread, threads);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra(state);
	}

	#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= last[j];
}

__constant__ uint2x4 blake2b_IV[2] = {
	0xf3bcc908lu, 0x6a09e667lu,
	0x84caa73blu, 0xbb67ae85lu,
	0xfe94f82blu, 0x3c6ef372lu,
	0x5f1d36f1lu, 0xa54ff53alu,
	0xade682d1lu, 0x510e527flu,
	0x2b3e6c1flu, 0x9b05688clu,
	0xfb41bd6blu, 0x1f83d9ablu,
	0x137e2179lu, 0x5be0cd19lu
};

__global__ __launch_bounds__(64, 1)
void lyra2_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		uint2x4 state[4];

		state[0].x = state[1].x = __ldg(&g_hash[thread + threads * 0]);
		state[0].y = state[1].y = __ldg(&g_hash[thread + threads * 1]);
		state[0].z = state[1].z = __ldg(&g_hash[thread + threads * 2]);
		state[0].w = state[1].w = __ldg(&g_hash[thread + threads * 3]);
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

		for (int i = 0; i<24; i++)
			round_lyra(state); //because 12 is not enough

		((uint2x4*)DMatrix)[threads * 0 + thread] = state[0];
		((uint2x4*)DMatrix)[threads * 1 + thread] = state[1];
		((uint2x4*)DMatrix)[threads * 2 + thread] = state[2];
		((uint2x4*)DMatrix)[threads * 3 + thread] = state[3];
	}
}

__global__
__launch_bounds__(TPB52, 1)
void lyra2_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;

	if (thread < threads)
	{
		uint2 state[4];
		state[0] = __ldg(&DMatrix[(0 * threads + thread) * blockDim.x + threadIdx.x]);
		state[1] = __ldg(&DMatrix[(1 * threads + thread) * blockDim.x + threadIdx.x]);
		state[2] = __ldg(&DMatrix[(2 * threads + thread) * blockDim.x + threadIdx.x]);
		state[3] = __ldg(&DMatrix[(3 * threads + thread) * blockDim.x + threadIdx.x]);

		reduceDuplex(state, thread, threads);
		reduceDuplexRowSetup(1, 0, 2, state, thread, threads);
		reduceDuplexRowSetup(2, 1, 3, state, thread, threads);
		reduceDuplexRowSetup(3, 0, 4, state, thread, threads);
		reduceDuplexRowSetup(4, 3, 5, state, thread, threads);
		reduceDuplexRowSetup(5, 2, 6, state, thread, threads);
		reduceDuplexRowSetup(6, 1, 7, state, thread, threads);

		uint32_t rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(7, rowa, 0, state, thread, threads);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(0, rowa, 3, state, thread, threads);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(3, rowa, 6, state, thread, threads);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(6, rowa, 1, state, thread, threads);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(1, rowa, 4, state, thread, threads);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(4, rowa, 7, state, thread, threads);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(7, rowa, 2, state, thread, threads);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt_8(rowa, state, thread, threads);

		DMatrix[(0 * threads + thread) * blockDim.x + threadIdx.x] = state[0];
		DMatrix[(1 * threads + thread) * blockDim.x + threadIdx.x] = state[1];
		DMatrix[(2 * threads + thread) * blockDim.x + threadIdx.x] = state[2];
		DMatrix[(3 * threads + thread) * blockDim.x + threadIdx.x] = state[3];
	}
}

__global__ __launch_bounds__(64, 1)
void lyra2_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint28 state[4];

	if (thread < threads)
	{
		state[0] = __ldg4(&((uint2x4*)DMatrix)[threads * 0 + thread]);
		state[1] = __ldg4(&((uint2x4*)DMatrix)[threads * 1 + thread]);
		state[2] = __ldg4(&((uint2x4*)DMatrix)[threads * 2 + thread]);
		state[3] = __ldg4(&((uint2x4*)DMatrix)[threads * 3 + thread]);

		for (int i = 0; i < 12; i++)
			round_lyra(state);

		g_hash[thread + threads * 0] = state[0].x;
		g_hash[thread + threads * 1] = state[0].y;
		g_hash[thread + threads * 2] = state[0].z;
		g_hash[thread + threads * 3] = state[0].w;

	} //thread
}
#else
#if __CUDA_ARCH__ < 500

/* for unsupported SM arch */
__device__ void* DMatrix;
#endif
__global__ void lyra2_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *g_hash) {}
__global__ void lyra2_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash) {}
__global__ void lyra2_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *g_hash) {}
#endif

__host__
void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix)
{
	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
}

__host__
void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, bool gtx750ti)
{
	int dev_id = device_map[thr_id % MAX_GPUS];

	uint32_t tpb = TPB52;

	if (cuda_arch[dev_id] >= 520) tpb = TPB52;
	else if (cuda_arch[dev_id] >= 500) tpb = TPB50;
	else if (cuda_arch[dev_id] >= 200) tpb = TPB20;

	dim3 grid1((threads * 4 + tpb - 1) / tpb);
	dim3 block1(4, tpb >> 2);

	dim3 grid2((threads + 64 - 1) / 64);
	dim3 block2(64);

	dim3 grid3((threads + tpb - 1) / tpb);
	dim3 block3(tpb);

	if (cuda_arch[dev_id] >= 520)
	{
		lyra2_gpu_hash_32_1 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);

		lyra2_gpu_hash_32_2 <<< grid1, block1, 24 * (8 - 0) * sizeof(uint2) * tpb >>> (threads, startNounce, d_hash);

		lyra2_gpu_hash_32_3 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);
	}
	else if (cuda_arch[dev_id] >= 500)
	{
		size_t shared_mem = 0;

		if (gtx750ti)
			// 8Warpに調整のため、8192バイト確保する
			shared_mem = 8192;
		else
			// 10Warpに調整のため、6144バイト確保する
			shared_mem = 6144;

		lyra2_gpu_hash_32_1_sm5 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);

		lyra2_gpu_hash_32_2_sm5 <<< grid1, block1, shared_mem >>> (threads, startNounce, (uint2*)d_hash);

		lyra2_gpu_hash_32_3_sm5 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);
	}
	else
		lyra2_gpu_hash_32_sm2 <<< grid3, block3 >>> (threads, startNounce, d_hash);
}
=======


#include <stdio.h>
#include <memory.h>
#include "cuda_vector.h"
#define TPB 8
//



#if __CUDA_ARCH__ == 500
#define u64type uint2
#define vectype uint28
#define memshift 3
#else 
#define u64type uint2
#define vectype uint28
#define memshift 4   
#endif
__device__ vectype  *DMatrix;

 
static __device__ __forceinline__ void Gfunc_v35(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
{

	a += b; d = eorswap32(a, d);
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);

}

static __device__ __forceinline__ void round_lyra_v35(vectype* s)
{

	Gfunc_v35(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v35(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v35(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v35(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v35(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v35(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v35(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v35(s[0].w, s[1].x, s[2].y, s[3].z);

}



static __device__ __forceinline__ void reduceDuplex(vectype state[4], uint32_t thread)
{


	    vectype state1[3]; 
		uint32_t ps1 = (256 * thread);
		uint32_t ps2 = (memshift * 7 + memshift * 8 + 256 * thread);

#pragma unroll 4
	for (int i = 0; i < 8; i++)
	{
        uint32_t s1 = ps1 + i*memshift;
        uint32_t s2 = ps2 - i*memshift;  
		
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix+s1)[j]); 
 
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];
		round_lyra_v35(state); 
		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];

		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state1[j];

	}

}

static __device__ __forceinline__ void reduceDuplexV3(vectype state[4], uint32_t thread)
{


	vectype state1[3];
	uint32_t ps1 = (256 * thread);
//                     colomn             row
	uint32_t ps2 = (memshift * 7 * 8 + memshift * 1 + 64 * memshift * thread);

#pragma unroll 4
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + 8 * i *memshift;
		uint32_t s2 = ps2 - 8 * i *memshift;

		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);

		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];
		round_lyra_v35(state);

		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];


		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state1[j];

	}

}

static __device__ __forceinline__ void reduceDuplexRowSetupV2(const int rowIn, const int rowInOut, const int rowOut, vectype state[4], uint32_t thread)
{


		vectype state2[3],state1[3];

		uint32_t ps1 = (              memshift * 8 * rowIn    + 256 * thread);
		uint32_t ps2 = (              memshift * 8 * rowInOut + 256 * thread);
		uint32_t ps3 = (memshift*7  + memshift * 8 * rowOut   + 256 * thread);


#pragma unroll 1
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 + i*memshift;
		uint32_t s3 = ps3 - i*memshift;

		for (int j = 0; j < 3; j++) 
			state1[j]= __ldg4(&(DMatrix + s1)[j]);
		for (int j = 0; j < 3; j++)
			state2[j]= __ldg4(&(DMatrix + s2)[j]);
		for (int j = 0; j < 3; j++) {
			vectype tmp = state1[j] + state2[j];
			state[j] ^= tmp;
		}
		

		round_lyra_v35(state);

		for (int j = 0; j < 3; j++) {
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}
 
		   ((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++) 
			((uint2*)state2)[j+1] ^= ((uint2*)state)[j];



		for (int j = 0; j < 3; j++)
		    (DMatrix + s2)[j] = state2[j];
		
	}


}

static __device__ __forceinline__ void reduceDuplexRowSetupV3(const int rowIn, const int rowInOut, const int rowOut, vectype state[4], uint32_t thread)
{


	vectype state2[3], state1[3];
	
	uint32_t ps1 = (                  memshift *  rowIn    + 64 * memshift * thread);
	uint32_t ps2 = (memshift * rowInOut +                    64 * memshift* thread);
	uint32_t ps3 = (8 * memshift * 7 + memshift *  rowOut +  64 * memshift * thread);
	/*
	uint32_t ps1 = (256 * thread);
	uint32_t ps2 = (256 * thread);
	uint32_t ps3 = (256 * thread);
    */
#pragma nounroll 
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + 8*i*memshift;
		uint32_t s2 = ps2 + 8*i*memshift;
		uint32_t s3 = ps3 - 8*i*memshift;

		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1 )[j]);
		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2 )[j]);
		for (int j = 0; j < 3; j++) {
			vectype tmp = state1[j] + state2[j];
			state[j] ^= tmp;
		}


		round_lyra_v35(state);

		for (int j = 0; j < 3; j++) {
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];



		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state2[j];

	}


}


static __device__ __forceinline__ void reduceDuplexRowtV2(const int rowIn, const int rowInOut, const int rowOut, vectype* state, uint32_t thread)
{

		vectype state1[3],state2[3];
		uint32_t ps1 = (memshift * 8 * rowIn + 256 * thread);
		uint32_t ps2 = (memshift * 8 * rowInOut + 256 * thread);
		uint32_t ps3 = (memshift * 8 * rowOut + 256 * thread);

#pragma unroll 1
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 + i*memshift;
		uint32_t s3 = ps3 + i*memshift;


		for (int j = 0; j < 3; j++)  
			state1[j] = __ldg4(&(DMatrix + s1)[j]);


		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);


		for (int j = 0; j < 3; j++)
			          state1[j] += state2[j];

		for (int j = 0; j < 3; j++)
			          state[j] ^= state1[j];


		round_lyra_v35(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++)
		((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];

if (rowInOut != rowOut) {

	for (int j = 0; j < 3; j++)
		(DMatrix + s2)[j] = state2[j];

	for (int j = 0; j < 3; j++)
		(DMatrix + s3)[j] ^= state[j];

} else {

	for (int j = 0; j < 3; j++)
		state2[j] ^= state[j];

	for (int j = 0; j < 3; j++)
		(DMatrix + s2)[j]=state2[j];
}






	}
}

static __device__ __forceinline__ void reduceDuplexRowtV3(const int rowIn, const int rowInOut, const int rowOut, vectype* state, uint32_t thread)
{

	vectype state1[3], state2[3];
	uint32_t ps1 = (memshift * rowIn + 64 * memshift * thread);
	uint32_t ps2 = (memshift * rowInOut + 64 * memshift * thread);
	uint32_t ps3 = (memshift * rowOut + 64 *memshift * thread);

#pragma nounroll 
	for (int i = 0; i < 8; i++)
	{
		uint32_t s1 = ps1 + 8 * i*memshift;
		uint32_t s2 = ps2 + 8 * i*memshift;
		uint32_t s3 = ps3 + 8 * i*memshift;


		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);


		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);


		for (int j = 0; j < 3; j++)
			state1[j] += state2[j];

		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];


		round_lyra_v35(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];

		if (rowInOut != rowOut) {

			for (int j = 0; j < 3; j++)
				(DMatrix + s2)[j] = state2[j];

			for (int j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		}
		else {

			for (int j = 0; j < 3; j++)
				state2[j] ^= state[j];

			for (int j = 0; j < 3; j++)
				(DMatrix + s2)[j] = state2[j];
		}






	}
}



#if __CUDA_ARCH__ < 500
__global__	__launch_bounds__(48, 1)
#elif __CUDA_ARCH__ == 500
__global__	__launch_bounds__(16, 1)
#else
__global__	__launch_bounds__(TPB, 1)
#endif
void lyra2_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	   vectype state[4];
#if __CUDA_ARCH__ > 350
	const uint28 blake2b_IV[2] = {
		{{ 0xf3bcc908, 0x6a09e667 },
		{ 0x84caa73b, 0xbb67ae85 },
		{ 0xfe94f82b, 0x3c6ef372 },
		{ 0x5f1d36f1, 0xa54ff53a }},
		{{ 0xade682d1, 0x510e527f },
		{ 0x2b3e6c1f, 0x9b05688c },
		{ 0xfb41bd6b, 0x1f83d9ab },
		{ 0x137e2179, 0x5be0cd19 }}};
#else 
		const ulonglong4 blake2b_IV[2] = {
			{ 0x6a09e667f3bcc908,  
			  0xbb67ae8584caa73b,  
			  0x3c6ef372fe94f82b,  
			  0xa54ff53a5f1d36f1   },
			{ 0x510e527fade682d1,  
			  0x9b05688c2b3e6c1f,  
			  0x1f83d9abfb41bd6b,  
			  0x5be0cd19137e2179  } };
#endif
	
#if __CUDA_ARCH__ == 350
	if (thread < threads)
#endif
	{
 
		 ((uint2*)state)[0] = __ldg(&outputHash[thread]);
		 ((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		 ((uint2*)state)[2] = __ldg(&outputHash[thread + 2 * threads]);
		 ((uint2*)state)[3] = __ldg(&outputHash[thread + 3 * threads]);
//		 state[0] = __ldg4(&((vectype*)outputHash)[thread]);
		 state[1] = state[0];
		 state[2] = ((vectype*)blake2b_IV)[0];
		 state[3] = ((vectype*)blake2b_IV)[1];

 
		for (int i = 0; i<24; i++) { round_lyra_v35(state); } //because 12 is not enough

             uint32_t ps1 = (memshift * 7  + 256 * thread);

		for (int i = 0; i < 8; i++)
		{
			uint32_t s1 = ps1 - memshift * i;
			for (int j = 0; j < 3; j++)
			    (DMatrix + s1)[j] = (state)[j];

			round_lyra_v35(state);
		}


		reduceDuplex(state, thread);

		reduceDuplexRowSetupV2(1, 0, 2, state,  thread);
		reduceDuplexRowSetupV2(2, 1, 3, state,  thread);
		reduceDuplexRowSetupV2(3, 0, 4, state,  thread);
		reduceDuplexRowSetupV2(4, 3, 5, state,  thread);
		reduceDuplexRowSetupV2(5, 2, 6, state,  thread);
		reduceDuplexRowSetupV2(6, 1, 7, state,  thread);
		uint32_t rowa = ((uint2*)state)[0].x & 7;

		reduceDuplexRowtV2(7, rowa, 0, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(0, rowa, 3, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(3, rowa, 6, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(6, rowa, 1, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(1, rowa, 4, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(4, rowa, 7, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(7, rowa, 2, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV2(2, rowa, 5, state, thread);

		uint32_t shift = (memshift * 8 * rowa + 256 * thread);

		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
        			round_lyra_v35(state);
		

		outputHash[thread]=            ((uint2*)state)[0];
		outputHash[thread + threads] = ((uint2*)state)[1];
		outputHash[thread + 2 * threads] = ((uint2*)state)[2]; 
		outputHash[thread + 3 * threads] = ((uint2*)state)[3];
//		((vectype*)outputHash)[thread] = state[0];

	} //thread
}

#if __CUDA_ARCH__ < 500
__global__	__launch_bounds__(48, 1)
#elif __CUDA_ARCH__ == 500
__global__	__launch_bounds__(16, 1)
#else
__global__	__launch_bounds__(TPB, 1)
#endif
void lyra2_gpu_hash_32_v3(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	vectype state[4];

#if __CUDA_ARCH__ > 350
	const uint28 blake2b_IV[2] = {
		{ { 0xf3bcc908, 0x6a09e667 },
		{ 0x84caa73b, 0xbb67ae85 },
		{ 0xfe94f82b, 0x3c6ef372 },
		{ 0x5f1d36f1, 0xa54ff53a } },
		{ { 0xade682d1, 0x510e527f },
		{ 0x2b3e6c1f, 0x9b05688c },
		{ 0xfb41bd6b, 0x1f83d9ab },
		{ 0x137e2179, 0x5be0cd19 } } };
#else 
	const ulonglong4 blake2b_IV[2] = {
		{ 0x6a09e667f3bcc908,
		0xbb67ae8584caa73b,
		0x3c6ef372fe94f82b,
		0xa54ff53a5f1d36f1 },
		{ 0x510e527fade682d1,
		0x9b05688c2b3e6c1f,
		0x1f83d9abfb41bd6b,
		0x5be0cd19137e2179 } };
#endif


#if __CUDA_ARCH__ == 350
	if (thread < threads)
#endif
	{

		((uint2*)state)[0] = __ldg(&outputHash[thread]);
		((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		((uint2*)state)[2] = __ldg(&outputHash[thread + 2 * threads]);
		((uint2*)state)[3] = __ldg(&outputHash[thread + 3 * threads]);
		
		state[1] = state[0];

		state[2] = ((vectype*)blake2b_IV)[0];
		state[3] = ((vectype*)blake2b_IV)[1];

		for (int i = 0; i<24; i++) 
                round_lyra_v35(state);  //because 12 is not enough

		uint32_t ps1 = (8 * memshift * 7 + 64 * memshift * thread);


		for (int i = 0; i < 8; i++)
		{
			uint32_t s1 = ps1 - 8 * memshift * i;
			for (int j = 0; j < 3; j++)
				(DMatrix + s1)[j] = (state)[j];

			round_lyra_v35(state);
		}


		reduceDuplexV3(state, thread);

		reduceDuplexRowSetupV3(1, 0, 2, state, thread);
		reduceDuplexRowSetupV3(2, 1, 3, state, thread);
		reduceDuplexRowSetupV3(3, 0, 4, state, thread);
		reduceDuplexRowSetupV3(4, 3, 5, state, thread);
		reduceDuplexRowSetupV3(5, 2, 6, state, thread);
		reduceDuplexRowSetupV3(6, 1, 7, state, thread);
		uint32_t rowa = ((uint2*)state)[0].x & 7;

		reduceDuplexRowtV3(7, rowa, 0, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(0, rowa, 3, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(3, rowa, 6, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(6, rowa, 1, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(1, rowa, 4, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(4, rowa, 7, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(7, rowa, 2, state, thread);
		rowa = ((uint2*)state)[0].x & 7;
		reduceDuplexRowtV3(2, rowa, 5, state, thread);

		uint32_t shift = (memshift * rowa + 64 * memshift * thread);

		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
			round_lyra_v35(state);


		outputHash[thread] = ((uint2*)state)[0];
		outputHash[thread + threads] = ((uint2*)state)[1];
		outputHash[thread + 2 * threads] = ((uint2*)state)[2];
		outputHash[thread + 3 * threads] = ((uint2*)state)[3];
		
	} //thread
}




__host__
void lyra2_cpu_init(int thr_id, uint32_t threads,uint64_t *hash)
{
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
}



__host__ 
void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash)
{
uint32_t tpb;
	if (device_sm[device_map[thr_id]]==500)
      tpb = 16; 
    else 
      tpb = TPB;
	dim3 grid((threads + tpb - 1) / tpb);
	dim3 block(tpb);

	if (device_sm[device_map[thr_id]] == 500)
		lyra2_gpu_hash_32 << <grid, block >> > (threads, startNounce, (uint2*)d_outputHash);
    else 
    	lyra2_gpu_hash_32_v3 <<<grid, block>>> (threads, startNounce,(uint2*) d_outputHash);


}

  
>>>>>>> 8c320ca... added xevan
