#include "hip/hip_runtime.h"
/**
<<<<<<< HEAD
 * S3 Hash (Also called Triple S - Used by 1Coin)
=======
 * S3 Hash (Also called 3S - Used by 1Coin)
>>>>>>> 8c320ca... added xevan
 */

extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
}

#include "miner.h"
#include "cuda_helper.h"
<<<<<<< HEAD
#include "cuda_x11.h"

extern void x11_shavite512_setBlock_80(void *pdata);
extern void x11_shavite512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);
=======
>>>>>>> 8c320ca... added xevan

#include <stdint.h>

static uint32_t *d_hash[MAX_GPUS];

<<<<<<< HEAD
=======
extern void x11_shavite512_cpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void x11_shavite512_setBlock_80(void *pdata);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const uint32_t simdthreads);

extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
//extern void quark_skein512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, uint32_t *h_found, uint32_t target);
extern void quark_skein512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

>>>>>>> 8c320ca... added xevan
/* CPU HASH */
extern "C" void s3hash(void *output, const void *input)
{
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_skein512_context ctx_skein;

	unsigned char hash[64];

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, input, 80);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	memcpy(output, hash, 32);
}

<<<<<<< HEAD
#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 32); \
		hipMemcpy(debugbuf, d_hash[thr_id], 32, hipMemcpyDeviceToHost); \
		printf("S3 %s %08x %08x %08x %08x...%08x\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3]), swab32(debugbuf[7])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

static bool init[MAX_GPUS] = { 0 };

/* Main S3 entry point */
extern "C" int scanhash_s3(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = 20; // 256*256*8*2;
=======
static bool init[MAX_GPUS] = { 0 };

/* Main S3 entry point */
extern "C" int scanhash_s3(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t intensity = 20; // 256*256*8*2;
>>>>>>> 8c320ca... added xevan
#ifdef WIN32
	// reduce by one the intensity on windows
	intensity--;
#endif
<<<<<<< HEAD
	uint32_t throughput =  cuda_default_throughput(thr_id, 1 << intensity);
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0xF;
=======
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, 1 << intensity);
	throughput = min(throughput, (max_nonce - first_nonce));
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 64 : 32;
	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000000fu;
>>>>>>> 8c320ca... added xevan

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
<<<<<<< HEAD
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));

		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
=======
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}

		x11_simd512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput), 0);
>>>>>>> 8c320ca... added xevan

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
<<<<<<< HEAD
		be32enc(&endiandata[k], pdata[k]);
=======
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
>>>>>>> 8c320ca... added xevan

	x11_shavite512_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
<<<<<<< HEAD
		int order = 0;

		x11_shavite512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		TRACE("shavite:");
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("simd   :");
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("skein  :");

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			s3hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					s3hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);
=======

		x11_shavite512_cpu_hash_80(throughput, pdata[19], d_hash[thr_id]);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], simdthreads);
		quark_skein512_cpu_hash_64(throughput, pdata[19], NULL, d_hash[thr_id]);
		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);

		if (foundNonce != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			s3hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				/*
				if (h_found[thr_id][1] != 0xffffffff)
				{
					pdata[21] = h_found[thr_id][1];
					res++;
					if (opt_benchmark)
						applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, h_found[thr_id][1]);
				}
				*/
				pdata[19] = foundNonce;
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", thr_id, foundNonce);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce);
				}
			}
		}
		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
>>>>>>> 8c320ca... added xevan

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
<<<<<<< HEAD

// cleanup
extern "C" void free_s3(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
=======
>>>>>>> 8c320ca... added xevan
