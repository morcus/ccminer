#include "hip/hip_runtime.h"
<<<<<<< HEAD
#include <cuda_helper.h>
#include <cuda_vectors.h>

#define CUBEHASH_ROUNDS 16 /* this is r for CubeHashr/b */
#define CUBEHASH_BLOCKBYTES 32 /* this is b for CubeHashr/b */

#if __CUDA_ARCH__ < 350
#define LROT(x,bits) ((x << bits) | (x >> (32 - bits)))
#else
#define LROT(x, bits) __funnelshift_l(x, x, bits)
#endif

#define ROTATEUPWARDS7(a)  LROT(a,7)
#define ROTATEUPWARDS11(a) LROT(a,11)

#define SWAP(a,b) { uint32_t u = a; a = b; b = u; }

__device__ __constant__
static const uint32_t c_IV_512[32] = {
	0x2AEA2A61, 0x50F494D4, 0x2D538B8B, 0x4167D83E,
	0x3FEE2313, 0xC701CF8C, 0xCC39968E, 0x50AC5695,
	0x4D42C787, 0xA647A8B3, 0x97CF0BEF, 0x825B4537,
	0xEEF864D2, 0xF22090C4, 0xD0E5CD33, 0xA23911AE,
	0xFCD398D9, 0x148FE485, 0x1B017BEF, 0xB6444532,
	0x6A536159, 0x2FF5781C, 0x91FA7934, 0x0DBADEA9,
	0xD65C8A2B, 0xA5A70E75, 0xB1C62456, 0xBC796576,
	0x1921C8F7, 0xE7989AF1, 0x7795D246, 0xD43E3B44
};

__device__ __forceinline__
static void rrounds(uint32_t x[2][2][2][2][2])
{
    int r;
    int j;
    int k;
    int l;
    int m;

//#pragma unroll 16
    for (r = 0;r < CUBEHASH_ROUNDS;++r) {

        /* "add x_0jklm into x_1jklmn modulo 2^32" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[1][j][k][l][m] += x[0][j][k][l][m];

        /* "rotate x_0jklm upwards by 7 bits" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] = ROTATEUPWARDS7(x[0][j][k][l][m]);

        /* "swap x_00klm with x_01klm" */
#pragma unroll 2
        for (k = 0;k < 2;++k)
#pragma unroll 2
            for (l = 0;l < 2;++l)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    SWAP(x[0][0][k][l][m],x[0][1][k][l][m])

        /* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] ^= x[1][j][k][l][m];

        /* "swap x_1jk0m with x_1jk1m" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    SWAP(x[1][j][k][0][m],x[1][j][k][1][m])

        /* "add x_0jklm into x_1jklm modulo 2^32" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[1][j][k][l][m] += x[0][j][k][l][m];

        /* "rotate x_0jklm upwards by 11 bits" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] = ROTATEUPWARDS11(x[0][j][k][l][m]);

        /* "swap x_0j0lm with x_0j1lm" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (l = 0;l < 2;++l)
#pragma unroll 2
                for (m = 0;m < 2;++m)
                    SWAP(x[0][j][0][l][m],x[0][j][1][l][m])

        /* "xor x_1jklm into x_0jklm" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
#pragma unroll 2
                    for (m = 0;m < 2;++m)
                        x[0][j][k][l][m] ^= x[1][j][k][l][m];

        /* "swap x_1jkl0 with x_1jkl1" */
#pragma unroll 2
        for (j = 0;j < 2;++j)
#pragma unroll 2
            for (k = 0;k < 2;++k)
#pragma unroll 2
                for (l = 0;l < 2;++l)
                    SWAP(x[1][j][k][l][0],x[1][j][k][l][1])

    }
}

__device__ __forceinline__
static void block_tox(uint32_t* const block, uint32_t x[2][2][2][2][2])
{
	// read 32 bytes input from global mem with uint2 chunks
	AS_UINT2(x[0][0][0][0]) ^= AS_UINT2(&block[0]);
	AS_UINT2(x[0][0][0][1]) ^= AS_UINT2(&block[2]);
	AS_UINT2(x[0][0][1][0]) ^= AS_UINT2(&block[4]);
	AS_UINT2(x[0][0][1][1]) ^= AS_UINT2(&block[6]);
}

__device__ __forceinline__
static void hash_fromx(uint32_t hash[16], uint32_t const x[2][2][2][2][2])
{
	// used to write final hash to global mem
	AS_UINT2(&hash[ 0]) = AS_UINT2(x[0][0][0][0]);
	AS_UINT2(&hash[ 2]) = AS_UINT2(x[0][0][0][1]);
	AS_UINT2(&hash[ 4]) = AS_UINT2(x[0][0][1][0]);
	AS_UINT2(&hash[ 6]) = AS_UINT2(x[0][0][1][1]);
	AS_UINT2(&hash[ 8]) = AS_UINT2(x[0][1][0][0]);
	AS_UINT2(&hash[10]) = AS_UINT2(x[0][1][0][1]);
	AS_UINT2(&hash[12]) = AS_UINT2(x[0][1][1][0]);
	AS_UINT2(&hash[14]) = AS_UINT2(x[0][1][1][1]);
}

#define Init(x) \
	AS_UINT2(x[0][0][0][0]) = AS_UINT2(&c_IV_512[ 0]); \
	AS_UINT2(x[0][0][0][1]) = AS_UINT2(&c_IV_512[ 2]); \
	AS_UINT2(x[0][0][1][0]) = AS_UINT2(&c_IV_512[ 4]); \
	AS_UINT2(x[0][0][1][1]) = AS_UINT2(&c_IV_512[ 6]); \
	AS_UINT2(x[0][1][0][0]) = AS_UINT2(&c_IV_512[ 8]); \
	AS_UINT2(x[0][1][0][1]) = AS_UINT2(&c_IV_512[10]); \
	AS_UINT2(x[0][1][1][0]) = AS_UINT2(&c_IV_512[12]); \
	AS_UINT2(x[0][1][1][1]) = AS_UINT2(&c_IV_512[14]); \
	AS_UINT2(x[1][0][0][0]) = AS_UINT2(&c_IV_512[16]); \
	AS_UINT2(x[1][0][0][1]) = AS_UINT2(&c_IV_512[18]); \
	AS_UINT2(x[1][0][1][0]) = AS_UINT2(&c_IV_512[20]); \
	AS_UINT2(x[1][0][1][1]) = AS_UINT2(&c_IV_512[22]); \
	AS_UINT2(x[1][1][0][0]) = AS_UINT2(&c_IV_512[24]); \
	AS_UINT2(x[1][1][0][1]) = AS_UINT2(&c_IV_512[26]); \
	AS_UINT2(x[1][1][1][0]) = AS_UINT2(&c_IV_512[28]); \
	AS_UINT2(x[1][1][1][1]) = AS_UINT2(&c_IV_512[30]);

__device__ __forceinline__
static void Update32(uint32_t x[2][2][2][2][2], uint32_t* const data)
{
	/* "xor the block into the first b bytes of the state" */
	block_tox(data, x);
	/* "and then transform the state invertibly through r identical rounds" */
	rrounds(x);
}

__device__ __forceinline__
static void Final(uint32_t x[2][2][2][2][2], uint32_t *hashval)
{
	/* "the integer 1 is xored into the last state word x_11111" */
	x[1][1][1][1][1] ^= 1;

	/* "the state is then transformed invertibly through 10r identical rounds" */
	#pragma unroll 10
	for (int i = 0; i < 10; i++) rrounds(x);

	/* "output the first h/8 bytes of the state" */
	hash_fromx(hashval, x);
}


/***************************************************/

__global__
void x11_cubehash512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint32_t *Hash = (uint32_t*)&g_hash[8 * hashPosition];

		uint32_t x[2][2][2][2][2];
		Init(x);

		Update32(x, &Hash[0]);
		Update32(x, &Hash[8]);

		// Padding Block
		uint32_t last[8];
		last[0] = 0x80;
		#pragma unroll 7
		for (int i=1; i < 8; i++) last[i] = 0;
		Update32(x, last);

		Final(x, Hash);
	}
}

__host__
void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;

	x11_cubehash512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
}

__host__
void x11_cubehash512_cpu_init(int thr_id, uint32_t threads) { }


/***************************************************/

#define WANT_CUBEHASH80
#ifdef WANT_CUBEHASH80

__constant__
static uint32_t c_PaddedMessage80[20];

__host__
void cubehash512_setBlock_80(int thr_id, uint32_t* endiandata)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), endiandata, sizeof(c_PaddedMessage80), 0, hipMemcpyHostToDevice);
}

__global__
void cubehash512_gpu_hash_80(const uint32_t threads, const uint32_t startNounce, uint64_t *g_outhash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNounce + thread;

		uint32_t x[2][2][2][2][2];
		Init(x);

		uint32_t message[8];
		// first 32 bytes
		AS_UINT4(&message[0]) = AS_UINT4(&c_PaddedMessage80[0]);
		AS_UINT4(&message[4]) = AS_UINT4(&c_PaddedMessage80[4]);
		Update32(x, message);

		// second 32 bytes
		AS_UINT4(&message[0]) = AS_UINT4(&c_PaddedMessage80[8]);
		AS_UINT4(&message[4]) = AS_UINT4(&c_PaddedMessage80[12]);
		Update32(x, message);

		// last 16 bytes + Padding
		AS_UINT4(&message[0]) = AS_UINT4(&c_PaddedMessage80[16]);
		message[3] = cuda_swab32(nonce);
		message[4] = 0x80;
		message[5] = 0;
		message[6] = 0;
		message[7] = 0;
		Update32(x, message);

		uint32_t* output = (uint32_t*) (&g_outhash[(size_t)8 * thread]);
		Final(x, output);
	}
}

__host__
void cubehash512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 256;
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cubehash512_gpu_hash_80 <<<grid, block>>> (threads, startNounce, (uint64_t*) d_hash);
}

#endif
=======
#include "cuda_helper.h"

#define ROUND_EVEN   \
		xg = (x0 + xg); \
		x0 = ROTL32c(x0, 7); \
		xh = (x1 + xh); \
		x1 = ROTL32c(x1, 7); \
		xi = (x2 + xi); \
		x2 = ROTL32c(x2, 7); \
		xj = (x3 + xj); \
		x3 = ROTL32c(x3, 7); \
		xk = (x4 + xk); \
		x4 = ROTL32c(x4, 7); \
		xl = (x5 + xl); \
		x5 = ROTL32c(x5, 7); \
		xm = (x6 + xm); \
		x6 = ROTL32c(x6, 7); \
		xn = (x7 + xn); \
		x7 = ROTL32c(x7, 7); \
		xo = (x8 + xo); \
		x8 = ROTL32c(x8, 7); \
		xp = (x9 + xp); \
		x9 = ROTL32c(x9, 7); \
		xq = (xa + xq); \
		xa = ROTL32c(xa, 7); \
		xr = (xb + xr); \
		xb = ROTL32c(xb, 7); \
		xs = (xc + xs); \
		xc = ROTL32c(xc, 7); \
		xt = (xd + xt); \
		xd = ROTL32c(xd, 7); \
		xu = (xe + xu); \
		xe = ROTL32c(xe, 7); \
		xv = (xf + xv); \
		xf = ROTL32c(xf, 7); \
		x8 ^= xg; \
		x9 ^= xh; \
		xa ^= xi; \
		xb ^= xj; \
		xc ^= xk; \
		xd ^= xl; \
		xe ^= xm; \
		xf ^= xn; \
		x0 ^= xo; \
		x1 ^= xp; \
		x2 ^= xq; \
		x3 ^= xr; \
		x4 ^= xs; \
		x5 ^= xt; \
		x6 ^= xu; \
		x7 ^= xv; \
		xi = (x8 + xi); \
		x8 = ROTL32c(x8, 11); \
		xj = (x9 + xj); \
		x9 = ROTL32c(x9, 11); \
		xg = (xa + xg); \
		xa = ROTL32c(xa, 11); \
		xh = (xb + xh); \
		xb = ROTL32c(xb, 11); \
		xm = (xc + xm); \
		xc = ROTL32c(xc, 11); \
		xn = (xd + xn); \
		xd = ROTL32c(xd, 11); \
		xk = (xe + xk); \
		xe = ROTL32c(xe, 11); \
		xl = (xf + xl); \
		xf = ROTL32c(xf, 11); \
		xq = (x0 + xq); \
		x0 = ROTL32c(x0, 11); \
		xr = (x1 + xr); \
		x1 = ROTL32c(x1, 11); \
		xo = (x2 + xo); \
		x2 = ROTL32c(x2, 11); \
		xp = (x3 + xp); \
		x3 = ROTL32c(x3, 11); \
		xu = (x4 + xu); \
		x4 = ROTL32c(x4, 11); \
		xv = (x5 + xv); \
		x5 = ROTL32c(x5, 11); \
		xs = (x6 + xs); \
		x6 = ROTL32c(x6, 11); \
		xt = (x7 + xt); \
		x7 = ROTL32c(x7, 11); \
		xc ^= xi; \
		xd ^= xj; \
		xe ^= xg; \
		xf ^= xh; \
		x8 ^= xm; \
		x9 ^= xn; \
		xa ^= xk; \
		xb ^= xl; \
		x4 ^= xq; \
		x5 ^= xr; \
		x6 ^= xo; \
		x7 ^= xp; \
		x0 ^= xu; \
		x1 ^= xv; \
		x2 ^= xs; \
		x3 ^= xt; 

#define ROUND_ODD    \
		xj = (xc + xj); \
		xc = ROTL32c(xc, 7); \
		xi = (xd + xi); \
		xd = ROTL32c(xd, 7); \
		xh = (xe + xh); \
		xe = ROTL32c(xe, 7); \
		xg = (xf + xg); \
		xf = ROTL32c(xf, 7); \
		xn = (x8 + xn); \
		x8 = ROTL32c(x8, 7); \
		xm = (x9 + xm); \
		x9 = ROTL32c(x9, 7); \
		xl = (xa + xl); \
		xa = ROTL32c(xa, 7); \
		xk = (xb + xk); \
		xb = ROTL32c(xb, 7); \
		xr = (x4 + xr); \
		x4 = ROTL32c(x4, 7); \
		xq = (x5 + xq); \
		x5 = ROTL32c(x5, 7); \
		xp = (x6 + xp); \
		x6 = ROTL32c(x6, 7); \
		xo = (x7 + xo); \
		x7 = ROTL32c(x7, 7); \
		xv = (x0 + xv); \
		x0 = ROTL32c(x0, 7); \
		xu = (x1 + xu); \
		x1 = ROTL32c(x1, 7); \
		xt = (x2 + xt); \
		x2 = ROTL32c(x2, 7); \
		xs = (x3 + xs); \
		x3 = ROTL32c(x3, 7); \
		x4 ^= xj; \
		x5 ^= xi; \
		x6 ^= xh; \
		x7 ^= xg; \
		x0 ^= xn; \
		x1 ^= xm; \
		x2 ^= xl; \
		x3 ^= xk; \
		xc ^= xr; \
		xd ^= xq; \
		xe ^= xp; \
		xf ^= xo; \
		x8 ^= xv; \
		x9 ^= xu; \
		xa ^= xt; \
		xb ^= xs; \
		xh = (x4 + xh); \
		x4 = ROTL32c(x4, 11); \
		xg = (x5 + xg); \
		x5 = ROTL32c(x5, 11); \
		xj = (x6 + xj); \
		x6 = ROTL32c(x6, 11); \
		xi = (x7 + xi); \
		x7 = ROTL32c(x7, 11); \
		xl = (x0 + xl); \
		x0 = ROTL32c(x0, 11); \
		xk = (x1 + xk); \
		x1 = ROTL32c(x1, 11); \
		xn = (x2 + xn); \
		x2 = ROTL32c(x2, 11); \
		xm = (x3 + xm); \
		x3 = ROTL32c(x3, 11); \
		xp = (xc + xp); \
		xc = ROTL32c(xc, 11); \
		xo = (xd + xo); \
		xd = ROTL32c(xd, 11); \
		xr = (xe + xr); \
		xe = ROTL32c(xe, 11); \
		xq = (xf + xq); \
		xf = ROTL32c(xf, 11); \
		xt = (x8 + xt); \
		x8 = ROTL32c(x8, 11); \
		xs = (x9 + xs); \
		x9 = ROTL32c(x9, 11); \
		xv = (xa + xv); \
		xa = ROTL32c(xa, 11); \
		xu = (xb + xu); \
		xb = ROTL32c(xb, 11); \
		x0 ^= xh; \
		x1 ^= xg; \
		x2 ^= xj; \
		x3 ^= xi; \
		x4 ^= xl; \
		x5 ^= xk; \
		x6 ^= xn; \
		x7 ^= xm; \
		x8 ^= xp; \
		x9 ^= xo; \
		xa ^= xr; \
		xb ^= xq; \
		xc ^= xt; \
		xd ^= xs; \
		xe ^= xv; \
		xf ^= xu; 

#define SIXTEEN_ROUNDS \
		for (int j = 0; j < 8; j ++) { \
			ROUND_EVEN; \
			ROUND_ODD;}
__global__	
void x11_cubehash512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *g_hash)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (startNounce + thread);
        int hashPosition = nounce - startNounce;
		int i, j;
        uint32_t *Hash = &g_hash[16 * hashPosition];

		uint32_t x0 = 0x2AEA2A61 ^ Hash[0];
		uint32_t x1 = 0x50F494D4 ^ Hash[1];
		uint32_t x2 = 0x2D538B8B ^ Hash[2];
		uint32_t x3 = 0x4167D83E ^ Hash[3];
		uint32_t x4 = 0x3FEE2313 ^ Hash[4];
		uint32_t x5 = 0xC701CF8C ^ Hash[5];
		uint32_t x6 = 0xCC39968E ^ Hash[6];
		uint32_t x7 = 0x50AC5695 ^ Hash[7];
		uint32_t x8 = 0x4D42C787, x9 = 0xA647A8B3, xa = 0x97CF0BEF, xb = 0x825B4537;
		uint32_t xc = 0xEEF864D2, xd = 0xF22090C4, xe = 0xD0E5CD33, xf = 0xA23911AE;
		uint32_t xg = 0xFCD398D9 + x0, xh = 0x148FE485 + x1, xi = 0x1B017BEF + x2, xj = 0xB6444532 + x3;
		uint32_t xk = 0x6A536159 + x4, xl = 0x2FF5781C + x5, xm = 0x91FA7934 + x6, xn = 0x0DBADEA9 + x7;
		uint32_t xo = 0xD65C8A2B + x8, xp = 0xA5A70E75 + x9, xq = 0xB1C62456 + xa, xr = 0xBC796576 + xb;
		uint32_t xs = 0x1921C8F7 + xc, xt = 0xE7989AF1 + xd, xu = 0x7795D246 + xe, xv = 0xD43E3B44 + xf;


		x0 = ROTL32c(x0, 7);
		x1 = ROTL32c(x1, 7);
		x2 = ROTL32c(x2, 7);
		x3 = ROTL32c(x3, 7);
		x4 = ROTL32c(x4, 7);
		x5 = ROTL32c(x5, 7);
		x6 = ROTL32c(x6, 7);
		x7 = ROTL32c(x7, 7);
		x8 = ROTL32c(x8, 7);
		x9 = ROTL32c(x9, 7);
		xa = ROTL32c(xa, 7);
		xb = ROTL32c(xb, 7);
		xc = ROTL32c(xc, 7);
		xd = ROTL32c(xd, 7);
		xe = ROTL32c(xe, 7);
		xf = ROTL32c(xf, 7);
		x8 ^= xg;
		x9 ^= xh;
		xa ^= xi;
		xb ^= xj;
		xc ^= xk;
		xd ^= xl;
		xe ^= xm;
		xf ^= xn;
		x0 ^= xo;
		x1 ^= xp;
		x2 ^= xq;
		x3 ^= xr;
		x4 ^= xs;
		x5 ^= xt;
		x6 ^= xu;
		x7 ^= xv;
		xi = (x8 + xi);
		x8 = ROTL32c(x8, 11);
		xj = (x9 + xj);
		x9 = ROTL32c(x9, 11);
		xg = (xa + xg);
		xa = ROTL32c(xa, 11);
		xh = (xb + xh);
		xb = ROTL32c(xb, 11);
		xm = (xc + xm);
		xc = ROTL32c(xc, 11);
		xn = (xd + xn);
		xd = ROTL32c(xd, 11);
		xk = (xe + xk);
		xe = ROTL32c(xe, 11);
		xl = (xf + xl);
		xf = ROTL32c(xf, 11);
		xq = (x0 + xq);
		x0 = ROTL32c(x0, 11);
		xr = (x1 + xr);
		x1 = ROTL32c(x1, 11);
		xo = (x2 + xo);
		x2 = ROTL32c(x2, 11);
		xp = (x3 + xp);
		x3 = ROTL32c(x3, 11);
		xu = (x4 + xu);
		x4 = ROTL32c(x4, 11);
		xv = (x5 + xv);
		x5 = ROTL32c(x5, 11);
		xs = (x6 + xs);
		x6 = ROTL32c(x6, 11);
		xt = (x7 + xt);
		x7 = ROTL32c(x7, 11);
		xc ^= xi;
		xd ^= xj;
		xe ^= xg;
		xf ^= xh;
		x8 ^= xm;
		x9 ^= xn;
		xa ^= xk;
		xb ^= xl;
		x4 ^= xq;
		x5 ^= xr;
		x6 ^= xo;
		x7 ^= xp;
		x0 ^= xu;
		x1 ^= xv;
		x2 ^= xs;
		x3 ^= xt;

		xj = (xc + xj);
		xc = ROTL32c(xc, 7);
		xi = (xd + xi);
		xd = ROTL32c(xd, 7);
		xh = (xe + xh);
		xe = ROTL32c(xe, 7);
		xg = (xf + xg);
		xf = ROTL32c(xf, 7);
		xn = (x8 + xn);
		x8 = ROTL32c(x8, 7);
		xm = (x9 + xm);
		x9 = ROTL32c(x9, 7);
		xl = (xa + xl);
		xa = ROTL32c(xa, 7);
		xk = (xb + xk);
		xb = ROTL32c(xb, 7);
		xr = (x4 + xr);
		x4 = ROTL32c(x4, 7);
		xq = (x5 + xq);
		x5 = ROTL32c(x5, 7);
		xp = (x6 + xp);
		x6 = ROTL32c(x6, 7);
		xo = (x7 + xo);
		x7 = ROTL32c(x7, 7);
		xv = (x0 + xv);
		x0 = ROTL32c(x0, 7);
		xu = (x1 + xu);
		x1 = ROTL32c(x1, 7);
		xt = (x2 + xt);
		x2 = ROTL32c(x2, 7);
		xs = (x3 + xs);
		x3 = ROTL32c(x3, 7);
		x4 ^= xj;
		x5 ^= xi;
		x6 ^= xh;
		x7 ^= xg;
		x0 ^= xn;
		x1 ^= xm;
		x2 ^= xl;
		x3 ^= xk;
		xc ^= xr;
		xd ^= xq;
		xe ^= xp;
		xf ^= xo;
		x8 ^= xv;
		x9 ^= xu;
		xa ^= xt;
		xb ^= xs;
		xh = (x4 + xh);
		x4 = ROTL32c(x4, 11);
		xg = (x5 + xg);
		x5 = ROTL32c(x5, 11);
		xj = (x6 + xj);
		x6 = ROTL32c(x6, 11);
		xi = (x7 + xi);
		x7 = ROTL32c(x7, 11);
		xl = (x0 + xl);
		x0 = ROTL32c(x0, 11);
		xk = (x1 + xk);
		x1 = ROTL32c(x1, 11);
		xn = (x2 + xn);
		x2 = ROTL32c(x2, 11);
		xm = (x3 + xm);
		x3 = ROTL32c(x3, 11);
		xp = (xc + xp);
		xc = ROTL32c(xc, 11);
		xo = (xd + xo);
		xd = ROTL32c(xd, 11);
		xr = (xe + xr);
		xe = ROTL32c(xe, 11);
		xq = (xf + xq);
		xf = ROTL32c(xf, 11);
		xt = (x8 + xt);
		x8 = ROTL32c(x8, 11);
		xs = (x9 + xs);
		x9 = ROTL32c(x9, 11);
		xv = (xa + xv);
		xa = ROTL32c(xa, 11);
		xu = (xb + xu);
		xb = ROTL32c(xb, 11);
		x0 ^= xh;
		x1 ^= xg;
		x2 ^= xj;
		x3 ^= xi;
		x4 ^= xl;
		x5 ^= xk;
		x6 ^= xn;
		x7 ^= xm;
		x8 ^= xp;
		x9 ^= xo;
		xa ^= xr;
		xb ^= xq;
		xc ^= xt;
		xd ^= xs;
		xe ^= xv;
		xf ^= xu;

		for (j = 1; j < 8; j++)
		{
			ROUND_EVEN;
			ROUND_ODD;
		}
		x0 ^= (Hash[8]);
		x1 ^= (Hash[9]);
		x2 ^= (Hash[10]);
		x3 ^= (Hash[11]);
		x4 ^= (Hash[12]);
		x5 ^= (Hash[13]);
		x6 ^= (Hash[14]);
		x7 ^= (Hash[15]);


		#pragma unroll 1
		for (j = 0; j < 8; j++)
		{
			ROUND_EVEN;
			ROUND_ODD;
		}
		x0 ^= 0x80;

		#pragma unroll 1
		for (int j = 0; j < 8; j++)
		{
			ROUND_EVEN;
			ROUND_ODD;
		}
		xv ^= 1;

		for (i = 3; i < 12; i++)
		{
#if __CUDA_ARCH__ > 500
#pragma unroll
			for (j = 0; j < 8; j++)
#else
#pragma unroll 1
			for (j = 0; j < 8; j++)
#endif
			{
				ROUND_EVEN;
				ROUND_ODD;
			}
		}
		#pragma unroll
		for (j = 0; j < 8; j++)
		{
			ROUND_EVEN;
			ROUND_ODD;
		}

		Hash[0] = x0;
		Hash[1] = x1;
		Hash[2] = x2;
		Hash[3] = x3;
		Hash[4] = x4;
		Hash[5] = x5;
		Hash[6] = x6;
		Hash[7] = x7;
		Hash[8] = x8;
		Hash[9] = x9;
		Hash[10] = xa;
		Hash[11] = xb;
		Hash[12] = xc;
		Hash[13] = xd;
		Hash[14] = xe;
		Hash[15] = xf;
	}
}
__host__
void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 256;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    x11_cubehash512_gpu_hash_64<<<grid, block>>>(threads, startNounce, d_hash);
}

>>>>>>> 8c320ca... added xevan
