#include "hip/hip_runtime.h"
<<<<<<< HEAD
#include <memory.h> // memcpy()

#include "cuda_helper.h"

extern __device__ __device_builtin__ void __threadfence_block(void);

#define TPB 128

__constant__ uint32_t c_PaddedMessage80[32]; // padded message (80 bytes + padding)

#include "cuda_x11_aes.cuh"

__device__ __forceinline__
static void AES_ROUND_NOKEY(
	const uint32_t* __restrict__ sharedMemory,
	uint32_t &x0, uint32_t &x1, uint32_t &x2, uint32_t &x3)
{
	uint32_t y0, y1, y2, y3;
	aes_round(sharedMemory,
		x0, x1, x2, x3,
		y0, y1, y2, y3);

	x0 = y0;
	x1 = y1;
	x2 = y2;
	x3 = y3;
}

__device__ __forceinline__
static void KEY_EXPAND_ELT(
=======
#include "cuda_helper.h"
#include <memory.h> // memcpy()
#include "cuda_vector.h"

#if __CUDA_ARCH__ == 500
#define TPB 384
#else
#define TPB 352
#endif
__constant__ uint32_t c_PaddedMessage80[32]; // padded message (80 bytes + padding)

#include "cuda_x11_aes.cu"

__device__ __forceinline__
 void AES_ROUND_NOKEY(
	const uint32_t* __restrict__ sharedMemory,
	uint32_t &x0, uint32_t &x1, uint32_t &x2, uint32_t &x3)
{
	aes_round(sharedMemory,
		x0, x1, x2, x3,
		x0, x1, x2, x3);
}

__device__ __forceinline__
void KEY_EXPAND_ELT(
>>>>>>> 8c320ca... added xevan
	const uint32_t* __restrict__ sharedMemory,
	uint32_t &k0, uint32_t &k1, uint32_t &k2, uint32_t &k3)
{
	uint32_t y0, y1, y2, y3;
	aes_round(sharedMemory,
		k0, k1, k2, k3,
		y0, y1, y2, y3);

	k0 = y1;
	k1 = y2;
	k2 = y3;
	k3 = y0;
}
<<<<<<< HEAD

__device__ __forceinline__
static void c512(const uint32_t* sharedMemory, uint32_t *state, uint32_t *msg, const uint32_t count)
{
	uint32_t p0, p1, p2, p3, p4, p5, p6, p7;
	uint32_t p8, p9, pA, pB, pC, pD, pE, pF;
	uint32_t x0, x1, x2, x3;
	uint32_t rk00, rk01, rk02, rk03, rk04, rk05, rk06, rk07;
	uint32_t rk08, rk09, rk0A, rk0B, rk0C, rk0D, rk0E, rk0F;
	uint32_t rk10, rk11, rk12, rk13, rk14, rk15, rk16, rk17;
	uint32_t rk18, rk19, rk1A, rk1B, rk1C, rk1D, rk1E, rk1F;
	const uint32_t counter = count;

	p0 = state[0x0];
	p1 = state[0x1];
	p2 = state[0x2];
	p3 = state[0x3];
	p4 = state[0x4];
	p5 = state[0x5];
	p6 = state[0x6];
	p7 = state[0x7];
	p8 = state[0x8];
	p9 = state[0x9];
	pA = state[0xA];
	pB = state[0xB];
	pC = state[0xC];
	pD = state[0xD];
	pE = state[0xE];
	pF = state[0xF];

	/* round 0 */
	rk00 = msg[0];
	x0 = p4 ^ msg[0];
	rk01 = msg[1];
	x1 = p5 ^ msg[1];
	rk02 = msg[2];
	x2 = p6 ^ msg[2];
	rk03 = msg[3];
	x3 = p7 ^ msg[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 = msg[4];
	x0 ^= msg[4];
	rk05 = msg[5];
	x1 ^= msg[5];
	rk06 = msg[6];
	x2 ^= msg[6];
	rk07 = msg[7];
	x3 ^= msg[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 = msg[8];
	x0 ^= msg[8];
	rk09 = msg[9];
	x1 ^= msg[9];
	rk0A = msg[10];
	x2 ^= msg[10];
	rk0B = msg[11];
	x3 ^= msg[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C = msg[12];
	x0 ^= msg[12];
	rk0D = msg[13];
	x1 ^= msg[13];
	rk0E = msg[14];
	x2 ^= msg[14];
	rk0F = msg[15];
	x3 ^= msg[15];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
=======
__device__ __forceinline__
void shavite_gpu_init(uint32_t *sharedMemory)
{
	/* each thread startup will fill a uint32 */
	if (threadIdx.x < 256) {
		/* each thread startup will fill a uint32 */
		sharedMemory[threadIdx.x] = d_AES0[threadIdx.x];
		sharedMemory[threadIdx.x + 256] = ROL8(sharedMemory[threadIdx.x]);
		sharedMemory[threadIdx.x + 512] = ROL16(sharedMemory[threadIdx.x]);
		sharedMemory[threadIdx.x + 768] = ROL24(sharedMemory[threadIdx.x]);
		//		sharedMemory[threadIdx.x + 64 * 2 ] = d_AES0[threadIdx.x + 64 * 2];
		//		sharedMemory[threadIdx.x + 64 * 2 + 256] = d_AES1[threadIdx.x + 64 * 2];
		//		sharedMemory[threadIdx.x + 64 * 2 + 512] = d_AES2[threadIdx.x + 64 * 2];
		//		sharedMemory[threadIdx.x + 64 * 2 + 768] = d_AES3[threadIdx.x + 64 * 2];
	}
}

__device__ __forceinline__
static void c512(const uint32_t*const __restrict__ sharedMemory, uint32_t *const __restrict__  state, uint32_t *const __restrict__  msg)
{
	//	uint32_t p0, p1, p2, p3, p4, p5, p6, p7;
	//	uint32_t p8, p9, pA, pB, pC, pD, pE, pF;
	//	uint32_t x0, x1, x2, x3;
	uint32_t rk[32];
	//	uint32_t i;
	const uint32_t counter = 640;

	uint32_t p0 = state[0x0];
	uint32_t p1 = state[0x1];
	uint32_t p2 = state[0x2];
	uint32_t p3 = state[0x3];
	uint32_t p4 = state[0x4];
	uint32_t p5 = state[0x5];
	uint32_t p6 = state[0x6];
	uint32_t p7 = state[0x7];
	uint32_t p8 = state[0x8];
	uint32_t p9 = state[0x9];
	uint32_t pA = state[0xA];
	uint32_t pB = state[0xB];
	uint32_t pC = state[0xC];
	uint32_t pD = state[0xD];
	uint32_t pE = state[0xE];
	uint32_t pF = state[0xF];

	uint32_t x0 = p4;
	uint32_t x1 = p5;
	uint32_t x2 = p6;
	uint32_t x3 = p7;
#pragma nounroll
	for (int i = 0; i<16; i += 4)
	{
		rk[i] = msg[i];
		x0 ^= msg[i];
		rk[i + 1] = msg[i + 1];
		x1 ^= msg[i + 1];
		rk[i + 2] = msg[i + 2];
		x2 ^= msg[i + 2];
		rk[i + 3] = msg[i + 3];
		x3 ^= msg[i + 3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	}

>>>>>>> 8c320ca... added xevan
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
<<<<<<< HEAD
	if (count == 512)
	{
		rk10 = 0x80U;
		x0 = pC ^ 0x80U;
		rk11 = 0;
		x1 = pD;
		rk12 = 0;
		x2 = pE;
		rk13 = 0;
		x3 = pF;
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk14 = 0;
		rk15 = 0;
		rk16 = 0;
		rk17 = 0;
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk18 = 0;
		rk19 = 0;
		rk1A = 0;
		rk1B = 0x02000000U;
		x3 ^= 0x02000000U;
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk1C = 0;
		rk1D = 0;
		rk1E = 0;
		rk1F = 0x02000000;
		x3 ^= 0x02000000;
	}
	else
	{
		rk10 = msg[16];
		x0 = pC ^ msg[16];
		rk11 = msg[17];
		x1 = pD ^ msg[17];
		rk12 = msg[18];
		x2 = pE ^ msg[18];
		rk13 = msg[19];
		x3 = pF ^ msg[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk14 = msg[20];
		x0 ^= msg[20];
		rk15 = msg[21];
		x1 ^= msg[21];
		rk16 = msg[22];
		x2 ^= msg[22];
		rk17 = msg[23];
		x3 ^= msg[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk18 = msg[24];
		x0 ^= msg[24];
		rk19 = msg[25];
		x1 ^= msg[25];
		rk1A = msg[26];
		x2 ^= msg[26];
		rk1B = msg[27];
		x3 ^= msg[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk1C = msg[28];
		x0 ^= msg[28];
		rk1D = msg[29];
		x1 ^= msg[29];
		rk1E = msg[30];
		x2 ^= msg[30];
		rk1F = msg[31];
		x3 ^= msg[31];
	}
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
=======
	x0 = pC;
	x1 = pD;
	x2 = pE;
	x3 = pF;

#pragma nounroll
	for (int i = 16; i<32; i += 4)
	{
		rk[i] = msg[i];
		x0 ^= msg[i];
		rk[i + 1] = msg[i + 1];
		x1 ^= msg[i + 1];
		rk[i + 2] = msg[i + 2];
		x2 ^= msg[i + 2];
		rk[i + 3] = msg[i + 3];
		x3 ^= msg[i + 3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	}
>>>>>>> 8c320ca... added xevan
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;

	// 1
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	rk00 ^= counter;
	rk03 ^= 0xFFFFFFFF;
	x0 = p0 ^ rk00;
	x1 = p1 ^ rk01;
	x2 = p2 ^ rk02;
	x3 = p3 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);

	rk[0] ^= rk[28];
	rk[1] ^= rk[29];
	rk[2] ^= rk[30];
	rk[3] ^= ~rk[31];
	rk[0] ^= counter;
	//rk[3] ^= 0xFFFFFFFF;
	x0 = p0 ^ rk[0];
	x1 = p1 ^ rk[1];
	x2 = p2 ^ rk[2];
	x3 = p3 ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
	rk[4] ^= rk[0];
	rk[5] ^= rk[1];
	rk[6] ^= rk[2];
	rk[7] ^= rk[3];
	x0 ^= rk[4];
	x1 ^= rk[5];
	x2 ^= rk[6];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
	rk[8] ^= rk[4];
	rk[9] ^= rk[5];
	rk[10] ^= rk[6];
	rk[11] ^= rk[7];
	x0 ^= rk[8];
	x1 ^= rk[9];
	x2 ^= rk[10];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
	rk[12] ^= rk[8];
	rk[13] ^= rk[9];
	rk[14] ^= rk[10];
	rk[15] ^= rk[11];
	x0 ^= rk[12];
	x1 ^= rk[13];
	x2 ^= rk[14];
	x3 ^= rk[15];

>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p8 ^ rk10;
	x1 = p9 ^ rk11;
	x2 = pA ^ rk12;
	x3 = pB ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
=======

	KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
	rk[16] ^= rk[12];
	rk[17] ^= rk[13];
	rk[18] ^= rk[14];
	rk[19] ^= rk[15];
	x0 = p8 ^ rk[16];
	x1 = p9 ^ rk[17];
	x2 = pA ^ rk[18];
	x3 = pB ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
	rk[20] ^= rk[16];
	rk[21] ^= rk[17];
	rk[22] ^= rk[18];
	rk[23] ^= rk[19];
	x0 ^= rk[20];
	x1 ^= rk[21];
	x2 ^= rk[22];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
	rk[24] ^= rk[20];
	rk[25] ^= rk[21];
	rk[26] ^= rk[22];
	rk[27] ^= rk[23];
	x0 ^= rk[24];
	x1 ^= rk[25];
	x2 ^= rk[26];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
	rk[28] ^= rk[24];
	rk[29] ^= rk[25];
	rk[30] ^= rk[26];
	rk[31] ^= rk[27];
	x0 ^= rk[28];
	x1 ^= rk[29];
	x2 ^= rk[30];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;

<<<<<<< HEAD
	rk00 ^= rk19;
	x0 = pC ^ rk00;
	rk01 ^= rk1A;
	x1 = pD ^ rk01;
	rk02 ^= rk1B;
	x2 = pE ^ rk02;
	rk03 ^= rk1C;
	x3 = pF ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
=======
	rk[0] ^= rk[25];
	x0 = pC ^ rk[0];
	rk[1] ^= rk[26];
	x1 = pD ^ rk[1];
	rk[2] ^= rk[27];
	x2 = pE ^ rk[2];
	rk[3] ^= rk[28];
	x3 = pF ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[4] ^= rk[29];
	x0 ^= rk[4];
	rk[5] ^= rk[30];
	x1 ^= rk[5];
	rk[6] ^= rk[31];
	x2 ^= rk[6];
	rk[7] ^= rk[0];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[8] ^= rk[1];
	x0 ^= rk[8];
	rk[9] ^= rk[2];
	x1 ^= rk[9];
	rk[10] ^= rk[3];
	x2 ^= rk[10];
	rk[11] ^= rk[4];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[12] ^= rk[5];
	x0 ^= rk[12];
	rk[13] ^= rk[6];
	x1 ^= rk[13];
	rk[14] ^= rk[7];
	x2 ^= rk[14];
	rk[15] ^= rk[8];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;
<<<<<<< HEAD
	rk10 ^= rk09;
	x0 = p4 ^ rk10;
	rk11 ^= rk0A;
	x1 = p5 ^ rk11;
	rk12 ^= rk0B;
	x2 = p6 ^ rk12;
	rk13 ^= rk0C;
	x3 = p7 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
=======
	rk[16] ^= rk[9];
	x0 = p4 ^ rk[16];
	rk[17] ^= rk[10];
	x1 = p5 ^ rk[17];
	rk[18] ^= rk[11];
	x2 = p6 ^ rk[18];
	rk[19] ^= rk[12];
	x3 = p7 ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[20] ^= rk[13];
	x0 ^= rk[20];
	rk[21] ^= rk[14];
	x1 ^= rk[21];
	rk[22] ^= rk[15];
	x2 ^= rk[22];
	rk[23] ^= rk[16];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[24] ^= rk[17];
	x0 ^= rk[24];
	rk[25] ^= rk[18];
	x1 ^= rk[25];
	rk[26] ^= rk[19];
	x2 ^= rk[26];
	rk[27] ^= rk[20];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[28] ^= rk[21];
	x0 ^= rk[28];
	rk[29] ^= rk[22];
	x1 ^= rk[29];
	rk[30] ^= rk[23];
	x2 ^= rk[30];
	rk[31] ^= rk[24];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
<<<<<<< HEAD

	/* round 3, 7, 11 */
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p8 ^ rk00;
	x1 = p9 ^ rk01;
	x2 = pA ^ rk02;
	x3 = pB ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
=======
	/* round 3, 7, 11 */
	KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
	rk[0] ^= rk[28];
	rk[1] ^= rk[29];
	rk[2] ^= rk[30];
	rk[3] ^= rk[31];
	x0 = p8 ^ rk[0];
	x1 = p9 ^ rk[1];
	x2 = pA ^ rk[2];
	x3 = pB ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
	rk[4] ^= rk[0];
	rk[5] ^= rk[1];
	rk[6] ^= rk[2];
	rk[7] ^= rk[3];
	x0 ^= rk[4];
	x1 ^= rk[5];
	x2 ^= rk[6];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
	rk[8] ^= rk[4];
	rk[9] ^= rk[5];
	rk[10] ^= rk[6];
	rk[11] ^= rk[7];
	x0 ^= rk[8];
	x1 ^= rk[9];
	x2 ^= rk[10];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
	rk[12] ^= rk[8];
	rk[13] ^= rk[9];
	rk[14] ^= rk[10];
	rk[15] ^= rk[11];
	x0 ^= rk[12];
	x1 ^= rk[13];
	x2 ^= rk[14];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p0 ^ rk10;
	x1 = p1 ^ rk11;
	x2 = p2 ^ rk12;
	x3 = p3 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
	rk[16] ^= rk[12];
	rk[17] ^= rk[13];
	rk[18] ^= rk[14];
	rk[19] ^= rk[15];
	x0 = p0 ^ rk[16];
	x1 = p1 ^ rk[17];
	x2 = p2 ^ rk[18];
	x3 = p3 ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
	rk[20] ^= rk[16];
	rk[21] ^= rk[17];
	rk[22] ^= rk[18];
	rk[23] ^= rk[19];
	x0 ^= rk[20];
	x1 ^= rk[21];
	x2 ^= rk[22];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
	rk[24] ^= rk[20];
	rk[25] ^= rk[21];
	rk[26] ^= rk[22];
	rk[27] ^= rk[23];
	x0 ^= rk[24];
	x1 ^= rk[25];
	x2 ^= rk[26];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
	rk[28] ^= rk[24];
	rk[29] ^= rk[25];
	rk[30] ^= rk[26];
	rk[31] ^= rk[27];
	x0 ^= rk[28];
	x1 ^= rk[29];
	x2 ^= rk[30];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
<<<<<<< HEAD

	/* round 4, 8, 12 */
	rk00 ^= rk19;
	x0 = p4 ^ rk00;
	rk01 ^= rk1A;
	x1 = p5 ^ rk01;
	rk02 ^= rk1B;
	x2 = p6 ^ rk02;
	rk03 ^= rk1C;
	x3 = p7 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
=======
	/* round 4, 8, 12 */
	rk[0] ^= rk[25];
	x0 = p4 ^ rk[0];
	rk[1] ^= rk[26];
	x1 = p5 ^ rk[1];
	rk[2] ^= rk[27];
	x2 = p6 ^ rk[2];
	rk[3] ^= rk[28];
	x3 = p7 ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[4] ^= rk[29];
	x0 ^= rk[4];
	rk[5] ^= rk[30];
	x1 ^= rk[5];
	rk[6] ^= rk[31];
	x2 ^= rk[6];
	rk[7] ^= rk[0];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[8] ^= rk[1];
	x0 ^= rk[8];
	rk[9] ^= rk[2];
	x1 ^= rk[9];
	rk[10] ^= rk[3];
	x2 ^= rk[10];
	rk[11] ^= rk[4];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[12] ^= rk[5];
	x0 ^= rk[12];
	rk[13] ^= rk[6];
	x1 ^= rk[13];
	rk[14] ^= rk[7];
	x2 ^= rk[14];
	rk[15] ^= rk[8];
	x3 ^= rk[15];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);

>>>>>>> 8c320ca... added xevan
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
<<<<<<< HEAD
	rk10 ^= rk09;
	x0 = pC ^ rk10;
	rk11 ^= rk0A;
	x1 = pD ^ rk11;
	rk12 ^= rk0B;
	x2 = pE ^ rk12;
	rk13 ^= rk0C;
	x3 = pF ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
=======
	rk[16] ^= rk[9];
	x0 = pC ^ rk[16];
	rk[17] ^= rk[10];
	x1 = pD ^ rk[17];
	rk[18] ^= rk[11];
	x2 = pE ^ rk[18];
	rk[19] ^= rk[12];
	x3 = pF ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[20] ^= rk[13];
	x0 ^= rk[20];
	rk[21] ^= rk[14];
	x1 ^= rk[21];
	rk[22] ^= rk[15];
	x2 ^= rk[22];
	rk[23] ^= rk[16];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[24] ^= rk[17];
	x0 ^= rk[24];
	rk[25] ^= rk[18];
	x1 ^= rk[25];
	rk[26] ^= rk[19];
	x2 ^= rk[26];
	rk[27] ^= rk[20];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[28] ^= rk[21];
	x0 ^= rk[28];
	rk[29] ^= rk[22];
	x1 ^= rk[29];
	rk[30] ^= rk[23];
	x2 ^= rk[30];
	rk[31] ^= rk[24];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;

	// 2
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p0 ^ rk00;
	x1 = p1 ^ rk01;
	x2 = p2 ^ rk02;
	x3 = p3 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	rk07 ^= SPH_T32(~counter);
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
	rk[0] ^= rk[28];
	rk[1] ^= rk[29];
	rk[2] ^= rk[30];
	rk[3] ^= rk[31];
	x0 = p0 ^ rk[0];
	x1 = p1 ^ rk[1];
	x2 = p2 ^ rk[2];
	x3 = p3 ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
	rk[4] ^= rk[0];
	rk[5] ^= rk[1];
	rk[6] ^= rk[2];
	rk[7] ^= rk[3];
	rk[7] ^= ~counter;
	x0 ^= rk[4];
	x1 ^= rk[5];
	x2 ^= rk[6];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
	rk[8] ^= rk[4];
	rk[9] ^= rk[5];
	rk[10] ^= rk[6];
	rk[11] ^= rk[7];
	x0 ^= rk[8];
	x1 ^= rk[9];
	x2 ^= rk[10];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
	rk[12] ^= rk[8];
	rk[13] ^= rk[9];
	rk[14] ^= rk[10];
	rk[15] ^= rk[11];
	x0 ^= rk[12];
	x1 ^= rk[13];
	x2 ^= rk[14];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p8 ^ rk10;
	x1 = p9 ^ rk11;
	x2 = pA ^ rk12;
	x3 = pB ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
	rk[16] ^= rk[12];
	rk[17] ^= rk[13];
	rk[18] ^= rk[14];
	rk[19] ^= rk[15];
	x0 = p8 ^ rk[16];
	x1 = p9 ^ rk[17];
	x2 = pA ^ rk[18];
	x3 = pB ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
	rk[20] ^= rk[16];
	rk[21] ^= rk[17];
	rk[22] ^= rk[18];
	rk[23] ^= rk[19];
	x0 ^= rk[20];
	x1 ^= rk[21];
	x2 ^= rk[22];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
	rk[24] ^= rk[20];
	rk[25] ^= rk[21];
	rk[26] ^= rk[22];
	rk[27] ^= rk[23];
	x0 ^= rk[24];
	x1 ^= rk[25];
	x2 ^= rk[26];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
	rk[28] ^= rk[24];
	rk[29] ^= rk[25];
	rk[30] ^= rk[26];
	rk[31] ^= rk[27];
	x0 ^= rk[28];
	x1 ^= rk[29];
	x2 ^= rk[30];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;

<<<<<<< HEAD
	rk00 ^= rk19;
	x0 = pC ^ rk00;
	rk01 ^= rk1A;
	x1 = pD ^ rk01;
	rk02 ^= rk1B;
	x2 = pE ^ rk02;
	rk03 ^= rk1C;
	x3 = pF ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
=======
	rk[0] ^= rk[25];
	x0 = pC ^ rk[0];
	rk[1] ^= rk[26];
	x1 = pD ^ rk[1];
	rk[2] ^= rk[27];
	x2 = pE ^ rk[2];
	rk[3] ^= rk[28];
	x3 = pF ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[4] ^= rk[29];
	x0 ^= rk[4];
	rk[5] ^= rk[30];
	x1 ^= rk[5];
	rk[6] ^= rk[31];
	x2 ^= rk[6];
	rk[7] ^= rk[0];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[8] ^= rk[1];
	x0 ^= rk[8];
	rk[9] ^= rk[2];
	x1 ^= rk[9];
	rk[10] ^= rk[3];
	x2 ^= rk[10];
	rk[11] ^= rk[4];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[12] ^= rk[5];
	x0 ^= rk[12];
	rk[13] ^= rk[6];
	x1 ^= rk[13];
	rk[14] ^= rk[7];
	x2 ^= rk[14];
	rk[15] ^= rk[8];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;
<<<<<<< HEAD
	rk10 ^= rk09;
	x0 = p4 ^ rk10;
	rk11 ^= rk0A;
	x1 = p5 ^ rk11;
	rk12 ^= rk0B;
	x2 = p6 ^ rk12;
	rk13 ^= rk0C;
	x3 = p7 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
=======
	rk[16] ^= rk[9];
	x0 = p4 ^ rk[16];
	rk[17] ^= rk[10];
	x1 = p5 ^ rk[17];
	rk[18] ^= rk[11];
	x2 = p6 ^ rk[18];
	rk[19] ^= rk[12];
	x3 = p7 ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[20] ^= rk[13];
	x0 ^= rk[20];
	rk[21] ^= rk[14];
	x1 ^= rk[21];
	rk[22] ^= rk[15];
	x2 ^= rk[22];
	rk[23] ^= rk[16];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[24] ^= rk[17];
	x0 ^= rk[24];
	rk[25] ^= rk[18];
	x1 ^= rk[25];
	rk[26] ^= rk[19];
	x2 ^= rk[26];
	rk[27] ^= rk[20];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[28] ^= rk[21];
	x0 ^= rk[28];
	rk[29] ^= rk[22];
	x1 ^= rk[29];
	rk[30] ^= rk[23];
	x2 ^= rk[30];
	rk[31] ^= rk[24];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
<<<<<<< HEAD

	/* round 3, 7, 11 */
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p8 ^ rk00;
	x1 = p9 ^ rk01;
	x2 = pA ^ rk02;
	x3 = pB ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
=======
	/* round 3, 7, 11 */
	KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
	rk[0] ^= rk[28];
	rk[1] ^= rk[29];
	rk[2] ^= rk[30];
	rk[3] ^= rk[31];
	x0 = p8 ^ rk[0];
	x1 = p9 ^ rk[1];
	x2 = pA ^ rk[2];
	x3 = pB ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
	rk[4] ^= rk[0];
	rk[5] ^= rk[1];
	rk[6] ^= rk[2];
	rk[7] ^= rk[3];
	x0 ^= rk[4];
	x1 ^= rk[5];
	x2 ^= rk[6];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
	rk[8] ^= rk[4];
	rk[9] ^= rk[5];
	rk[10] ^= rk[6];
	rk[11] ^= rk[7];
	x0 ^= rk[8];
	x1 ^= rk[9];
	x2 ^= rk[10];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
	rk[12] ^= rk[8];
	rk[13] ^= rk[9];
	rk[14] ^= rk[10];
	rk[15] ^= rk[11];
	x0 ^= rk[12];
	x1 ^= rk[13];
	x2 ^= rk[14];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p0 ^ rk10;
	x1 = p1 ^ rk11;
	x2 = p2 ^ rk12;
	x3 = p3 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
	rk[16] ^= rk[12];
	rk[17] ^= rk[13];
	rk[18] ^= rk[14];
	rk[19] ^= rk[15];
	x0 = p0 ^ rk[16];
	x1 = p1 ^ rk[17];
	x2 = p2 ^ rk[18];
	x3 = p3 ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
	rk[20] ^= rk[16];
	rk[21] ^= rk[17];
	rk[22] ^= rk[18];
	rk[23] ^= rk[19];
	x0 ^= rk[20];
	x1 ^= rk[21];
	x2 ^= rk[22];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
	rk[24] ^= rk[20];
	rk[25] ^= rk[21];
	rk[26] ^= rk[22];
	rk[27] ^= rk[23];
	x0 ^= rk[24];
	x1 ^= rk[25];
	x2 ^= rk[26];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
	rk[28] ^= rk[24];
	rk[29] ^= rk[25];
	rk[30] ^= rk[26];
	rk[31] ^= rk[27];
	x0 ^= rk[28];
	x1 ^= rk[29];
	x2 ^= rk[30];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
<<<<<<< HEAD

	/* round 4, 8, 12 */
	rk00 ^= rk19;
	x0 = p4 ^ rk00;
	rk01 ^= rk1A;
	x1 = p5 ^ rk01;
	rk02 ^= rk1B;
	x2 = p6 ^ rk02;
	rk03 ^= rk1C;
	x3 = p7 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
=======
	/* round 4, 8, 12 */
	rk[0] ^= rk[25];
	x0 = p4 ^ rk[0];
	rk[1] ^= rk[26];
	x1 = p5 ^ rk[1];
	rk[2] ^= rk[27];
	x2 = p6 ^ rk[2];
	rk[3] ^= rk[28];
	x3 = p7 ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[4] ^= rk[29];
	x0 ^= rk[4];
	rk[5] ^= rk[30];
	x1 ^= rk[5];
	rk[6] ^= rk[31];
	x2 ^= rk[6];
	rk[7] ^= rk[0];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[8] ^= rk[1];
	x0 ^= rk[8];
	rk[9] ^= rk[2];
	x1 ^= rk[9];
	rk[10] ^= rk[3];
	x2 ^= rk[10];
	rk[11] ^= rk[4];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[12] ^= rk[5];
	x0 ^= rk[12];
	rk[13] ^= rk[6];
	x1 ^= rk[13];
	rk[14] ^= rk[7];
	x2 ^= rk[14];
	rk[15] ^= rk[8];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
<<<<<<< HEAD
	rk10 ^= rk09;
	x0 = pC ^ rk10;
	rk11 ^= rk0A;
	x1 = pD ^ rk11;
	rk12 ^= rk0B;
	x2 = pE ^ rk12;
	rk13 ^= rk0C;
	x3 = pF ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
=======
	rk[16] ^= rk[9];
	x0 = pC ^ rk[16];
	rk[17] ^= rk[10];
	x1 = pD ^ rk[17];
	rk[18] ^= rk[11];
	x2 = pE ^ rk[18];
	rk[19] ^= rk[12];
	x3 = pF ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[20] ^= rk[13];
	x0 ^= rk[20];
	rk[21] ^= rk[14];
	x1 ^= rk[21];
	rk[22] ^= rk[15];
	x2 ^= rk[22];
	rk[23] ^= rk[16];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[24] ^= rk[17];
	x0 ^= rk[24];
	rk[25] ^= rk[18];
	x1 ^= rk[25];
	rk[26] ^= rk[19];
	x2 ^= rk[26];
	rk[27] ^= rk[20];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[28] ^= rk[21];
	x0 ^= rk[28];
	rk[29] ^= rk[22];
	x1 ^= rk[29];
	rk[30] ^= rk[23];
	x2 ^= rk[30];
	rk[31] ^= rk[24];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;

	// 3
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p0 ^ rk00;
	x1 = p1 ^ rk01;
	x2 = p2 ^ rk02;
	x3 = p3 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
	rk[0] ^= rk[28];
	rk[1] ^= rk[29];
	rk[2] ^= rk[30];
	rk[3] ^= rk[31];
	x0 = p0 ^ rk[0];
	x1 = p1 ^ rk[1];
	x2 = p2 ^ rk[2];
	x3 = p3 ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
	rk[4] ^= rk[0];
	rk[5] ^= rk[1];
	rk[6] ^= rk[2];
	rk[7] ^= rk[3];
	x0 ^= rk[4];
	x1 ^= rk[5];
	x2 ^= rk[6];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
	rk[8] ^= rk[4];
	rk[9] ^= rk[5];
	rk[10] ^= rk[6];
	rk[11] ^= rk[7];
	x0 ^= rk[8];
	x1 ^= rk[9];
	x2 ^= rk[10];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
	rk[12] ^= rk[8];
	rk[13] ^= rk[9];
	rk[14] ^= rk[10];
	rk[15] ^= rk[11];
	x0 ^= rk[12];
	x1 ^= rk[13];
	x2 ^= rk[14];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p8 ^ rk10;
	x1 = p9 ^ rk11;
	x2 = pA ^ rk12;
	x3 = pB ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	rk1E ^= counter;
	rk1F ^= 0xFFFFFFFF;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
	rk[16] ^= rk[12];
	rk[17] ^= rk[13];
	rk[18] ^= rk[14];
	rk[19] ^= rk[15];
	x0 = p8 ^ rk[16];
	x1 = p9 ^ rk[17];
	x2 = pA ^ rk[18];
	x3 = pB ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
	rk[20] ^= rk[16];
	rk[21] ^= rk[17];
	rk[22] ^= rk[18];
	rk[23] ^= rk[19];
	x0 ^= rk[20];
	x1 ^= rk[21];
	x2 ^= rk[22];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
	rk[24] ^= rk[20];
	rk[25] ^= rk[21];
	rk[26] ^= rk[22];
	rk[27] ^= rk[23];
	x0 ^= rk[24];
	x1 ^= rk[25];
	x2 ^= rk[26];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
	rk[28] ^= rk[24];
	rk[29] ^= rk[25];
	rk[30] ^= rk[26];
	rk[31] ^= ~rk[27];
	rk[30] ^= counter;
	//rk[31] ^= 0xFFFFFFFF;
	x0 ^= rk[28];
	x1 ^= rk[29];
	x2 ^= rk[30];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;

<<<<<<< HEAD
	rk00 ^= rk19;
	x0 = pC ^ rk00;
	rk01 ^= rk1A;
	x1 = pD ^ rk01;
	rk02 ^= rk1B;
	x2 = pE ^ rk02;
	rk03 ^= rk1C;
	x3 = pF ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
=======
	rk[0] ^= rk[25];
	x0 = pC ^ rk[0];
	rk[1] ^= rk[26];
	x1 = pD ^ rk[1];
	rk[2] ^= rk[27];
	x2 = pE ^ rk[2];
	rk[3] ^= rk[28];
	x3 = pF ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[4] ^= rk[29];
	x0 ^= rk[4];
	rk[5] ^= rk[30];
	x1 ^= rk[5];
	rk[6] ^= rk[31];
	x2 ^= rk[6];
	rk[7] ^= rk[0];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[8] ^= rk[1];
	x0 ^= rk[8];
	rk[9] ^= rk[2];
	x1 ^= rk[9];
	rk[10] ^= rk[3];
	x2 ^= rk[10];
	rk[11] ^= rk[4];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[12] ^= rk[5];
	x0 ^= rk[12];
	rk[13] ^= rk[6];
	x1 ^= rk[13];
	rk[14] ^= rk[7];
	x2 ^= rk[14];
	rk[15] ^= rk[8];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;
<<<<<<< HEAD
	rk10 ^= rk09;
	x0 = p4 ^ rk10;
	rk11 ^= rk0A;
	x1 = p5 ^ rk11;
	rk12 ^= rk0B;
	x2 = p6 ^ rk12;
	rk13 ^= rk0C;
	x3 = p7 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
=======
	rk[16] ^= rk[9];
	x0 = p4 ^ rk[16];
	rk[17] ^= rk[10];
	x1 = p5 ^ rk[17];
	rk[18] ^= rk[11];
	x2 = p6 ^ rk[18];
	rk[19] ^= rk[12];
	x3 = p7 ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[20] ^= rk[13];
	x0 ^= rk[20];
	rk[21] ^= rk[14];
	x1 ^= rk[21];
	rk[22] ^= rk[15];
	x2 ^= rk[22];
	rk[23] ^= rk[16];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[24] ^= rk[17];
	x0 ^= rk[24];
	rk[25] ^= rk[18];
	x1 ^= rk[25];
	rk[26] ^= rk[19];
	x2 ^= rk[26];
	rk[27] ^= rk[20];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[28] ^= rk[21];
	x0 ^= rk[28];
	rk[29] ^= rk[22];
	x1 ^= rk[29];
	rk[30] ^= rk[23];
	x2 ^= rk[30];
	rk[31] ^= rk[24];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;

<<<<<<< HEAD
	/* round 3, 7, 11 */
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p8 ^ rk00;
	x1 = p9 ^ rk01;
	x2 = pA ^ rk02;
	x3 = pB ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
=======

	/* round 3, 7, 11 */
	KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
	rk[0] ^= rk[28];
	rk[1] ^= rk[29];
	rk[2] ^= rk[30];
	rk[3] ^= rk[31];
	x0 = p8 ^ rk[0];
	x1 = p9 ^ rk[1];
	x2 = pA ^ rk[2];
	x3 = pB ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
	rk[4] ^= rk[0];
	rk[5] ^= rk[1];
	rk[6] ^= rk[2];
	rk[7] ^= rk[3];
	x0 ^= rk[4];
	x1 ^= rk[5];
	x2 ^= rk[6];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
	rk[8] ^= rk[4];
	rk[9] ^= rk[5];
	rk[10] ^= rk[6];
	rk[11] ^= rk[7];
	x0 ^= rk[8];
	x1 ^= rk[9];
	x2 ^= rk[10];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
	rk[12] ^= rk[8];
	rk[13] ^= rk[9];
	rk[14] ^= rk[10];
	rk[15] ^= rk[11];
	x0 ^= rk[12];
	x1 ^= rk[13];
	x2 ^= rk[14];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p0 ^ rk10;
	x1 = p1 ^ rk11;
	x2 = p2 ^ rk12;
	x3 = p3 ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15;
	rk1A ^= rk16;
	rk1B ^= rk17;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
	rk[16] ^= rk[12];
	rk[17] ^= rk[13];
	rk[18] ^= rk[14];
	rk[19] ^= rk[15];
	x0 = p0 ^ rk[16];
	x1 = p1 ^ rk[17];
	x2 = p2 ^ rk[18];
	x3 = p3 ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
	rk[20] ^= rk[16];
	rk[21] ^= rk[17];
	rk[22] ^= rk[18];
	rk[23] ^= rk[19];
	x0 ^= rk[20];
	x1 ^= rk[21];
	x2 ^= rk[22];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
	rk[24] ^= rk[20];
	rk[25] ^= rk[21];
	rk[26] ^= rk[22];
	rk[27] ^= rk[23];
	x0 ^= rk[24];
	x1 ^= rk[25];
	x2 ^= rk[26];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
	rk[28] ^= rk[24];
	rk[29] ^= rk[25];
	rk[30] ^= rk[26];
	rk[31] ^= rk[27];
	x0 ^= rk[28];
	x1 ^= rk[29];
	x2 ^= rk[30];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
	/* round 4, 8, 12 */
<<<<<<< HEAD
	rk00 ^= rk19;
	x0 = p4 ^ rk00;
	rk01 ^= rk1A;
	x1 = p5 ^ rk01;
	rk02 ^= rk1B;
	x2 = p6 ^ rk02;
	rk03 ^= rk1C;
	x3 = p7 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk04 ^= rk1D;
	x0 ^= rk04;
	rk05 ^= rk1E;
	x1 ^= rk05;
	rk06 ^= rk1F;
	x2 ^= rk06;
	rk07 ^= rk00;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk08 ^= rk01;
	x0 ^= rk08;
	rk09 ^= rk02;
	x1 ^= rk09;
	rk0A ^= rk03;
	x2 ^= rk0A;
	rk0B ^= rk04;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk0C ^= rk05;
	x0 ^= rk0C;
	rk0D ^= rk06;
	x1 ^= rk0D;
	rk0E ^= rk07;
	x2 ^= rk0E;
	rk0F ^= rk08;
	x3 ^= rk0F;
=======
	rk[0] ^= rk[25];
	x0 = p4 ^ rk[0];
	rk[1] ^= rk[26];
	x1 = p5 ^ rk[1];
	rk[2] ^= rk[27];
	x2 = p6 ^ rk[2];
	rk[3] ^= rk[28];
	x3 = p7 ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[4] ^= rk[29];
	x0 ^= rk[4];
	rk[5] ^= rk[30];
	x1 ^= rk[5];
	rk[6] ^= rk[31];
	x2 ^= rk[6];
	rk[7] ^= rk[0];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[8] ^= rk[1];
	x0 ^= rk[8];
	rk[9] ^= rk[2];
	x1 ^= rk[9];
	rk[10] ^= rk[3];
	x2 ^= rk[10];
	rk[11] ^= rk[4];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[12] ^= rk[5];
	x0 ^= rk[12];
	rk[13] ^= rk[6];
	x1 ^= rk[13];
	rk[14] ^= rk[7];
	x2 ^= rk[14];
	rk[15] ^= rk[8];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p0 ^= x0;
	p1 ^= x1;
	p2 ^= x2;
	p3 ^= x3;
<<<<<<< HEAD
	rk10 ^= rk09;
	x0 = pC ^ rk10;
	rk11 ^= rk0A;
	x1 = pD ^ rk11;
	rk12 ^= rk0B;
	x2 = pE ^ rk12;
	rk13 ^= rk0C;
	x3 = pF ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk14 ^= rk0D;
	x0 ^= rk14;
	rk15 ^= rk0E;
	x1 ^= rk15;
	rk16 ^= rk0F;
	x2 ^= rk16;
	rk17 ^= rk10;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk18 ^= rk11;
	x0 ^= rk18;
	rk19 ^= rk12;
	x1 ^= rk19;
	rk1A ^= rk13;
	x2 ^= rk1A;
	rk1B ^= rk14;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk1C ^= rk15;
	x0 ^= rk1C;
	rk1D ^= rk16;
	x1 ^= rk1D;
	rk1E ^= rk17;
	x2 ^= rk1E;
	rk1F ^= rk18;
	x3 ^= rk1F;
=======
	rk[16] ^= rk[9];
	x0 = pC ^ rk[16];
	rk[17] ^= rk[10];
	x1 = pD ^ rk[17];
	rk[18] ^= rk[11];
	x2 = pE ^ rk[18];
	rk[19] ^= rk[12];
	x3 = pF ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[20] ^= rk[13];
	x0 ^= rk[20];
	rk[21] ^= rk[14];
	x1 ^= rk[21];
	rk[22] ^= rk[15];
	x2 ^= rk[22];
	rk[23] ^= rk[16];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[24] ^= rk[17];
	x0 ^= rk[24];
	rk[25] ^= rk[18];
	x1 ^= rk[25];
	rk[26] ^= rk[19];
	x2 ^= rk[26];
	rk[27] ^= rk[20];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	rk[28] ^= rk[21];
	x0 ^= rk[28];
	rk[29] ^= rk[22];
	x1 ^= rk[29];
	rk[30] ^= rk[23];
	x2 ^= rk[30];
	rk[31] ^= rk[24];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p8 ^= x0;
	p9 ^= x1;
	pA ^= x2;
	pB ^= x3;

	/* round 13 */
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk00, rk01, rk02, rk03);
	rk00 ^= rk1C;
	rk01 ^= rk1D;
	rk02 ^= rk1E;
	rk03 ^= rk1F;
	x0 = p0 ^ rk00;
	x1 = p1 ^ rk01;
	x2 = p2 ^ rk02;
	x3 = p3 ^ rk03;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk04, rk05, rk06, rk07);
	rk04 ^= rk00;
	rk05 ^= rk01;
	rk06 ^= rk02;
	rk07 ^= rk03;
	x0 ^= rk04;
	x1 ^= rk05;
	x2 ^= rk06;
	x3 ^= rk07;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk08, rk09, rk0A, rk0B);
	rk08 ^= rk04;
	rk09 ^= rk05;
	rk0A ^= rk06;
	rk0B ^= rk07;
	x0 ^= rk08;
	x1 ^= rk09;
	x2 ^= rk0A;
	x3 ^= rk0B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk0C, rk0D, rk0E, rk0F);
	rk0C ^= rk08;
	rk0D ^= rk09;
	rk0E ^= rk0A;
	rk0F ^= rk0B;
	x0 ^= rk0C;
	x1 ^= rk0D;
	x2 ^= rk0E;
	x3 ^= rk0F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
	rk[0] ^= rk[28];
	rk[1] ^= rk[29];
	rk[2] ^= rk[30];
	rk[3] ^= rk[31];
	x0 = p0 ^ rk[0];
	x1 = p1 ^ rk[1];
	x2 = p2 ^ rk[2];
	x3 = p3 ^ rk[3];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
	rk[4] ^= rk[0];
	rk[5] ^= rk[1];
	rk[6] ^= rk[2];
	rk[7] ^= rk[3];
	x0 ^= rk[4];
	x1 ^= rk[5];
	x2 ^= rk[6];
	x3 ^= rk[7];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
	rk[8] ^= rk[4];
	rk[9] ^= rk[5];
	rk[10] ^= rk[6];
	rk[11] ^= rk[7];
	x0 ^= rk[8];
	x1 ^= rk[9];
	x2 ^= rk[10];
	x3 ^= rk[11];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
	rk[12] ^= rk[8];
	rk[13] ^= rk[9];
	rk[14] ^= rk[10];
	rk[15] ^= rk[11];
	x0 ^= rk[12];
	x1 ^= rk[13];
	x2 ^= rk[14];
	x3 ^= rk[15];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	pC ^= x0;
	pD ^= x1;
	pE ^= x2;
	pF ^= x3;
<<<<<<< HEAD
	KEY_EXPAND_ELT(sharedMemory, rk10, rk11, rk12, rk13);
	rk10 ^= rk0C;
	rk11 ^= rk0D;
	rk12 ^= rk0E;
	rk13 ^= rk0F;
	x0 = p8 ^ rk10;
	x1 = p9 ^ rk11;
	x2 = pA ^ rk12;
	x3 = pB ^ rk13;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk14, rk15, rk16, rk17);
	rk14 ^= rk10;
	rk15 ^= rk11;
	rk16 ^= rk12;
	rk17 ^= rk13;
	x0 ^= rk14;
	x1 ^= rk15;
	x2 ^= rk16;
	x3 ^= rk17;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk18, rk19, rk1A, rk1B);
	rk18 ^= rk14;
	rk19 ^= rk15 ^ counter;
	rk1A ^= rk16;
	rk1B ^= rk17 ^ 0xFFFFFFFF;
	x0 ^= rk18;
	x1 ^= rk19;
	x2 ^= rk1A;
	x3 ^= rk1B;
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk1C, rk1D, rk1E, rk1F);
	rk1C ^= rk18;
	rk1D ^= rk19;
	rk1E ^= rk1A;
	rk1F ^= rk1B;
	x0 ^= rk1C;
	x1 ^= rk1D;
	x2 ^= rk1E;
	x3 ^= rk1F;
=======
	KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
	rk[16] ^= rk[12];
	rk[17] ^= rk[13];
	rk[18] ^= rk[14];
	rk[19] ^= rk[15];
	x0 = p8 ^ rk[16];
	x1 = p9 ^ rk[17];
	x2 = pA ^ rk[18];
	x3 = pB ^ rk[19];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
	rk[20] ^= rk[16];
	rk[21] ^= rk[17];
	rk[22] ^= rk[18];
	rk[23] ^= rk[19];
	x0 ^= rk[20];
	x1 ^= rk[21];
	x2 ^= rk[22];
	x3 ^= rk[23];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
	rk[24] ^= rk[20];
	rk[25] ^= rk[21] ^ counter;
	rk[26] ^= rk[22];
	rk[27] ^= ~rk[23]; //^ 0xFFFFFFFF;
	x0 ^= rk[24];
	x1 ^= rk[25];
	x2 ^= rk[26];
	x3 ^= rk[27];
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
	rk[28] ^= rk[24];
	rk[29] ^= rk[25];
	rk[30] ^= rk[26];
	rk[31] ^= rk[27];
	x0 ^= rk[28];
	x1 ^= rk[29];
	x2 ^= rk[30];
	x3 ^= rk[31];
>>>>>>> 8c320ca... added xevan
	AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
	p4 ^= x0;
	p5 ^= x1;
	p6 ^= x2;
	p7 ^= x3;
	state[0x0] ^= p8;
	state[0x1] ^= p9;
	state[0x2] ^= pA;
	state[0x3] ^= pB;
	state[0x4] ^= pC;
	state[0x5] ^= pD;
	state[0x6] ^= pE;
	state[0x7] ^= pF;
	state[0x8] ^= p0;
	state[0x9] ^= p1;
	state[0xA] ^= p2;
	state[0xB] ^= p3;
	state[0xC] ^= p4;
	state[0xD] ^= p5;
	state[0xE] ^= p6;
	state[0xF] ^= p7;
}

<<<<<<< HEAD
__device__ __forceinline__
void shavite_gpu_init(uint32_t *sharedMemory)
{
	/* each thread startup will fill a uint32 */
	if (threadIdx.x < 128) {
		sharedMemory[threadIdx.x] = d_AES0[threadIdx.x];
		sharedMemory[threadIdx.x + 256] = d_AES1[threadIdx.x];
		sharedMemory[threadIdx.x + 512] = d_AES2[threadIdx.x];
		sharedMemory[threadIdx.x + 768] = d_AES3[threadIdx.x];

		sharedMemory[threadIdx.x + 64 * 2] = d_AES0[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 256] = d_AES1[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 512] = d_AES2[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 768] = d_AES3[threadIdx.x + 64 * 2];
	}
}

// GPU Hash
__global__ __launch_bounds__(TPB, 7) /* 64 registers with 128,8 - 72 regs with 128,7 */
void x11_shavite512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	__shared__ uint32_t sharedMemory[1024];

	shavite_gpu_init(sharedMemory);
	__threadfence_block();
=======
__constant__ uint32_t cstate[16] =
{
	(0x72FCCDD8), (0x79CA4727), (0x128A077B), (0x40D55AEC),
	(0xD1901A06), (0x430AE307), (0xB29F5CD1), (0xDF07FBFC),
	(0x8E45D73D), (0x681AB538), (0xBDE86578), (0xDD577E47),
	(0xE275EADE), (0x502D9FCD), (0xB9357178), (0x022A4B9A)
};

__global__ __launch_bounds__(TPB, 2)
void x11_shavite512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *const __restrict__ g_hash)
{
	__shared__  __align__(32) uint32_t sharedMemory[1024];

	shavite_gpu_init(sharedMemory);
>>>>>>> 8c320ca... added xevan

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
<<<<<<< HEAD
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint32_t *Hash = (uint32_t*)&g_hash[hashPosition<<3];

		// kopiere init-state
		uint32_t state[16] = {
			SPH_C32(0x72FCCDD8), SPH_C32(0x79CA4727), SPH_C32(0x128A077B), SPH_C32(0x40D55AEC),
			SPH_C32(0xD1901A06), SPH_C32(0x430AE307), SPH_C32(0xB29F5CD1), SPH_C32(0xDF07FBFC),
			SPH_C32(0x8E45D73D), SPH_C32(0x681AB538), SPH_C32(0xBDE86578), SPH_C32(0xDD577E47),
			SPH_C32(0xE275EADE), SPH_C32(0x502D9FCD), SPH_C32(0xB9357178), SPH_C32(0x022A4B9A)
		};

		// nachricht laden
		uint32_t msg[32];

		// fülle die Nachricht mit 64-byte (vorheriger Hash)
		#pragma unroll 16
		for(int i=0;i<16;i++)
			msg[i] = Hash[i];

		// Nachrichtenende
		msg[16] = 0x80;
		#pragma unroll 10
		for(int i=17;i<27;i++)
			msg[i] = 0;

		msg[27] = 0x02000000;
		msg[28] = 0;
		msg[29] = 0;
		msg[30] = 0;
		msg[31] = 0x02000000;

		c512(sharedMemory, state, msg, 512);

		#pragma unroll 16
		for(int i=0;i<16;i++)
			Hash[i] = state[i];
	}
}

__global__ __launch_bounds__(TPB, 7)
=======
		uint32_t nounce = (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint32_t *Hash = (uint32_t*)&g_hash[hashPosition*8];


		// kopiere init-state

		uint32_t rk[32];
		uint32_t msg[16];
//		{
//			Hash[0], Hash[1], Hash[2], Hash[3], Hash[4], Hash[5], Hash[6], Hash[7], Hash[8], Hash[9], Hash[10], Hash[11], Hash[12], Hash[13], Hash[14], Hash[15]
//		};


		uint28 *phash = (uint28*)Hash;
		uint28 *outpt = (uint28*)msg;
		outpt[0] = phash[0];
		outpt[1] = phash[1];

		uint32_t state[16]=
		{
			cstate[0], cstate[1], cstate[2], cstate[3],
			cstate[4], cstate[5], cstate[6], cstate[7],
			cstate[8], cstate[9], cstate[10], cstate[11],
			cstate[12], cstate[13], cstate[14], cstate[15],
		};

		/*
		if (threadIdx.x == 0) 
		{

			((uint16*)state)[0] = make_uint16(
				(0x72FCCDD8), (0x79CA4727), (0x128A077B), (0x40D55AEC),
				(0xD1901A06), (0x430AE307), (0xB29F5CD1), (0xDF07FBFC),
				(0x8E45D73D), (0x681AB538), (0xBDE86578), (0xDD577E47),
				(0xE275EADE), (0x502D9FCD), (0xB9357178), (0x022A4B9A)
				);
		}

*/

/*		uint32_t p0 = state[0x0];
		uint32_t p1 = state[0x1];
		uint32_t p2 = state[0x2];
		uint32_t p3 = state[0x3];
		uint32_t state[4] = state[0x4];
		uint32_t state[5] = state[0x5];
		uint32_t state[6] = state[0x6];
		uint32_t state[7] = state[0x7];
		uint32_t state[8] = state[0x8];
		uint32_t state[9] = state[0x9];
*/
//		uint32_t pA = state[0xA];
//		uint32_t pB = state[0xB];
//		uint32_t pC = state[0xC];
//		uint32_t pD = state[0xD];
//		uint32_t pE = state[0xE];
//		uint32_t pF = state[0xF];

		uint32_t x0 = state[0x4];
		uint32_t x1 = state[0x5];
		uint32_t x2 = state[0x6];
		uint32_t x3 = state[0x7];

		for (int i = 0; i < 16; i+=4)
		{

			rk[i + 0] = msg[i + 0];
			x0 ^= msg[i + 0];
			rk[i + 1] = msg[i + 1];
			x1 ^= msg[i + 1];
			rk[i + 2] = msg[i + 2];
			x2 ^= msg[i + 2];
			rk[i + 3] = msg[i + 3];
			x3 ^= msg[i + 3];
			AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		}
		state[0] ^= x0;
		state[1] ^= x1;
		state[2] ^= x2;
		state[3] ^= x3;

		// 1
		KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);

		rk[3] ^= (0x02000000UL ^ 0xFFFFFFFFUL);	//rk[31];
		rk[0] ^= 512;
		//	rk[3] ^= 0xFFFFFFFF;

		x0 = state[0] ^ rk[0];
		x1 = state[1] ^ rk[1];
		x2 = state[2] ^ rk[2];
		x3 = state[3] ^ rk[3];


		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
		rk[4] ^= rk[0];
		rk[5] ^= rk[1];
		rk[6] ^= rk[2];
		rk[7] ^= rk[3];
		x0 ^= rk[4];
		x1 ^= rk[5];
		x2 ^= rk[6];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
		rk[8] ^= rk[4];
		rk[9] ^= rk[5];
		rk[10] ^= rk[6];
		rk[11] ^= rk[7];
		x0 ^= rk[8];
		x1 ^= rk[9];
		x2 ^= rk[10];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
		rk[12] ^= rk[8];
		rk[13] ^= rk[9];
		rk[14] ^= rk[10];
		rk[15] ^= rk[11];
		x0 ^= rk[12];
		x1 ^= rk[13];
		x2 ^= rk[14];
		x3 ^= rk[15];

		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);

		state[8] ^= 0x32be246fUL;
		state[9] ^= 0xe33ad1e5UL;
		state[10] ^= 0xd659b13eUL;
		state[11] ^= 0xb6a1a92cUL;

		state[12] ^= x0;
		state[13] ^= x1;
		state[14] ^= x2;
		state[15] ^= x3;

		rk[16] = rk[12] ^ 0x63636363UL;
		rk[17] = rk[13] ^ 0x63636363UL;
		rk[18] = rk[14] ^ 0x63636363UL;
		rk[19] = rk[15] ^ 0x8acdcd24UL;
		x0 = state[8] ^ rk[16];
		x1 = state[9] ^ rk[17];
		x2 = state[10] ^ rk[18];
		x3 = state[11] ^ rk[19];
		rk[20] = 0x63636363UL ^ rk[16];
		rk[21] = 0x63636363UL ^ rk[17];
		rk[22] = 0x63636363UL ^ rk[18];
		rk[23] = 0x63636363UL ^ rk[19];
		rk[24] = 0x63636363UL ^ rk[20];
		rk[25] = 0x63636363UL ^ rk[21];
		rk[26] = 0x63636363UL ^ rk[22];
		rk[27] = 0x4b5f7777UL ^ rk[23];

		rk[28] = 0x63636363UL ^ rk[24];
		rk[29] = 0x63636363UL ^ rk[25];
		rk[30] = 0x63636363UL ^ rk[26];
		rk[31] = 0x4b5f7777UL ^ rk[27];



		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);

		x0 ^= rk[20];
		x1 ^= rk[21];
		x2 ^= rk[22];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);


		x0 ^= rk[24];
		x1 ^= rk[25];
		x2 ^= rk[26];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);

		x0 ^= rk[28];
		x1 ^= rk[29];
		x2 ^= rk[30];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);

		state[4] ^= x0;
		state[5] ^= x1;
		state[6] ^= x2;
		state[7] ^= x3;

		rk[0] ^= rk[25];
		x0 = state[12] ^ rk[0];
		rk[1] ^= rk[26];
		x1 = state[13] ^ rk[1];
		rk[2] ^= rk[27];
		x2 = state[14] ^ rk[2];
		rk[3] ^= rk[28];
		x3 = state[15] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[4] ^= rk[29];
		x0 ^= rk[4];
		rk[5] ^= rk[30];
		x1 ^= rk[5];
		rk[6] ^= rk[31];
		x2 ^= rk[6];
		rk[7] ^= rk[0];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[8] ^= rk[1];
		x0 ^= rk[8];
		rk[9] ^= rk[2];
		x1 ^= rk[9];
		rk[10] ^= rk[3];
		x2 ^= rk[10];
		rk[11] ^= rk[4];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[12] ^= rk[5];
		x0 ^= rk[12];
		rk[13] ^= rk[6];
		x1 ^= rk[13];
		rk[14] ^= rk[7];
		x2 ^= rk[14];
		rk[15] ^= rk[8];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[8] ^= x0;
		state[9] ^= x1;
		state[10] ^= x2;
		state[11] ^= x3;
		rk[16] ^= rk[9];
		x0 = state[4] ^ rk[16];
		rk[17] ^= rk[10];
		x1 = state[5] ^ rk[17];
		rk[18] ^= rk[11];
		x2 = state[6] ^ rk[18];
		rk[19] ^= rk[12];
		x3 = state[7] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[20] ^= rk[13];
		x0 ^= rk[20];
		rk[21] ^= rk[14];
		x1 ^= rk[21];
		rk[22] ^= rk[15];
		x2 ^= rk[22];
		rk[23] ^= rk[16];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[24] ^= rk[17];
		x0 ^= rk[24];
		rk[25] ^= rk[18];
		x1 ^= rk[25];
		rk[26] ^= rk[19];
		x2 ^= rk[26];
		rk[27] ^= rk[20];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[28] ^= rk[21];
		x0 ^= rk[28];
		rk[29] ^= rk[22];
		x1 ^= rk[29];
		rk[30] ^= rk[23];
		x2 ^= rk[30];
		rk[31] ^= rk[24];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[0] ^= x0;
		state[1] ^= x1;
		state[2] ^= x2;
		state[3] ^= x3;
		/* round 3, 7, 11 */
		KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
		rk[0] ^= rk[28];
		rk[1] ^= rk[29];
		rk[2] ^= rk[30];
		rk[3] ^= rk[31];
		x0 = state[8] ^ rk[0];
		x1 = state[9] ^ rk[1];
		x2 = state[10] ^ rk[2];
		x3 = state[11] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
		rk[4] ^= rk[0];
		rk[5] ^= rk[1];
		rk[6] ^= rk[2];
		rk[7] ^= rk[3];
		x0 ^= rk[4];
		x1 ^= rk[5];
		x2 ^= rk[6];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
		rk[8] ^= rk[4];
		rk[9] ^= rk[5];
		rk[10] ^= rk[6];
		rk[11] ^= rk[7];
		x0 ^= rk[8];
		x1 ^= rk[9];
		x2 ^= rk[10];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
		rk[12] ^= rk[8];
		rk[13] ^= rk[9];
		rk[14] ^= rk[10];
		rk[15] ^= rk[11];
		x0 ^= rk[12];
		x1 ^= rk[13];
		x2 ^= rk[14];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[4] ^= x0;
		state[5] ^= x1;
		state[6] ^= x2;
		state[7] ^= x3;
		KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
		rk[16] ^= rk[12];
		rk[17] ^= rk[13];
		rk[18] ^= rk[14];
		rk[19] ^= rk[15];
		x0 = state[0] ^ rk[16];
		x1 = state[1] ^ rk[17];
		x2 = state[2] ^ rk[18];
		x3 = state[3] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
		rk[20] ^= rk[16];
		rk[21] ^= rk[17];
		rk[22] ^= rk[18];
		rk[23] ^= rk[19];
		x0 ^= rk[20];
		x1 ^= rk[21];
		x2 ^= rk[22];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
		rk[24] ^= rk[20];
		rk[25] ^= rk[21];
		rk[26] ^= rk[22];
		rk[27] ^= rk[23];
		x0 ^= rk[24];
		x1 ^= rk[25];
		x2 ^= rk[26];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
		rk[28] ^= rk[24];
		rk[29] ^= rk[25];
		rk[30] ^= rk[26];
		rk[31] ^= rk[27];
		x0 ^= rk[28];
		x1 ^= rk[29];
		x2 ^= rk[30];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[12] ^= x0;
		state[13] ^= x1;
		state[14] ^= x2;
		state[15] ^= x3;
		/* round 4, 8, 12 */
		rk[0] ^= rk[25];
		x0 = state[4] ^ rk[0];
		rk[1] ^= rk[26];
		x1 = state[5] ^ rk[1];
		rk[2] ^= rk[27];
		x2 = state[6] ^ rk[2];
		rk[3] ^= rk[28];
		x3 = state[7] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[4] ^= rk[29];
		x0 ^= rk[4];
		rk[5] ^= rk[30];
		x1 ^= rk[5];
		rk[6] ^= rk[31];
		x2 ^= rk[6];
		rk[7] ^= rk[0];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[8] ^= rk[1];
		x0 ^= rk[8];
		rk[9] ^= rk[2];
		x1 ^= rk[9];
		rk[10] ^= rk[3];
		x2 ^= rk[10];
		rk[11] ^= rk[4];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[12] ^= rk[5];
		x0 ^= rk[12];
		rk[13] ^= rk[6];
		x1 ^= rk[13];
		rk[14] ^= rk[7];
		x2 ^= rk[14];
		rk[15] ^= rk[8];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);

		state[0] ^= x0;
		state[1] ^= x1;
		state[2] ^= x2;
		state[3] ^= x3;
		rk[16] ^= rk[9];
		x0 = state[12] ^ rk[16];
		rk[17] ^= rk[10];
		x1 = state[13] ^ rk[17];
		rk[18] ^= rk[11];
		x2 = state[14] ^ rk[18];
		rk[19] ^= rk[12];
		x3 = state[15] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[20] ^= rk[13];
		x0 ^= rk[20];
		rk[21] ^= rk[14];
		x1 ^= rk[21];
		rk[22] ^= rk[15];
		x2 ^= rk[22];
		rk[23] ^= rk[16];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[24] ^= rk[17];
		x0 ^= rk[24];
		rk[25] ^= rk[18];
		x1 ^= rk[25];
		rk[26] ^= rk[19];
		x2 ^= rk[26];
		rk[27] ^= rk[20];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[28] ^= rk[21];
		x0 ^= rk[28];
		rk[29] ^= rk[22];
		x1 ^= rk[29];
		rk[30] ^= rk[23];
		x2 ^= rk[30];
		rk[31] ^= rk[24];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[8] ^= x0;
		state[9] ^= x1;
		state[10] ^= x2;
		state[11] ^= x3;

		// 2
		KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
		rk[0] ^= rk[28];
		rk[1] ^= rk[29];
		rk[2] ^= rk[30];
		rk[3] ^= rk[31];
		x0 = state[0] ^ rk[0];
		x1 = state[1] ^ rk[1];
		x2 = state[2] ^ rk[2];
		x3 = state[3] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
		rk[4] ^= rk[0];
		rk[5] ^= rk[1];
		rk[6] ^= rk[2];
		rk[7] ^= rk[3];
		rk[7] ^= ~512;
		x0 ^= rk[4];
		x1 ^= rk[5];
		x2 ^= rk[6];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
		rk[8] ^= rk[4];
		rk[9] ^= rk[5];
		rk[10] ^= rk[6];
		rk[11] ^= rk[7];
		x0 ^= rk[8];
		x1 ^= rk[9];
		x2 ^= rk[10];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
		rk[12] ^= rk[8];
		rk[13] ^= rk[9];
		rk[14] ^= rk[10];
		rk[15] ^= rk[11];
		x0 ^= rk[12];
		x1 ^= rk[13];
		x2 ^= rk[14];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[12] ^= x0;
		state[13] ^= x1;
		state[14] ^= x2;
		state[15] ^= x3;
		KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
		rk[16] ^= rk[12];
		rk[17] ^= rk[13];
		rk[18] ^= rk[14];
		rk[19] ^= rk[15];
		x0 = state[8] ^ rk[16];
		x1 = state[9] ^ rk[17];
		x2 = state[10] ^ rk[18];
		x3 = state[11] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
		rk[20] ^= rk[16];
		rk[21] ^= rk[17];
		rk[22] ^= rk[18];
		rk[23] ^= rk[19];
		x0 ^= rk[20];
		x1 ^= rk[21];
		x2 ^= rk[22];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
		rk[24] ^= rk[20];
		rk[25] ^= rk[21];
		rk[26] ^= rk[22];
		rk[27] ^= rk[23];
		x0 ^= rk[24];
		x1 ^= rk[25];
		x2 ^= rk[26];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
		rk[28] ^= rk[24];
		rk[29] ^= rk[25];
		rk[30] ^= rk[26];
		rk[31] ^= rk[27];
		x0 ^= rk[28];
		x1 ^= rk[29];
		x2 ^= rk[30];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[4] ^= x0;
		state[5] ^= x1;
		state[6] ^= x2;
		state[7] ^= x3;

		rk[0] ^= rk[25];
		x0 = state[12] ^ rk[0];
		rk[1] ^= rk[26];
		x1 = state[13] ^ rk[1];
		rk[2] ^= rk[27];
		x2 = state[14] ^ rk[2];
		rk[3] ^= rk[28];
		x3 = state[15] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[4] ^= rk[29];
		x0 ^= rk[4];
		rk[5] ^= rk[30];
		x1 ^= rk[5];
		rk[6] ^= rk[31];
		x2 ^= rk[6];
		rk[7] ^= rk[0];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[8] ^= rk[1];
		x0 ^= rk[8];
		rk[9] ^= rk[2];
		x1 ^= rk[9];
		rk[10] ^= rk[3];
		x2 ^= rk[10];
		rk[11] ^= rk[4];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[12] ^= rk[5];
		x0 ^= rk[12];
		rk[13] ^= rk[6];
		x1 ^= rk[13];
		rk[14] ^= rk[7];
		x2 ^= rk[14];
		rk[15] ^= rk[8];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[8] ^= x0;
		state[9] ^= x1;
		state[10] ^= x2;
		state[11] ^= x3;
		rk[16] ^= rk[9];
		x0 = state[4] ^ rk[16];
		rk[17] ^= rk[10];
		x1 = state[5] ^ rk[17];
		rk[18] ^= rk[11];
		x2 = state[6] ^ rk[18];
		rk[19] ^= rk[12];
		x3 = state[7] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[20] ^= rk[13];
		x0 ^= rk[20];
		rk[21] ^= rk[14];
		x1 ^= rk[21];
		rk[22] ^= rk[15];
		x2 ^= rk[22];
		rk[23] ^= rk[16];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[24] ^= rk[17];
		x0 ^= rk[24];
		rk[25] ^= rk[18];
		x1 ^= rk[25];
		rk[26] ^= rk[19];
		x2 ^= rk[26];
		rk[27] ^= rk[20];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[28] ^= rk[21];
		x0 ^= rk[28];
		rk[29] ^= rk[22];
		x1 ^= rk[29];
		rk[30] ^= rk[23];
		x2 ^= rk[30];
		rk[31] ^= rk[24];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[0] ^= x0;
		state[1] ^= x1;
		state[2] ^= x2;
		state[3] ^= x3;
		/* round 3, 7, 11 */
		KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
		rk[0] ^= rk[28];
		rk[1] ^= rk[29];
		rk[2] ^= rk[30];
		rk[3] ^= rk[31];
		x0 = state[8] ^ rk[0];
		x1 = state[9] ^ rk[1];
		x2 = state[10] ^ rk[2];
		x3 = state[11] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
		rk[4] ^= rk[0];
		rk[5] ^= rk[1];
		rk[6] ^= rk[2];
		rk[7] ^= rk[3];
		x0 ^= rk[4];
		x1 ^= rk[5];
		x2 ^= rk[6];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
		rk[8] ^= rk[4];
		rk[9] ^= rk[5];
		rk[10] ^= rk[6];
		rk[11] ^= rk[7];
		x0 ^= rk[8];
		x1 ^= rk[9];
		x2 ^= rk[10];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
		rk[12] ^= rk[8];
		rk[13] ^= rk[9];
		rk[14] ^= rk[10];
		rk[15] ^= rk[11];
		x0 ^= rk[12];
		x1 ^= rk[13];
		x2 ^= rk[14];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[4] ^= x0;
		state[5] ^= x1;
		state[6] ^= x2;
		state[7] ^= x3;
		KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
		rk[16] ^= rk[12];
		rk[17] ^= rk[13];
		rk[18] ^= rk[14];
		rk[19] ^= rk[15];
		x0 = state[0] ^ rk[16];
		x1 = state[1] ^ rk[17];
		x2 = state[2] ^ rk[18];
		x3 = state[3] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
		rk[20] ^= rk[16];
		rk[21] ^= rk[17];
		rk[22] ^= rk[18];
		rk[23] ^= rk[19];
		x0 ^= rk[20];
		x1 ^= rk[21];
		x2 ^= rk[22];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
		rk[24] ^= rk[20];
		rk[25] ^= rk[21];
		rk[26] ^= rk[22];
		rk[27] ^= rk[23];
		x0 ^= rk[24];
		x1 ^= rk[25];
		x2 ^= rk[26];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
		rk[28] ^= rk[24];
		rk[29] ^= rk[25];
		rk[30] ^= rk[26];
		rk[31] ^= rk[27];
		x0 ^= rk[28];
		x1 ^= rk[29];
		x2 ^= rk[30];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[12] ^= x0;
		state[13] ^= x1;
		state[14] ^= x2;
		state[15] ^= x3;
		/* round 4, 8, 12 */
		rk[0] ^= rk[25];
		x0 = state[4] ^ rk[0];
		rk[1] ^= rk[26];
		x1 = state[5] ^ rk[1];
		rk[2] ^= rk[27];
		x2 = state[6] ^ rk[2];
		rk[3] ^= rk[28];
		x3 = state[7] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[4] ^= rk[29];
		x0 ^= rk[4];
		rk[5] ^= rk[30];
		x1 ^= rk[5];
		rk[6] ^= rk[31];
		x2 ^= rk[6];
		rk[7] ^= rk[0];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[8] ^= rk[1];
		x0 ^= rk[8];
		rk[9] ^= rk[2];
		x1 ^= rk[9];
		rk[10] ^= rk[3];
		x2 ^= rk[10];
		rk[11] ^= rk[4];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[12] ^= rk[5];
		x0 ^= rk[12];
		rk[13] ^= rk[6];
		x1 ^= rk[13];
		rk[14] ^= rk[7];
		x2 ^= rk[14];
		rk[15] ^= rk[8];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[0] ^= x0;
		state[1] ^= x1;
		state[2] ^= x2;
		state[3] ^= x3;
		rk[16] ^= rk[9];
		x0 = state[12] ^ rk[16];
		rk[17] ^= rk[10];
		x1 = state[13] ^ rk[17];
		rk[18] ^= rk[11];
		x2 = state[14] ^ rk[18];
		rk[19] ^= rk[12];
		x3 = state[15] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[20] ^= rk[13];
		x0 ^= rk[20];
		rk[21] ^= rk[14];
		x1 ^= rk[21];
		rk[22] ^= rk[15];
		x2 ^= rk[22];
		rk[23] ^= rk[16];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[24] ^= rk[17];
		x0 ^= rk[24];
		rk[25] ^= rk[18];
		x1 ^= rk[25];
		rk[26] ^= rk[19];
		x2 ^= rk[26];
		rk[27] ^= rk[20];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[28] ^= rk[21];
		x0 ^= rk[28];
		rk[29] ^= rk[22];
		x1 ^= rk[29];
		rk[30] ^= rk[23];
		x2 ^= rk[30];
		rk[31] ^= rk[24];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[8] ^= x0;
		state[9] ^= x1;
		state[10] ^= x2;
		state[11] ^= x3;

		// 3
		KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
		rk[0] ^= rk[28];
		rk[1] ^= rk[29];
		rk[2] ^= rk[30];
		rk[3] ^= rk[31];
		x0 = state[0] ^ rk[0];
		x1 = state[1] ^ rk[1];
		x2 = state[2] ^ rk[2];
		x3 = state[3] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
		rk[4] ^= rk[0];
		rk[5] ^= rk[1];
		rk[6] ^= rk[2];
		rk[7] ^= rk[3];
		x0 ^= rk[4];
		x1 ^= rk[5];
		x2 ^= rk[6];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
		rk[8] ^= rk[4];
		rk[9] ^= rk[5];
		rk[10] ^= rk[6];
		rk[11] ^= rk[7];
		x0 ^= rk[8];
		x1 ^= rk[9];
		x2 ^= rk[10];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
		rk[12] ^= rk[8];
		rk[13] ^= rk[9];
		rk[14] ^= rk[10];
		rk[15] ^= rk[11];
		x0 ^= rk[12];
		x1 ^= rk[13];
		x2 ^= rk[14];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[12] ^= x0;
		state[13] ^= x1;
		state[14] ^= x2;
		state[15] ^= x3;
		KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
		rk[16] ^= rk[12];
		rk[17] ^= rk[13];
		rk[18] ^= rk[14];
		rk[19] ^= rk[15];
		x0 = state[8] ^ rk[16];
		x1 = state[9] ^ rk[17];
		x2 = state[10] ^ rk[18];
		x3 = state[11] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
		rk[20] ^= rk[16];
		rk[21] ^= rk[17];
		rk[22] ^= rk[18];
		rk[23] ^= rk[19];
		x0 ^= rk[20];
		x1 ^= rk[21];
		x2 ^= rk[22];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
		rk[24] ^= rk[20];
		rk[25] ^= rk[21];
		rk[26] ^= rk[22];
		rk[27] ^= rk[23];
		x0 ^= rk[24];
		x1 ^= rk[25];
		x2 ^= rk[26];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
		rk[28] ^= rk[24];
		rk[29] ^= rk[25];
		rk[30] ^= rk[26];
		rk[31] ^= ~rk[27];
		rk[30] ^= 512;
//		rk[31] ^= 0xFFFFFFFF;
		x0 ^= rk[28];
		x1 ^= rk[29];
		x2 ^= rk[30];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[4] ^= x0;
		state[5] ^= x1;
		state[6] ^= x2;
		state[7] ^= x3;

		rk[0] ^= rk[25];
		x0 = state[12] ^ rk[0];
		rk[1] ^= rk[26];
		x1 = state[13] ^ rk[1];
		rk[2] ^= rk[27];
		x2 = state[14] ^ rk[2];
		rk[3] ^= rk[28];
		x3 = state[15] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[4] ^= rk[29];
		x0 ^= rk[4];
		rk[5] ^= rk[30];
		x1 ^= rk[5];
		rk[6] ^= rk[31];
		x2 ^= rk[6];
		rk[7] ^= rk[0];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[8] ^= rk[1];
		x0 ^= rk[8];
		rk[9] ^= rk[2];
		x1 ^= rk[9];
		rk[10] ^= rk[3];
		x2 ^= rk[10];
		rk[11] ^= rk[4];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[12] ^= rk[5];
		x0 ^= rk[12];
		rk[13] ^= rk[6];
		x1 ^= rk[13];
		rk[14] ^= rk[7];
		x2 ^= rk[14];
		rk[15] ^= rk[8];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[8] ^= x0;
		state[9] ^= x1;
		state[10] ^= x2;
		state[11] ^= x3;
		rk[16] ^= rk[9];
		x0 = state[4] ^ rk[16];
		rk[17] ^= rk[10];
		x1 = state[5] ^ rk[17];
		rk[18] ^= rk[11];
		x2 = state[6] ^ rk[18];
		rk[19] ^= rk[12];
		x3 = state[7] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[20] ^= rk[13];
		x0 ^= rk[20];
		rk[21] ^= rk[14];
		x1 ^= rk[21];
		rk[22] ^= rk[15];
		x2 ^= rk[22];
		rk[23] ^= rk[16];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[24] ^= rk[17];
		x0 ^= rk[24];
		rk[25] ^= rk[18];
		x1 ^= rk[25];
		rk[26] ^= rk[19];
		x2 ^= rk[26];
		rk[27] ^= rk[20];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[28] ^= rk[21];
		x0 ^= rk[28];
		rk[29] ^= rk[22];
		x1 ^= rk[29];
		rk[30] ^= rk[23];
		x2 ^= rk[30];
		rk[31] ^= rk[24];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[0] ^= x0;
		state[1] ^= x1;
		state[2] ^= x2;
		state[3] ^= x3;

		/* round 3, 7, 11 */
		KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
		rk[0] ^= rk[28];
		rk[1] ^= rk[29];
		rk[2] ^= rk[30];
		rk[3] ^= rk[31];
		x0 = state[8] ^ rk[0];
		x1 = state[9] ^ rk[1];
		x2 = state[10] ^ rk[2];
		x3 = state[11] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
		rk[4] ^= rk[0];
		rk[5] ^= rk[1];
		rk[6] ^= rk[2];
		rk[7] ^= rk[3];
		x0 ^= rk[4];
		x1 ^= rk[5];
		x2 ^= rk[6];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
		rk[8] ^= rk[4];
		rk[9] ^= rk[5];
		rk[10] ^= rk[6];
		rk[11] ^= rk[7];
		x0 ^= rk[8];
		x1 ^= rk[9];
		x2 ^= rk[10];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
		rk[12] ^= rk[8];
		rk[13] ^= rk[9];
		rk[14] ^= rk[10];
		rk[15] ^= rk[11];
		x0 ^= rk[12];
		x1 ^= rk[13];
		x2 ^= rk[14];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[4] ^= x0;
		state[5] ^= x1;
		state[6] ^= x2;
		state[7] ^= x3;
		KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
		rk[16] ^= rk[12];
		rk[17] ^= rk[13];
		rk[18] ^= rk[14];
		rk[19] ^= rk[15];
		x0 = state[0] ^ rk[16];
		x1 = state[1] ^ rk[17];
		x2 = state[2] ^ rk[18];
		x3 = state[3] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
		rk[20] ^= rk[16];
		rk[21] ^= rk[17];
		rk[22] ^= rk[18];
		rk[23] ^= rk[19];
		x0 ^= rk[20];
		x1 ^= rk[21];
		x2 ^= rk[22];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
		rk[24] ^= rk[20];
		rk[25] ^= rk[21];
		rk[26] ^= rk[22];
		rk[27] ^= rk[23];
		x0 ^= rk[24];
		x1 ^= rk[25];
		x2 ^= rk[26];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
		rk[28] ^= rk[24];
		rk[29] ^= rk[25];
		rk[30] ^= rk[26];
		rk[31] ^= rk[27];
		x0 ^= rk[28];
		x1 ^= rk[29];
		x2 ^= rk[30];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[12] ^= x0;
		state[13] ^= x1;
		state[14] ^= x2;
		state[15] ^= x3;
		/* round 4, 8, 12 */
		rk[0] ^= rk[25];
		x0 = state[4] ^ rk[0];
		rk[1] ^= rk[26];
		x1 = state[5] ^ rk[1];
		rk[2] ^= rk[27];
		x2 = state[6] ^ rk[2];
		rk[3] ^= rk[28];
		x3 = state[7] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[4] ^= rk[29];
		x0 ^= rk[4];
		rk[5] ^= rk[30];
		x1 ^= rk[5];
		rk[6] ^= rk[31];
		x2 ^= rk[6];
		rk[7] ^= rk[0];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[8] ^= rk[1];
		x0 ^= rk[8];
		rk[9] ^= rk[2];
		x1 ^= rk[9];
		rk[10] ^= rk[3];
		x2 ^= rk[10];
		rk[11] ^= rk[4];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[12] ^= rk[5];
		x0 ^= rk[12];
		rk[13] ^= rk[6];
		x1 ^= rk[13];
		rk[14] ^= rk[7];
		x2 ^= rk[14];
		rk[15] ^= rk[8];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[0] ^= x0;
		state[1] ^= x1;
		state[2] ^= x2;
		state[3] ^= x3;
		rk[16] ^= rk[9];
		x0 = state[12] ^ rk[16];
		rk[17] ^= rk[10];
		x1 = state[13] ^ rk[17];
		rk[18] ^= rk[11];
		x2 = state[14] ^ rk[18];
		rk[19] ^= rk[12];
		x3 = state[15] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[20] ^= rk[13];
		x0 ^= rk[20];
		rk[21] ^= rk[14];
		x1 ^= rk[21];
		rk[22] ^= rk[15];
		x2 ^= rk[22];
		rk[23] ^= rk[16];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[24] ^= rk[17];
		x0 ^= rk[24];
		rk[25] ^= rk[18];
		x1 ^= rk[25];
		rk[26] ^= rk[19];
		x2 ^= rk[26];
		rk[27] ^= rk[20];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		rk[28] ^= rk[21];
		x0 ^= rk[28];
		rk[29] ^= rk[22];
		x1 ^= rk[29];
		rk[30] ^= rk[23];
		x2 ^= rk[30];
		rk[31] ^= rk[24];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[8] ^= x0;
		state[9] ^= x1;
		state[10] ^= x2;
		state[11] ^= x3;

		/* round 13 */
		KEY_EXPAND_ELT(sharedMemory, rk[0], rk[1], rk[2], rk[3]);
		rk[0] ^= rk[28];
		rk[1] ^= rk[29];
		rk[2] ^= rk[30];
		rk[3] ^= rk[31];
		x0 = state[0] ^ rk[0];
		x1 = state[1] ^ rk[1];
		x2 = state[2] ^ rk[2];
		x3 = state[3] ^ rk[3];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[4], rk[5], rk[6], rk[7]);
		rk[4] ^= rk[0];
		rk[5] ^= rk[1];
		rk[6] ^= rk[2];
		rk[7] ^= rk[3];
		x0 ^= rk[4];
		x1 ^= rk[5];
		x2 ^= rk[6];
		x3 ^= rk[7];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[8], rk[9], rk[10], rk[11]);
		rk[8] ^= rk[4];
		rk[9] ^= rk[5];
		rk[10] ^= rk[6];
		rk[11] ^= rk[7];
		x0 ^= rk[8];
		x1 ^= rk[9];
		x2 ^= rk[10];
		x3 ^= rk[11];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[12], rk[13], rk[14], rk[15]);
		rk[12] ^= rk[8];
		rk[13] ^= rk[9];
		rk[14] ^= rk[10];
		rk[15] ^= rk[11];
		x0 ^= rk[12];
		x1 ^= rk[13];
		x2 ^= rk[14];
		x3 ^= rk[15];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[12] ^= x0;
		state[13] ^= x1;
		state[14] ^= x2;
		state[15] ^= x3;
		KEY_EXPAND_ELT(sharedMemory, rk[16], rk[17], rk[18], rk[19]);
		rk[16] ^= rk[12];
		rk[17] ^= rk[13];
		rk[18] ^= rk[14];
		rk[19] ^= rk[15];
		x0 = state[8] ^ rk[16];
		x1 = state[9] ^ rk[17];
		x2 = state[10] ^ rk[18];
		x3 = state[11] ^ rk[19];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[20], rk[21], rk[22], rk[23]);
		rk[20] ^= rk[16];
		rk[21] ^= rk[17];
		rk[22] ^= rk[18];
		rk[23] ^= rk[19];
		x0 ^= rk[20];
		x1 ^= rk[21];
		x2 ^= rk[22];
		x3 ^= rk[23];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[24], rk[25], rk[26], rk[27]);
		rk[24] ^= rk[20];
		rk[25] ^= rk[21] ^ 512;
		rk[26] ^= rk[22];
		rk[27] ^= ~rk[23]; //^ 0xFFFFFFFF;
		x0 ^= rk[24];
		x1 ^= rk[25];
		x2 ^= rk[26];
		x3 ^= rk[27];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		KEY_EXPAND_ELT(sharedMemory, rk[28], rk[29], rk[30], rk[31]);
		rk[28] ^= rk[24];
		rk[29] ^= rk[25];
		rk[30] ^= rk[26];
		rk[31] ^= rk[27];
		x0 ^= rk[28];
		x1 ^= rk[29];
		x2 ^= rk[30];
		x3 ^= rk[31];
		AES_ROUND_NOKEY(sharedMemory, x0, x1, x2, x3);
		state[4] ^= x0;
		state[5] ^= x1;
		state[6] ^= x2;
		state[7] ^= x3;




		Hash[0] = cstate[0x0] ^ state[8];
		Hash[1] = cstate[0x1] ^ state[9];
		Hash[2] = cstate[0x2] ^ state[10];
		Hash[3] = cstate[0x3] ^ state[11];
		Hash[4] = cstate[0x4] ^ state[12];
		Hash[5] = cstate[0x5] ^ state[13];
		Hash[6] = cstate[0x6] ^ state[14];
		Hash[7] = cstate[0x7] ^ state[15];
		Hash[8] = cstate[0x8] ^ state[0];
		Hash[9] = cstate[0x9] ^ state[1];
		Hash[10] = cstate[0xA] ^ state[2];
		Hash[11] = cstate[0xB] ^ state[3];
		Hash[12] = cstate[0xC] ^ state[4];
		Hash[13] = cstate[0xD] ^ state[5];
		Hash[14] = cstate[0xE] ^ state[6];
		Hash[15] = cstate[0xF] ^ state[7];
	}
}


__global__ __launch_bounds__(TPB, 2)
>>>>>>> 8c320ca... added xevan
void x11_shavite512_gpu_hash_80(uint32_t threads, uint32_t startNounce, void *outputHash)
{
	__shared__ uint32_t sharedMemory[1024];

<<<<<<< HEAD
	shavite_gpu_init(sharedMemory);
	__threadfence_block();
=======
	if (threadIdx.x < 256) 
	{
		sharedMemory[threadIdx.x] = d_AES0[threadIdx.x];
		sharedMemory[threadIdx.x + 256] = ROTL32(sharedMemory[threadIdx.x], 8);
		sharedMemory[threadIdx.x + 512] = ROTL32(sharedMemory[threadIdx.x], 16);
		sharedMemory[threadIdx.x + 768] = ROTL32(sharedMemory[threadIdx.x], 24);
	}
>>>>>>> 8c320ca... added xevan

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = startNounce + thread;

<<<<<<< HEAD
		// initial state
		uint32_t state[16] = {
			SPH_C32(0x72FCCDD8), SPH_C32(0x79CA4727), SPH_C32(0x128A077B), SPH_C32(0x40D55AEC),
			SPH_C32(0xD1901A06), SPH_C32(0x430AE307), SPH_C32(0xB29F5CD1), SPH_C32(0xDF07FBFC),
			SPH_C32(0x8E45D73D), SPH_C32(0x681AB538), SPH_C32(0xBDE86578), SPH_C32(0xDD577E47),
			SPH_C32(0xE275EADE), SPH_C32(0x502D9FCD), SPH_C32(0xB9357178), SPH_C32(0x022A4B9A)
=======
		// kopiere init-state
		uint32_t state[16] = {
			0x72FCCDD8, 0x79CA4727, 0x128A077B, 0x40D55AEC,
			0xD1901A06, 0x430AE307, 0xB29F5CD1, 0xDF07FBFC,
			0x8E45D73D, 0x681AB538, 0xBDE86578, 0xDD577E47,
			0xE275EADE, 0x502D9FCD, 0xB9357178, 0x022A4B9A
>>>>>>> 8c320ca... added xevan
		};

		uint32_t msg[32];

		#pragma unroll 32
		for(int i=0;i<32;i++) {
			msg[i] = c_PaddedMessage80[i];
		}
		msg[19] = cuda_swab32(nounce);
		msg[20] = 0x80;
		msg[27] = 0x2800000;
		msg[31] = 0x2000000;

<<<<<<< HEAD
		c512(sharedMemory, state, msg, 640);
=======
		c512(sharedMemory, state, msg);
>>>>>>> 8c320ca... added xevan

		uint32_t *outHash = (uint32_t *)outputHash + 16 * thread;

		#pragma unroll 16
		for(int i=0;i<16;i++)
			outHash[i] = state[i];

	} //thread < threads
}

<<<<<<< HEAD
__host__
void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const uint32_t threadsperblock = TPB;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// note: 128 threads minimum are required to init the shared memory array
	x11_shavite512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
	//MyStreamSynchronize(NULL, order, thr_id);
}

__host__
void x11_shavite512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash, int order)
{
	const uint32_t threadsperblock = TPB;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
=======
__host__ void x11_shavite512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t shavitethreads)
{
	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + shavitethreads - 1) / shavitethreads);
	dim3 block(shavitethreads);

	x11_shavite512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash);
}

__host__ void x11_shavite512_cpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash)
{

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + TPB - 1) / TPB);
	dim3 block(TPB);
>>>>>>> 8c320ca... added xevan

	x11_shavite512_gpu_hash_80<<<grid, block>>>(threads, startNounce, d_outputHash);
}

<<<<<<< HEAD
__host__
void x11_shavite512_cpu_init(int thr_id, uint32_t threads)
{
	aes_cpu_init(thr_id);
}

__host__
void x11_shavite512_setBlock_80(void *pdata)
{
	// Message with Padding
	// The nonce is at Byte 76.
=======
__host__ void x11_shavite512_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
>>>>>>> 8c320ca... added xevan
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);

	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 32*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}
<<<<<<< HEAD
=======

>>>>>>> 8c320ca... added xevan
