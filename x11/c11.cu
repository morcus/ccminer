#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"
<<<<<<< HEAD
#include "cuda_helper.h"
#include "cuda_x11.h"

void tribus_echo512_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t *d_resNonce, const uint64_t target);
=======
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
#include "cuda_helper.h"
>>>>>>> 8c320ca... added xevan

#include <stdio.h>
#include <memory.h>

<<<<<<< HEAD
static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];

// Flax/Chaincoin C11 CPU Hash
extern "C" void c11hash(void *output, const void *input)
{
	unsigned char hash[128] = { 0 };

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;

	sph_blake512_init(&ctx_blake);
	sph_blake512 (&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, (void*) hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*) hash);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*) hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512 (&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, (void*) hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512 (&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, (const void*) hash, 64);
	sph_luffa512_close (&ctx_luffa, (void*) hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*) hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512 (&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512 (&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(output, hash, 32);
}

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 8*sizeof(uint32_t)); \
		hipMemcpy(debugbuf, d_hash[thr_id], 8*sizeof(uint32_t), hipMemcpyDeviceToHost); \
		printf("X11 %s %08x %08x %08x %08x...\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

extern "C" int scanhash_c11(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 20 : 19;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 19=256*256*8;
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x5;

	if (!init[thr_id])
	{
		int dev_id = device_map[thr_id];
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);
		use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		x11_luffaCubehash512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		if (use_compat_kernels[thr_id])
			x11_echo512_cpu_init(thr_id, throughput);
		if (x11_simd512_cpu_init(thr_id, throughput) != 0) {
			return 0;
		}
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 64 * throughput), 0);
		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], 2 * sizeof(uint32_t)));

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	if (use_compat_kernels[thr_id])
		cuda_check_cpu_setTarget(ptarget);
	else
		hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));

	do {
		int order = 0;

		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		TRACE("blake  :");
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("bmw    :");
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("groestl:");
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("jh512  :");
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("keccak :");
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("skein  :");
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id], order++);
		TRACE("luffa+c:");
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("shavite:");
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("simd   :");

		if (use_compat_kernels[thr_id]) {
			x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
			work->nonces[1] = UINT32_MAX;
		} else {
			tribus_echo512_final(thr_id, throughput, d_hash[thr_id], d_resNonce[thr_id], AS_U64(&ptarget[6]));
			hipMemcpy(&work->nonces[0], d_resNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNounce = pdata[19];
			if (!use_compat_kernels[thr_id]) work->nonces[0] += startNounce;
			be32enc(&endiandata[19], work->nonces[0]);
			c11hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != UINT32_MAX) {
					work->nonces[1] += startNounce;
					be32enc(&endiandata[19], work->nonces[1]);
					c11hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				hipMemset(d_resNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);
=======

static uint32_t *d_hash[MAX_GPUS];
static uint32_t foundnonces[MAX_GPUS][2];


extern void quark_blake512_cpu_init(int thr_id);
extern void quark_blake512_cpu_setBlock_80(uint64_t *pdata);
extern void quark_blake512_cpu_setBlock_80_multi(uint32_t thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_bmw512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_groestl512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_skein512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void cuda_jh512Keccak512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_luffaCubehash512_cpu_init(int thr_id, uint32_t threads);
extern void x11_luffaCubehash512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t luffacubehashthreads);

extern void x11_shavite512_cpu_init(int thr_id, uint32_t threads);
extern void x11_shavite512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t shavitethreads);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const uint32_t simdthreads);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void x11_echo512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t target, uint32_t *h_found);
extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes,
	uint32_t *d_noncesTrue, uint32_t *nrmTrue, uint32_t *d_noncesFalse, uint32_t *nrmFalse);

extern "C" void c11hash(void *output, const void *input)
{
			// blake1-bmw2-grs3-skein4-jh5-keccak6-luffa7-cubehash8-shavite9-simd10-echo11
		sph_blake512_context ctx_blake;
		sph_bmw512_context ctx_bmw;
		sph_groestl512_context ctx_groestl;
		sph_jh512_context ctx_jh;
		sph_keccak512_context ctx_keccak;
		sph_skein512_context ctx_skein;
		sph_luffa512_context ctx_luffa;
		sph_cubehash512_context ctx_cubehash;
		sph_shavite512_context ctx_shavite;
		sph_simd512_context ctx_simd;
		sph_echo512_context ctx_echo;
		
		unsigned char hash[128];
		memset(hash, 0, sizeof hash);
		
		sph_blake512_init(&ctx_blake);
		sph_blake512(&ctx_blake, input, 80);
		sph_blake512_close(&ctx_blake, (void*)hash);
		
		sph_bmw512_init(&ctx_bmw);
		sph_bmw512(&ctx_bmw, (const void*)hash, 64);
		sph_bmw512_close(&ctx_bmw, (void*)hash);
		
		sph_groestl512_init(&ctx_groestl);
		sph_groestl512(&ctx_groestl, (const void*)hash, 64);
		sph_groestl512_close(&ctx_groestl, (void*)hash);
		
		sph_jh512_init(&ctx_jh);
		sph_jh512(&ctx_jh, (const void*)hash, 64);
		sph_jh512_close(&ctx_jh, (void*)hash);
		
		sph_keccak512_init(&ctx_keccak);
		sph_keccak512(&ctx_keccak, (const void*)hash, 64);
		sph_keccak512_close(&ctx_keccak, (void*)hash);
		
		sph_skein512_init(&ctx_skein);
		sph_skein512(&ctx_skein, (const void*)hash, 64);
		sph_skein512_close(&ctx_skein, (void*)hash);
		
		sph_luffa512_init(&ctx_luffa);
		sph_luffa512(&ctx_luffa, (const void*)hash, 64);
		sph_luffa512_close(&ctx_luffa, (void*)hash);
		
		sph_cubehash512_init(&ctx_cubehash);
		sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
		sph_cubehash512_close(&ctx_cubehash, (void*)hash);
		
		sph_shavite512_init(&ctx_shavite);
		sph_shavite512(&ctx_shavite, (const void*)hash, 64);
		sph_shavite512_close(&ctx_shavite, (void*)hash);
		
		sph_simd512_init(&ctx_simd);
		sph_simd512(&ctx_simd, (const void*)hash, 64);
		sph_simd512_close(&ctx_simd, (void*)hash);
		
		sph_echo512_init(&ctx_echo);
		sph_echo512(&ctx_echo, (const void*)hash, 64);
		sph_echo512_close(&ctx_echo, (void*)hash);
		
		memcpy(output, hash, 32);
}
static bool init[MAX_GPUS] = { 0 };
static uint32_t endiandata[MAX_GPUS][20];

extern "C" int scanhash_c11(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	int intensity = (device_sm[device_map[thr_id]] > 500) ? 256 * 256 * 21 : 256 * 256 * 10;
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 64 : 32;
	uint32_t shavitethreads = (device_sm[device_map[thr_id]] == 500) ? 256 : 320;
	uint32_t luffacubehashthreads = (device_sm[device_map[thr_id]] == 500) ? 512 : 256;

	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x4f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}

		x11_echo512_cpu_init(thr_id, throughput);
		if (x11_simd512_cpu_init(thr_id, throughput) != 0) {
			return 0;
		}
		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 64 * throughput), 0); // why 64 ?
		quark_blake512_cpu_init(thr_id);
		init[thr_id] = true;
	}
	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[thr_id][k], ((uint32_t*)pdata)[k]);


	if (opt_n_gputhreads > 1)
	{
		quark_blake512_cpu_setBlock_80_multi(thr_id, (uint64_t *)endiandata[thr_id]);
	}
	else
	{
		quark_blake512_cpu_setBlock_80((uint64_t *)endiandata[thr_id]);
	}

	do {

		quark_blake512_cpu_hash_80(throughput, pdata[19], d_hash[thr_id]);
		quark_bmw512_cpu_hash_64(throughput, pdata[19], NULL, d_hash[thr_id]);
		quark_groestl512_cpu_hash_64(throughput, pdata[19], NULL, d_hash[thr_id]);
		cuda_jh512Keccak512_cpu_hash_64(throughput, pdata[19], d_hash[thr_id]);
		quark_skein512_cpu_hash_64(throughput, pdata[19], NULL, d_hash[thr_id]);
		x11_luffaCubehash512_cpu_hash_64(throughput, pdata[19], d_hash[thr_id], luffacubehashthreads);
		x11_shavite512_cpu_hash_64(throughput, pdata[19], d_hash[thr_id], shavitethreads);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], simdthreads);
		x11_echo512_cpu_hash_64_final(thr_id, throughput, pdata[19], d_hash[thr_id], ptarget[7], foundnonces[thr_id]);
		if (foundnonces[thr_id][0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[thr_id][19], foundnonces[thr_id][0]);
			c11hash(vhash64, endiandata[thr_id]);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundnonces[thr_id][1] != 0xffffffff)
				{
					pdata[21] = foundnonces[thr_id][1];
					res++;
					if (opt_benchmark)
						applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, foundnonces[thr_id][1], vhash64[7], Htarg);
				}
				pdata[19] = foundnonces[thr_id][0];
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", thr_id, foundnonces[thr_id][0], vhash64[7], Htarg);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundnonces[thr_id][0]);
				}
			}
		}
		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
>>>>>>> 8c320ca... added xevan

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
<<<<<<< HEAD

// cleanup
extern "C" void free_c11(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_resNonce[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
=======
>>>>>>> 8c320ca... added xevan
