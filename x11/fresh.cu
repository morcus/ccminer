#include "hip/hip_runtime.h"
/**
 * Fresh algorithm
 */
extern "C" {
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}
#include "miner.h"
#include "cuda_helper.h"

// to test gpu hash on a null buffer
#define NULLTEST 0

static uint32_t *d_hash[MAX_GPUS];
<<<<<<< HEAD

extern void x11_shavite512_cpu_init(int thr_id, uint32_t threads);
extern void x11_shavite512_setBlock_80(void *pdata);
extern void x11_shavite512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void x11_simd512_cpu_free(int thr_id);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

=======
static uint32_t *h_found[MAX_GPUS];

extern void x11_shavite512_setBlock_80(void *pdata);
extern void x11_shavite512_cpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void x11_shavite512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t shavitethreads);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const uint32_t simdthreads);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
//extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void x11_echo512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t target, uint32_t *h_found);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes,
											uint32_t *d_noncesTrue, uint32_t *nrmTrue, uint32_t *d_noncesFalse, uint32_t *nrmFalse);
>>>>>>> 8c320ca... added xevan

// CPU Hash
extern "C" void fresh_hash(void *state, const void *input)
{
	// shavite-simd-shavite-simd-echo

	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashA hash
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, input, 80);
	sph_shavite512_close(&ctx_shavite, hashA);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hashA, 64);
	sph_simd512_close(&ctx_simd, hashB);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, hashB, 64);
	sph_shavite512_close(&ctx_shavite, hashA);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hashA, 64);
	sph_simd512_close(&ctx_simd, hashB);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hashB, 64);
	sph_echo512_close(&ctx_echo, hashA);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

<<<<<<< HEAD
extern "C" int scanhash_fresh(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];

	uint32_t throughput =  cuda_default_throughput(thr_id, 1 << 19);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x00ff;
=======
extern "C" int scanhash_fresh(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];

	uint32_t throughput = device_intensity(device_map[thr_id], __func__, 1 << 19);
	throughput = min(throughput, (max_nonce - first_nonce));
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 64 : 32;
	uint32_t shavitethreads = (device_sm[device_map[thr_id]] == 500) ? 256 : 320;

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0xf;
>>>>>>> 8c320ca... added xevan

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
<<<<<<< HEAD
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t)64 * throughput + 4), -1);

		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);

=======
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}

		x11_simd512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput + 4), 0);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&(h_found[thr_id]), 4 * sizeof(uint32_t)), 0);

>>>>>>> 8c320ca... added xevan
		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
<<<<<<< HEAD
		be32enc(&endiandata[k], pdata[k]);
	
	x11_shavite512_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);
	do {
		int order = 0;

		// GPU Hash
		x11_shavite512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

#if NULLTEST
		uint32_t buf[8]; memset(buf, 0, sizeof buf);
		CUDA_SAFE_CALL(hipMemcpy(buf, d_hash[thr_id], sizeof buf, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		print_hash((unsigned char*)buf); printf("\n");
#endif
		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			fresh_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					fresh_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}

// cleanup
extern "C" void free_fresh(int thr_id)
{
	if (!init[thr_id])
		return;

	hipSetDevice(device_map[thr_id]);

	hipFree(d_hash[thr_id]);
	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
=======
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	
	x11_shavite512_setBlock_80((void*)endiandata);

	do {
		uint32_t Htarg = ptarget[7];

	//	uint32_t foundNonce;

		// GPU Hash

		x11_shavite512_cpu_hash_80(throughput, pdata[19], d_hash[thr_id]);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], simdthreads);
		x11_shavite512_cpu_hash_64(throughput, pdata[19], d_hash[thr_id], shavitethreads);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], simdthreads);
		x11_echo512_cpu_hash_64_final(thr_id, throughput, pdata[19], d_hash[thr_id], ptarget[7], h_found[thr_id]);

	//	hipDeviceSynchronize();
		if (h_found[thr_id][0] != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], h_found[thr_id][0]);
			fresh_hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", thr_id, h_found[thr_id][0], vhash64[7], Htarg);

				if (h_found[thr_id][1] != 0xffffffff)
				{
					pdata[21] =h_found[thr_id][1];
					res++;
				}
				pdata[19] = h_found[thr_id][0];
				return res;
			}
			else
			{
				if(vhash64[7] != Htarg) applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, h_found[thr_id][0]);
			}
		}
		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
>>>>>>> 8c320ca... added xevan
