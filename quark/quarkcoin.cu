#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"
<<<<<<< HEAD
#include "cuda_quark.h"

#include <stdio.h>

extern uint32_t quark_filter_cpu_sm2(const int thr_id, const uint32_t threads, const uint32_t *inpHashes, uint32_t* d_branch2);
extern void quark_merge_cpu_sm2(const int thr_id, const uint32_t threads, uint32_t *outpHashes, uint32_t* d_branch2);

static uint32_t *d_hash[MAX_GPUS];
static uint32_t* d_hash_br2[MAX_GPUS];  // SM 2

// Speicher zur Generierung der Noncevektoren für die bedingten Hashes
static uint32_t *d_branch1Nonces[MAX_GPUS];
static uint32_t *d_branch2Nonces[MAX_GPUS];
static uint32_t *d_branch3Nonces[MAX_GPUS];

// Original Quarkhash Funktion aus einem miner Quelltext
extern "C" void quarkhash(void *state, const void *input)
{
	unsigned char _ALIGN(128) hash[64];

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;

	sph_blake512_init(&ctx_blake);
	sph_blake512 (&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, (void*) hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*) hash);

	if (hash[0] & 0x8)
	{
		sph_groestl512_init(&ctx_groestl);
		sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
		sph_groestl512_close(&ctx_groestl, (void*) hash);
	}
	else
	{
		sph_skein512_init(&ctx_skein);
		sph_skein512 (&ctx_skein, (const void*) hash, 64);
		sph_skein512_close(&ctx_skein, (void*) hash);
	}

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*) hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512 (&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, (void*) hash);

	if (hash[0] & 0x8)
	{
		sph_blake512_init(&ctx_blake);
		sph_blake512 (&ctx_blake, (const void*) hash, 64);
		sph_blake512_close(&ctx_blake, (void*) hash);
	}
	else
	{
		sph_bmw512_init(&ctx_bmw);
		sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
		sph_bmw512_close(&ctx_bmw, (void*) hash);
	}

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512 (&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	if (hash[0] & 0x8)
	{
		sph_keccak512_init(&ctx_keccak);
		sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
		sph_keccak512_close(&ctx_keccak, (void*) hash);
	}
	else
	{
		sph_jh512_init(&ctx_jh);
		sph_jh512 (&ctx_jh, (const void*) hash, 64);
		sph_jh512_close(&ctx_jh, (void*) hash);
	}

	memcpy(state, hash, 32);
}

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 32); \
		hipMemcpy(debugbuf, d_hash[thr_id], 32, hipMemcpyDeviceToHost); \
		printf("quark %s %08x %08x %08x %08x...%08x... \n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3]), swab32(debugbuf[7])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_quark(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	int dev_id = device_map[thr_id];
	uint32_t def_thr = 1U << 20; // 256*4096
	uint32_t throughput = cuda_default_throughput(thr_id, def_thr);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x00F;

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		hipGetLastError();
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));

		quark_blake512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_compactTest_cpu_init(thr_id, throughput);

		if (cuda_arch[dev_id] >= 300) {
			hipMalloc(&d_branch1Nonces[thr_id], sizeof(uint32_t)*throughput);
			hipMalloc(&d_branch2Nonces[thr_id], sizeof(uint32_t)*throughput);
			hipMalloc(&d_branch3Nonces[thr_id], sizeof(uint32_t)*throughput);
		} else {
			hipMalloc(&d_hash_br2[thr_id], (size_t) 64 * throughput);
		}

		cuda_check_cpu_init(thr_id, throughput);
		CUDA_SAFE_CALL(hipGetLastError());

=======

static uint32_t *d_hash[MAX_GPUS];

// Speicher zur Generierung der Noncevektoren für die bedingten Hashes
uint32_t *d_branch1Nonces[MAX_GPUS];
uint32_t *d_branch2Nonces[MAX_GPUS];
uint32_t *d_branch3Nonces[MAX_GPUS];


extern void quark_blake512_cpu_init(int thr_id);
extern void quark_blake512_cpu_setBlock_80(uint64_t *pdata);
extern void quark_blake512_cpu_setBlock_80_multi(uint32_t thr_id, uint64_t *pdata);

extern void quark_blake512_cpu_hash_80(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void quark_blake512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_bmw512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_bmw512_cpu_hash_64_quark(uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_groestl512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_keccakskein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_keccak512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, uint32_t target, uint32_t *h_found);
extern void quark_keccak512_cpu_init(int thr_id);


extern void quark_jh512_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_jh512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, uint32_t target, uint32_t *h_found);
extern void quark_jh512_cpu_init(int thr_id);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
											uint32_t *d_nonces1, uint32_t *nrm1,
											uint32_t *d_nonces2, uint32_t *nrm2);
extern void quark_compactTest_single_false_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
	uint32_t *d_nonces1, uint32_t *nrm1);

extern uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash);
extern void cuda_check_quarkcoin(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, uint32_t *foundnonces);

// Original Quarkhash Funktion aus einem miner Quelltext
extern "C" void quarkhash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_bmw512_context ctx_bmw;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    
    unsigned char hash[64];

    sph_blake512_init(&ctx_blake);
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close(&ctx_blake, (void*) hash);
    
    sph_bmw512_init(&ctx_bmw);
    sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
    sph_bmw512_close(&ctx_bmw, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_groestl512_init(&ctx_groestl);
        sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
        sph_groestl512_close(&ctx_groestl, (void*) hash);
    }
    else
    {
        sph_skein512_init(&ctx_skein);
        sph_skein512 (&ctx_skein, (const void*) hash, 64);
        sph_skein512_close(&ctx_skein, (void*) hash);
    }
    
    sph_groestl512_init(&ctx_groestl);
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_blake512_init(&ctx_blake);
        sph_blake512 (&ctx_blake, (const void*) hash, 64);
        sph_blake512_close(&ctx_blake, (void*) hash);
    }
    else
    {
        sph_bmw512_init(&ctx_bmw);
        sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
        sph_bmw512_close(&ctx_bmw, (void*) hash);
    }

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_keccak512_init(&ctx_keccak);
        sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
        sph_keccak512_close(&ctx_keccak, (void*) hash);
    }
    else
    {
        sph_jh512_init(&ctx_jh);
        sph_jh512 (&ctx_jh, (const void*) hash, 64);
        sph_jh512_close(&ctx_jh, (void*) hash);
    }

    memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };
static uint32_t endiandata[MAX_GPUS][20];
static uint32_t foundnonces[MAX_GPUS][2];
static uint32_t foundnonces2[MAX_GPUS][2];

extern "C" int scanhash_quark(int thr_id, uint32_t *pdata,
    uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	uint32_t intensity = 256*256*57;
	intensity = intensity + ((1 << 22));
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);

	if (device_sm[device_map[thr_id]] > 500) intensity= 1 << 24;

	if (strstr(props.name, "980 Ti"))
	{
		intensity = 1 << 25;
	} else
	if (strstr(props.name, "980"))
	{
		intensity = 1 << 25;
	}

	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] =0x2f;

	if (!init[thr_id])
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}

		// Konstanten kopieren, Speicher belegen
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		quark_compactTest_cpu_init(thr_id, throughput);

		uint32_t noncebuffersize = throughput * 7 / 10;
		uint32_t noncebuffersize2 = (throughput * 7 / 10)*7/10;

		hipMalloc(&d_branch1Nonces[thr_id], sizeof(uint32_t)*noncebuffersize2);
		hipMalloc(&d_branch2Nonces[thr_id], sizeof(uint32_t)*noncebuffersize2);
		hipMalloc(&d_branch3Nonces[thr_id], sizeof(uint32_t)*noncebuffersize);
		quark_blake512_cpu_init(thr_id);
		quark_keccak512_cpu_init(thr_id);
		quark_jh512_cpu_init(thr_id);
		CUDA_SAFE_CALL(hipGetLastError());
>>>>>>> 8c320ca... added xevan
		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
<<<<<<< HEAD
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t nrm1=0, nrm2=0, nrm3=0;

		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		TRACE("blake  :");
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("bmw    :");

		if (cuda_arch[dev_id] >= 300) {

			quark_compactTest_single_false_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], NULL,
				d_branch3Nonces[thr_id], &nrm3, order++);

			// nur den Skein Branch weiterverfolgen
			quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der unbedingte Branch für Groestl512
			quark_groestl512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der unbedingte Branch für JH512
			quark_jh512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
			quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
				d_branch1Nonces[thr_id], &nrm1,
				d_branch2Nonces[thr_id], &nrm2,
				order++);

			// das ist der bedingte Branch für Blake512
			quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der bedingte Branch für Bmw512
			quark_bmw512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der unbedingte Branch für Keccak512
			quark_keccak512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// das ist der unbedingte Branch für Skein512
			quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);

			// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
			quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
				d_branch1Nonces[thr_id], &nrm1,
				d_branch2Nonces[thr_id], &nrm2,
				order++);

			quark_keccak512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id], order++);

			work->nonces[0] = cuda_check_hash_branch(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], order++);
			work->nonces[1] = 0;
		} else {
			/* algo permutations are made with 2 different buffers */

			quark_filter_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			quark_merge_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			TRACE("perm1  :");

			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("groestl:");
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("jh512  :");

			quark_filter_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			quark_merge_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			TRACE("perm2  :");

			quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("keccak :");
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("skein  :");

			quark_filter_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			quark_merge_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			TRACE("perm3  :");

			CUDA_LOG_ERROR();
			work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
			work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			quarkhash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					quarkhash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	return 0;
}

// cleanup
extern "C" void free_quark(int thr_id)
{
	int dev_id = device_map[thr_id];
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	if (cuda_arch[dev_id] >= 300) {
		hipFree(d_branch1Nonces[thr_id]);
		hipFree(d_branch2Nonces[thr_id]);
		hipFree(d_branch3Nonces[thr_id]);
	} else {
		hipFree(d_hash_br2[thr_id]);
	}

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	quark_compactTest_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
=======
		be32enc(&endiandata[thr_id][k], ((uint32_t*)pdata)[k]);
	cuda_check_cpu_setTarget(ptarget);
	if (opt_n_gputhreads > 1)
	{
		quark_blake512_cpu_setBlock_80_multi(thr_id, (uint64_t *)endiandata[thr_id]);
	}
	else
	{
		quark_blake512_cpu_setBlock_80((uint64_t *)endiandata[thr_id]);
	}

	do {

		uint32_t nrm1 = 0, nrm2 = 0, nrm3 = 0;

		quark_blake512_cpu_hash_80( throughput, pdata[19], d_hash[thr_id]);
		quark_bmw512_cpu_hash_64_quark(throughput, pdata[19],d_hash[thr_id]);

		quark_compactTest_single_false_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], NULL,
			d_branch3Nonces[thr_id], &nrm3);

		// nur den Skein Branch weiterverfolgen
		quark_skein512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// das ist der unbedingte Branch für Groestl512
		quark_groestl512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// das ist der unbedingte Branch für JH512
		quark_jh512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
		quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2);

		// das ist der bedingte Branch für Blake512
		quark_blake512_cpu_hash_64(nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id]);

		// das ist der bedingte Branch für Bmw512
		quark_bmw512_cpu_hash_64(nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id]);

		quark_keccakskein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
		quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch3Nonces[thr_id], &nrm2);
		
		quark_keccak512_cpu_hash_64_final(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id], ptarget[7], &foundnonces2[thr_id][0]);
		quark_jh512_cpu_hash_64_final(thr_id, nrm2, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], ptarget[7], &foundnonces[thr_id][0]);

		if (foundnonces[thr_id][0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[thr_id][19], foundnonces[thr_id][0]);
			quarkhash(vhash64, endiandata[thr_id]);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				// check if there was some other ones...
				if (foundnonces2[thr_id][0] != 0xffffffff)
				{
					const uint32_t Htarg = ptarget[7];
					uint32_t vhash64[8];
					be32enc(&endiandata[thr_id][19], foundnonces2[thr_id][0]);
					quarkhash(vhash64, endiandata[thr_id]);

					if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = foundnonces2[thr_id][0];
						res++;
						if (opt_benchmark) applog(LOG_INFO, "GPU #%d: Found second nonce $%08X", thr_id, foundnonces2[thr_id][0]);
					}
					else
					{
						if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
							applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundnonces2[thr_id][0]);
					}
				} else	if (foundnonces[thr_id][1] != 0xffffffff)
				{
					pdata[21] = foundnonces[thr_id][1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d: Found second nonce $%08X", thr_id, foundnonces[thr_id][1]);
				}				
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d: Found nonce $%08X", thr_id, foundnonces[thr_id][0]);
				pdata[19] = foundnonces[thr_id][0];
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundnonces[thr_id][0]);
			}
		}

		if (foundnonces2[thr_id][0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[thr_id][19], foundnonces2[thr_id][0]);
			quarkhash(vhash64, endiandata[thr_id]);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				// check if there was some other ones...
				if (foundnonces2[thr_id][1] != 0xffffffff)
				{
					pdata[21] = foundnonces2[thr_id][1];
					res++;
					if (opt_benchmark)  applog(LOG_INFO, "GPU #%d: Found second nonce $%08X", thr_id, foundnonces2[thr_id][1]);
				}
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d: Found nonce $%08X", thr_id, foundnonces2[thr_id][0]);
				pdata[19] = foundnonces2[thr_id][0];
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundnonces2[thr_id][0]);
			}
		}


		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce;
	return 0;
>>>>>>> 8c320ca... added xevan
}
