#include "hip/hip_runtime.h"
/*
<<<<<<< HEAD
 * whirlpool routine
 */
extern "C" {
#include <sph/sph_whirlpool.h>
#include <miner.h>
}

#include <cuda_helper.h>

//#define SM3_VARIANT

#ifdef SM3_VARIANT
static uint32_t *d_hash[MAX_GPUS];
extern void whirlpool512_init_sm3(int thr_id, uint32_t threads, int mode);
extern void whirlpool512_free_sm3(int thr_id);
extern void whirlpool512_setBlock_80_sm3(void *pdata, const void *ptarget);
extern void whirlpool512_hash_64_sm3(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void whirlpool512_hash_80_sm3(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern uint32_t whirlpool512_finalhash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
//#define _DEBUG
#define _DEBUG_PREFIX "whirl"
#include <cuda_debug.cuh>
#else
extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode);
extern void x15_whirlpool_cpu_free(int thr_id);
extern void whirlpool512_setBlock_80(void *pdata, const void *ptarget);
extern void whirlpool512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *resNonces, const uint64_t target);
#endif
=======
 * whirlpool routine djm&SP
 */
extern "C"
{
#include "sph/sph_whirlpool.h"
#include "miner.h"
}


#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce,  uint32_t *d_hash);

extern void whirlpool512_setBlock_80(void *pdata, const void *ptarget);
extern void whirlpool512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
//extern uint32_t* whirlpool512_cpu_finalhash_64(int thr_id, uint32_t threads, uint32_t startNounce,  uint32_t *d_hash);
extern void whirlpool512_cpu_finalhash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t *foundnonce);
>>>>>>> 8c320ca... added xevan


// CPU Hash function
extern "C" void wcoinhash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

<<<<<<< HEAD
=======
	// shavite 1
>>>>>>> 8c320ca... added xevan
	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, input, 80);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hashB);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hashB, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	memcpy(state, hash, 32);
}

<<<<<<< HEAD
void whirl_midstate(void *state, const void *input)
{
	sph_whirlpool_context ctx;

	sph_whirlpool1_init(&ctx);
	sph_whirlpool1(&ctx, input, 64);

	memcpy(state, ctx.state, 64);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whirl(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(128) endiandata[20];
	uint32_t* pdata = work->data;
	uint32_t* ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 19); // 19=256*256*8;
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);
	if (init[thr_id]) throughput = max(throughput, 256); // shared mem requirement

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000ff;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
#ifdef SM3_VARIANT
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput));
		whirlpool512_init_sm3(thr_id, throughput, 1 /* old whirlpool */);
#else
		x15_whirlpool_cpu_init(thr_id, throughput, 1 /* old whirlpool */);
#endif
=======
static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_whc(int thr_id, uint32_t *pdata,
    uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughput = device_intensity(thr_id, __func__, 1U << 20); // 19=256*256*8;
	throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id]) {
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));
		x15_whirlpool_cpu_init(thr_id, throughput, 1 /* old whirlpool */);

>>>>>>> 8c320ca... added xevan
		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
<<<<<<< HEAD
		be32enc(&endiandata[k], pdata[k]);
	}

#ifdef SM3_VARIANT
	whirlpool512_setBlock_80_sm3((void*)endiandata, ptarget);
#else
	whirlpool512_setBlock_80((void*)endiandata, ptarget);
#endif

	do {
#ifdef SM3_VARIANT
		int order = 1;
		whirlpool512_hash_80_sm3(thr_id, throughput, pdata[19], d_hash[thr_id]);
		TRACE64(" 80 :", d_hash);
		whirlpool512_hash_64_sm3(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE64(" 64 :", d_hash);
		whirlpool512_hash_64_sm3(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE64(" 64 :", d_hash);
		work->nonces[0] = whirlpool512_finalhash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
#else
		whirlpool512_cpu_hash_80(thr_id, throughput, pdata[19], work->nonces, *(uint64_t*)&ptarget[6]);
#endif
		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != UINT32_MAX && bench_algo < 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			wcoinhash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				pdata[19] = work->nonces[0] + 1; // cursor
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}
		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);
=======
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	whirlpool512_setBlock_80((void*)endiandata, ptarget);

	do {
		uint32_t foundNonce[2] = { 0, 0 };;

		whirlpool512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);

		whirlpool512_cpu_finalhash_64(thr_id, throughput, pdata[19], d_hash[thr_id], &foundNonce[0]);
		if (foundNonce[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce[0]);
			wcoinhash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != UINT32_MAX)
				{
					be32enc(&endiandata[19], foundNonce[1]);
					wcoinhash(vhash64, endiandata);
					if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						if (opt_benchmark) applog(LOG_INFO, "GPU #%d: found second nounce %08x", thr_id, foundNonce[1]);
						pdata[21] = foundNonce[1];
						res++;
					}
					else
					{
						if (vhash64[7] != Htarg)
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce[1]);
					}
				}
				
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d: found nounce %08x", thr_id, foundNonce[0]);

				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce[0]);
			}
		}
		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
>>>>>>> 8c320ca... added xevan

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
<<<<<<< HEAD

// cleanup
extern "C" void free_whirl(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

#ifdef SM3_VARIANT
	hipFree(d_hash[thr_id]);
	whirlpool512_free_sm3(thr_id);
#else
	x15_whirlpool_cpu_free(thr_id);
#endif
	init[thr_id] = false;

	hipDeviceSynchronize();
}

=======
>>>>>>> 8c320ca... added xevan
