#include "hip/hip_runtime.h"
/*
 * whirlpool routine (djm)
<<<<<<< HEAD
 * whirlpoolx routine (provos alexis, tpruvot)
 */
extern "C" {
#include "sph/sph_whirlpool.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS] = { 0 };

extern void whirlpoolx_cpu_init(int thr_id, uint32_t threads);
extern void whirlpoolx_cpu_free(int thr_id);
extern void whirlpoolx_setBlock_80(void *pdata, const void *ptarget);
extern uint32_t whirlpoolx_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce);
=======
 * whirlpoolx routine (provos alexis)
 */
extern "C"
{
#include "sph/sph_whirlpool.h"
#include "miner.h"
}

#include "cuda_helper.h"

extern void whirlpoolx_cpu_init(int thr_id, uint32_t threads);
extern void whirlpoolx_setBlock_80(void *pdata, const void *ptarget);
extern void cpu_whirlpoolx(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *foundNonce);
>>>>>>> 8c320ca... added xevan
extern void whirlpoolx_precompute(int thr_id);

// CPU Hash function
extern "C" void whirlxHash(void *state, const void *input)
{
<<<<<<< HEAD
=======

>>>>>>> 8c320ca... added xevan
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[64];
	unsigned char hash_xored[32];

<<<<<<< HEAD
=======
	memset(hash, 0, sizeof(hash));

>>>>>>> 8c320ca... added xevan
	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, input, 80);
	sph_whirlpool_close(&ctx_whirlpool, hash);

<<<<<<< HEAD
	// compress the 48 first bytes of the hash to 32
	for (int i = 0; i < 32; i++) {
		hash_xored[i] = hash[i] ^ hash[i + 16];
=======
    
	for (uint32_t i = 0; i < 32; i++){
	        hash_xored[i] = hash[i] ^ hash[i + 16];
>>>>>>> 8c320ca... added xevan
	}
	memcpy(state, hash_xored, 32);
}

static bool init[MAX_GPUS] = { 0 };

<<<<<<< HEAD
extern "C" int scanhash_whirlx(int thr_id,  struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	int intensity = is_windows() ? 20 : 22;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), -1);

		whirlpoolx_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpoolx_setBlock_80((void*)endiandata, ptarget);
	whirlpoolx_precompute(thr_id);

	do {
		uint32_t foundNonce = whirlpoolx_cpu_hash(thr_id, throughput, pdata[19]);

		*(hashes_done) = pdata[19] - first_nonce + throughput;

		if (foundNonce != UINT32_MAX && bench_algo < 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			whirlxHash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				work_set_target_ratio(work, vhash64);
				pdata[19] = foundNonce;
				return 1;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*(hashes_done) = pdata[19] - first_nonce;

	return 0;
}

// cleanup
extern "C" void free_whirlx(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	whirlpoolx_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
=======
int scanhash_whirlpoolx(int thr_id, uint32_t *pdata, uint32_t *ptarget, uint32_t max_nonce, uint32_t *hashes_done)
{
	uint32_t foundNonce[MAX_GPUS][4];
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, (1 << 25));
	throughput = min(throughput, max_nonce - first_nonce);
	if (opt_benchmark)
		ptarget[7] = 0x5;

	if (!init[thr_id])
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		if (!opt_cpumining) hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		whirlpoolx_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
	{
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpoolx_setBlock_80((void*)endiandata, &ptarget[6]);
	whirlpoolx_precompute(thr_id);
	do {
		cpu_whirlpoolx(thr_id, throughput, pdata[19], foundNonce[thr_id]);
//		CUDA_SAFE_CALL(hipGetLastError());
		if (foundNonce[thr_id][0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			/* check now with the CPU to confirm */
			be32enc(&endiandata[19], foundNonce[thr_id][0]);
			whirlxHash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
			/*		if (foundNonce[thr_id][1] != UINT32_MAX)
				{
					be32enc(&endiandata[19], foundNonce[thr_id][1]);
					whirlxHash(vhash64, endiandata);
					if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = foundNonce[thr_id][1];
						res++;
						if (opt_benchmark) applog(LOG_INFO, "GPU #%d: found nonce %08x", thr_id, foundNonce[thr_id][1]);
					}
					else
					{
						if (vhash64[7] != Htarg)
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce[thr_id][1]);
					}
				}
				*/

				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d: found nonce %08x", thr_id, foundNonce[thr_id][0], vhash64[7]);
				pdata[19] = foundNonce[thr_id][0];
				return res;
			}
			else
			{
				if(vhash64[7] != Htarg)
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce[thr_id][0]);
			}
		}
		pdata[19] += throughput;
	} while (!scan_abort_flag && !work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
	*hashes_done = pdata[19] - first_nonce;
	return 0;
}
>>>>>>> 8c320ca... added xevan
